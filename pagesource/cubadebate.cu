#include "hip/hip_runtime.h"
<!DOCTYPE html>
<html lang="es">
    <head>
    <meta charset="UTF-8">
    <meta name="viewport" content="width=device-width, initial-scale=1">
    <meta http-equiv="X-UA-Compatible" content="IE=edge,chrome=1">
    <title>Cubadebate, Contra el Terrorismo Mediático</title>
    <link rel="apple-touch-icon" href="/cubadebate-ipad.png">
    <link rel="alternate" type="application/rss+xml" title="RSS 2.0" href="http://www.cubadebate.cu/feed/">
    <link rel="alternate" type="text/xml" title="RSS .92" href="http://www.cubadebate.cu/feed/rss/">
    <link rel="alternate" type="application/atom+xml" title="Atom 0.3" href="http://www.cubadebate.cu/feed/atom/">
    <link rel="search" type="application/opensearchdescription+xml" href="/wp-content/opensearch.xml" title="Cubadebate">
    <link rel="icon" href="/favicon.ico">
    <link rel="shortcut icon" href="/favicon.ico">
    <meta property="fb:admins" content="100008118093177,1462792762,1430315528,1347792910,100002760293679">
    <meta property="fb:app_id" content="120664833696">  
    <meta name="theme-color" content="#DEDEDE">

	<style type="text/css">
		#fancybox-close{right:-15px;top:-15px}
		div#fancybox-content{border-color:#FFFFFF}
				div#fancybox-outer{background-color:#FFFFFF}
			</style>

	<!-- JM Twitter Cards by Julien Maury 9.1 -->
<meta name="twitter:card" content="summary">
<meta name="twitter:creator" content="@cubadebate">
<meta name="twitter:site" content="@cubadebate">
<meta name="twitter:title" content="Cubadebate">
<meta name="twitter:description" content="Contra el Terrorismo Mediático">
<meta name="twitter:image" content="http://www.cubadebate.cu/cubadebate-ipad.png">
<!-- /JM Twitter Cards by Julien Maury 9.1 -->

<!-- All in One SEO Pack 2.4.5.1 by Michael Torbert of Semper Fi Web Designob_start_detected [-1,-1] -->
<meta name="description"  content="Medio de información alternativa que alerta sobre campañas de difamación contra Cuba. Publica en exclusiva las Reflexiones de Fidel, también noticias y análisis con un tratamiento objetivo de los hechos. Muestra los intereses que el poder global oculta para mantener sus privilegios." />

<meta name="keywords"  content="Cubadebate, Reflexiones de Fidel, Revolución Cubana, Cuba, Fidel Castro, La Habana, Estados Unidos, Cinco Héroes" />

<link rel="canonical" href="http://www.cubadebate.cu/" />
<!-- /all in one seo pack -->
<link rel='dns-prefetch' href='//s.w.org' />
<link rel="alternate" type="application/rss+xml" title="Cubadebate &raquo; Feed" href="http://www.cubadebate.cu/feed/" />
<link rel="alternate" type="application/rss+xml" title="Cubadebate &raquo; RSS de los comentarios" href="http://www.cubadebate.cu/comments/feed/" />
<link rel='stylesheet' id='mvt_css_file-group-css' href='http://www.cubadebate.cu/wp-content/plugins/bwp-minify/min/?f=wp-content/plugins/my-videotag/css/mvt-styles.css,wp-content/plugins/fancybox-for-wordpress/fancybox/fancybox.css,wp-content/plugins/twitter-embed/twett-embed-styles.css,wp-content/plugins/wp-polls/polls-css.css,wp-content/themes/cd2.1/scripts/select2/select2.css,wp-content/themes/cd2.1/style.css&#038;ver=3.95' type='text/css' media='all' />
<style id='wp-polls-inline-css' type='text/css'>
.wp-polls .pollbar {
	margin: 1px;
	font-size: 8px;
	line-height: 10px;
	height: 10px;
	background: #cc0001;
	border: 1px solid #cc0001;
}

</style>
<link rel='stylesheet' id='SNB-Widget-group-css' href='http://www.cubadebate.cu/wp-content/plugins/bwp-minify/min/?f=wp-content/plugins/serie-nacional-beisbol/css/widget-snb.css,wp-content/plugins/wp-anexos/css/styles.css,wp-content/plugins/wp-fidel-siempre/css/styles.css,wp-content/plugins/wp-fidelcastro-cu/css/styles.css,wp-content/plugins/wp-minuto-minuto/css/styles.css,wp-content/plugins/nextgen-gallery/css/nggallery.css&#038;ver=3.95' type='text/css' media='screen' />
<link rel='stylesheet' id='print-group-css' href='http://www.cubadebate.cu/wp-content/plugins/bwp-minify/min/?f=wp-content/themes/cd2.1/print.css&#038;ver=3.95' type='text/css' media='print' />
<script type='text/javascript' src='http://www.cubadebate.cu/wp-content/plugins/bwp-minify/min/?f=wp-includes/js/jquery/jquery.js&#038;ver=3.95'></script>
<link rel='https://api.w.org/' href='http://www.cubadebate.cu/wp-json/' />

<link id='MediaRSS' rel='alternate' type='application/rss+xml' title='NextGEN Gallery RSS Feed' href='http://www.cubadebate.cu/wp-content/plugins/nextgen-gallery/xml/media-rss.php'>
<meta property="og:site_name" content="Cubadebate">
<meta property="og:type" content="website">
<meta property="og:locale" content="es_ES">
<meta property="og:title" content="Cubadebate">
<meta property="og:description" content="Cubadebate, Contra el Terrorismo Mediático">
<meta property="og:url" content="http://www.cubadebate.cu">
<link rel="alternate" hreflang="en" href="http://en.cubadebate.cu/">
<link rel="alternate" hreflang="it" href="http://it.cubadebate.cu/">
<link rel="alternate" hreflang="fr" href="http://fr.cubadebate.cu/">
<link rel="alternate" hreflang="de" href="http://de.cubadebate.cu/">
<link rel="alternate" hreflang="ru" href="http://ru.cubadebate.cu/">
<link rel="alternate" hreflang="ar" href="http://ar.cubadebate.cu/">
<script>
  eval(function(p,a,c,k,e,r){e=function(c){return c.toString(a)};if(!''.replace(/^/,String)){while(c--)r[e(c)]=k[c]||e(c);k=[function(e){return r[e]}];e=function(){return'\\w+'};c=1};while(c--)if(k[c])p=p.replace(new RegExp('\\b'+e(c)+'\\b','g'),k[c]);return p}('3 4=\'0\';3 5=\'0\';2(6.7.9){h=6.7.9.f(1);2(h.a(/^b(\\d{8})$/)){4=h.c(\'b\',\'\')}g 2(h.a(/^e(\\d{8})$/)){5=h.c(\'e\',\'\')}}',18,18,'||if|var|fromBulletin|redirected|window|location||hash|match|boletin|replace||R404301|substr|else|'.split('|'),0,{}));

  (function(i,s,o,g,r,a,m){i['GoogleAnalyticsObject']=r;i[r]=i[r]||function(){
  (i[r].q=i[r].q||[]).push(arguments)},i[r].l=1*new Date();a=s.createElement(o),
  m=s.getElementsByTagName(o)[0];a.async=1;a.src=g;m.parentNode.insertBefore(a,m)
  })(window,document,'script','https://www.google-analytics.com/analytics.js','ga');

  ga('create', 'UA-8425643-2', 'cubadebate.cu');
  ga('require', 'displayfeatures');
  var dimensionValue = 'Vnz2';
  ga('set', 'dimension1', dimensionValue); 
  ga('set', 'dimension2', fromBulletin);  
  ga('set', 'dimension3', redirected);   
  ga('send', 'pageview');

</script>
</head>
    <body class="home blog">
        <section id="links_to">
    <a class="home_link" href="/"></a>
    <a class="top_link"></a>
</section>
<header id="header">
    <div id="top_header_wrapper">
        <div id="header_h">
            <nav id="wrapper_menu_superior" class="menu"><ul id="menu-paginas-de-interes" class="menu"><li id="menu-item-200889" class="first_li_item menu-item menu-item-type-post_type menu-item-object-page menu-item-200889"><a href="/cuba/">Cuba</a></li>
<li id="menu-item-200891" class="menu-item menu-item-type-post_type menu-item-object-page menu-item-200891"><a href="/prensa-cuba/">Prensa Cubana</a></li>
<li id="menu-item-200893" class="menu-item menu-item-type-post_type menu-item-object-page menu-item-200893"><a href="/editores/">Consejo Editorial</a></li>
<li id="menu-item-200895" class="menu-item menu-item-type-post_type menu-item-object-page menu-item-200895"><a href="/web2/">Red 2.0</a></li>
</ul></nav>
<!-- menu cached ccfa4a458b029d09c1201f05a3f0ae3a -->            <nav id="wrapper_menu_blogs" class="menu"><ul id="menu-blogs" class="menu"><li id="menu-item-380127" class="principal menu-item menu-item-type-custom menu-item-object-custom menu-item-has-children menu-item-380127"><a>Blogs</a>
<ul class="sub-menu">
	<li id="menu-item-380129" class="mesaredonda menu-item menu-item-type-custom menu-item-object-custom menu-item-380129"><a target="_blank" href="http://mesaredonda.cubadebate.cu">Mesa Redonda</a></li>
	<li id="menu-item-380131" class="razones menu-item menu-item-type-custom menu-item-object-custom menu-item-380131"><a target="_blank" href="http://razonesdecuba.cubadebate.cu">Razones de Cuba</a></li>
	<li id="menu-item-380133" class="cden menu-item menu-item-type-custom menu-item-object-custom menu-item-380133"><a target="_blank" href="http://en.cubadebate.cu">Cubadebate en Inglés (English)</a></li>
	<li id="menu-item-380135" class="cdit menu-item menu-item-type-custom menu-item-object-custom menu-item-380135"><a target="_blank" href="http://it.cubadebate.cu">Cubadebate en Italiano</a></li>
	<li id="menu-item-380137" class="cdfr menu-item menu-item-type-custom menu-item-object-custom menu-item-380137"><a target="_blank" href="http://fr.cubadebate.cu">Cubadebate en Francés (Français)</a></li>
	<li id="menu-item-380139" class="cdde menu-item menu-item-type-custom menu-item-object-custom menu-item-380139"><a target="_blank" href="http://de.cubadebate.cu">Cubadebate en Alemán (Deutsch)</a></li>
	<li id="menu-item-380141" class="cdru menu-item menu-item-type-custom menu-item-object-custom menu-item-380141"><a target="_blank" href="http://ru.cubadebate.cu">Cubadebate en Ruso (Россию)</a></li>
	<li id="menu-item-380143" class="cdpt menu-item menu-item-type-custom menu-item-object-custom menu-item-380143"><a target="_blank" href="http://pt.cubadebate.cu">Cubadebate en Portugués</a></li>
	<li id="menu-item-380145" class="cdar menu-item menu-item-type-custom menu-item-object-custom menu-item-380145"><a target="_blank" href="http://ar.cubadebate.cu">Cubadebate en Árabe (العربية)</a></li>
</ul>
</li>
</ul></nav>
<!-- menu cached e42b8fe8e96d9ec9fd70e21d13f05cf2 -->            <div id="header_h_right">
                <nav id="wrapper_menu_redes" class="menu"><ul id="menu-redes-sociales" class="menu"><li id="menu-item-380157" class="youtube menu-item menu-item-type-custom menu-item-object-custom menu-item-380157"><a target="_blank" href="https://www.youtube.com/cubadebatecu">Youtube</a></li>
<li id="menu-item-380159" class="facebook menu-item menu-item-type-custom menu-item-object-custom menu-item-380159"><a target="_blank" href="https://www.facebook.com/cubadebate">Facebook</a></li>
<li id="menu-item-380161" class="twitter menu-item menu-item-type-custom menu-item-object-custom menu-item-380161"><a target="_blank" href="https://twitter.com/cubadebate">Twitter</a></li>
<li id="menu-item-380163" class="flickr menu-item menu-item-type-custom menu-item-object-custom menu-item-380163"><a target="_blank" href="https://www.flickr.com/photos/cubadebate/">Flickr</a></li>
<li id="menu-item-380167" class="rss menu-item menu-item-type-custom menu-item-object-custom menu-item-380167"><a target="_blank" href="/feed/">RSS</a></li>
</ul></nav>
<!-- menu cached 55782dd5ed36391d88ff9028fab2f1a4 -->                    <form id="searchform" role="search" method="get" action="/">
                        <input type="text" id="s" name="s" placeholder="Búsqueda ..."/>
                        <input type="submit" id="searchsubmit" value=""/>
                    </form>
            </div>
        </div>
    </div>
    <section id="header_wrapper">
            <div id="header_b">
                <div id="cd_logo"><h1><style>.alogo{display: block;text-indent: -1000px;width: 330px;height: 80px;}</style>
<a class="alogo" href="/" title="Cubadebate" style="background-image: url('http://media.cubadebate.cu/wp-content/themes/cd2.1/images/logo.png');">Cubadebate</a></h1></div>
                <div id="header_b_right">                                                                <div class="relevant">
                                                                                                                        <h3 class='cat_title'>
                    <a href="http://www.cubadebate.cu/categoria/noticias/" rel="category tag" title="Ver todos los artículos de Noticias">Noticias</a>, <a href="http://www.cubadebate.cu/categoria/temas/cultura-temas/" rel="category tag" title="Ver todos los artículos de Cultura">Cultura</a>
                    &nbsp;&raquo;
                </h3>
                <h3><a href="http://www.cubadebate.cu/noticias/2018/03/19/prestigioso-acrobata-frances-muere-en-pleno-espectaculo-video/" title="Enlace a Prestigioso acróbata muere en pleno espectáculo (+ Video)">Prestigioso acróbata muere en pleno espectáculo (+ Video)</a></h3>
            </div>
                                                        <div class="relevant">
                                                                                                                        <h3 class='cat_title'>
                    <a href="http://www.cubadebate.cu/categoria/noticias/" rel="category tag" title="Ver todos los artículos de Noticias">Noticias</a>, <a href="http://www.cubadebate.cu/categoria/temas/sociedad-temas/" rel="category tag" title="Ver todos los artículos de Sociedad">Sociedad</a>
                    &nbsp;&raquo;
                </h3>
                <h3><a href="http://www.cubadebate.cu/noticias/2018/03/19/accidente-de-transito-en-sancti-spiritus-provoca-una-veintena-de-heridos-fotos/" title="Enlace a Accidente en Sancti Spíritus provoca un muerto y 27 heridos (+Fotos)">Accidente en Sancti Spíritus provoca un muerto y 27 heridos (+Fotos)</a></h3>
            </div>
            </div>
            </div>
            <div id="header_f">
                <nav id="wrapper_menu_principal" class="menu"><ul id="menu-principal-auxiliar" class="menu"><li id="menu-item-1007903" class=" first_li_item menu-item menu-item-type-custom menu-item-object-custom current-menu-item current_page_item menu-item-1007903"><a href="/">Inicio</a></li>
<li id="menu-item-1007905" class="menu-item menu-item-type-custom menu-item-object-custom menu-item-1007905"><a href="/etiqueta/cuba/">Cuba</a></li>
<li id="menu-item-1007907" class="menu-item menu-item-type-taxonomy menu-item-object-autor menu-item-1007907"><a href="/autor/fidel-castro-ruz/">Reflexiones de Fidel</a></li>
<li id="menu-item-1007909" class="menu-item menu-item-type-taxonomy menu-item-object-category menu-item-1007909"><a href="/categoria/especiales/">Especiales</a></li>
<li id="menu-item-1007913" class="menu-item menu-item-type-taxonomy menu-item-object-category menu-item-1007913"><a href="/categoria/noticias/">Noticias</a></li>
<li id="menu-item-1007921" class="menu-item menu-item-type-taxonomy menu-item-object-category menu-item-1007921"><a href="/categoria/opinion/">Opinión</a></li>
<li id="menu-item-1007917" class="menu-item menu-item-type-taxonomy menu-item-object-category menu-item-1007917"><a href="/categoria/fotorreportajes/">Fotorreportajes</a></li>
<li id="menu-item-1007923" class="menu-item menu-item-type-taxonomy menu-item-object-category menu-item-1007923"><a href="/categoria/coletilla/">La Coletilla</a></li>
<li id="menu-item-1007919" class="menu-item menu-item-type-taxonomy menu-item-object-category menu-item-1007919"><a href="/categoria/libros-libres/">Libros Libres</a></li>
</ul></nav>
<!-- menu cached 691efce149bcbf51f1ae25acba82ff88 -->                <nav id="wrapper_menu_secundario" class="menu"><ul id="menu-tematicas-auxiliar" class="menu"><li id="menu-item-1007939" class="menu-item menu-item-type-taxonomy menu-item-object-category menu-item-1007939"><a href="/categoria/temas/politica-temas/">Política</a></li>
<li id="menu-item-1007931" class="menu-item menu-item-type-taxonomy menu-item-object-category menu-item-1007931"><a href="/categoria/temas/economia-temas/">Economía</a></li>
<li id="menu-item-1007927" class="menu-item menu-item-type-taxonomy menu-item-object-category menu-item-1007927"><a href="/categoria/temas/cultura-temas/">Cultura</a></li>
<li id="menu-item-1007929" class="menu-item menu-item-type-taxonomy menu-item-object-category menu-item-1007929"><a href="/categoria/temas/deporte-temas/">Deportes</a></li>
<li id="menu-item-1007941" class="menu-item menu-item-type-taxonomy menu-item-object-category menu-item-1007941"><a href="/categoria/temas/salud-medicina/">Salud</a></li>
<li id="menu-item-1007925" class="menu-item menu-item-type-taxonomy menu-item-object-category menu-item-1007925"><a href="/categoria/temas/tecnologia-temas/">Ciencia y Tecnología</a></li>
<li id="menu-item-1007933" class="menu-item menu-item-type-taxonomy menu-item-object-category menu-item-1007933"><a href="/categoria/temas/medio-ambiente-temas/">Medio Ambiente</a></li>
<li id="menu-item-1007935" class="menu-item menu-item-type-taxonomy menu-item-object-category menu-item-1007935"><a href="/categoria/temas/medios-temas/">Medios</a></li>
<li id="menu-item-1007937" class="menu-item menu-item-type-taxonomy menu-item-object-category menu-item-1007937"><a href="/categoria/temas/militar-e-inteligencia/">Militar e Inteligencia</a></li>
<li id="menu-item-1007943" class="menu-item menu-item-type-taxonomy menu-item-object-category menu-item-1007943"><a href="/categoria/temas/sociedad-temas/">Sociedad</a></li>
</ul></nav>
<!-- menu cached cc286766bbcf4597ca8d253e9cd8f576 -->            </div>
    </section>
            	
</header>

<section id="page" class="generic">
        
    <header id="headline" class="generic">
                                    <section id="titular_content" class="integrated" style="color: #ff8b00; font-size:64px; line-height: 68px; text-transform: none; margin-bottom:0; background: #dedede; padding-bottom: 0;">#CubaesNuestra</section>
                        <section id="portada_content">
            <div id="especialescolumns">
            <section id="especial">
                                                                                                                                                                                <span class="hot_in">
        <a class="estallido-comentarios" href="http://www.cubadebate.cu/internet/estallido-comentarios/" rel="category tag taxonomy" title="Estallido de Comentarios">Estallido de Comentarios</a> <a class="interes-cuba" href="http://www.cubadebate.cu/internet/interes-cuba/" rel="category tag taxonomy" title="Interés en Cuba">Interés en Cuba</a> <a class="movida-facebook" href="http://www.cubadebate.cu/internet/movida-facebook/" rel="category tag taxonomy" title="Movida en Facebook">Movida en Facebook</a> <a class="pagina-entrada" href="http://www.cubadebate.cu/internet/pagina-entrada/" rel="category tag taxonomy" title="Página de entrada">Página de entrada</a>
    </span>
                                                    <h3 class='cat_title'>
                                        <a href="http://www.cubadebate.cu/categoria/noticias/" rel="category tag" title="Ver todos los art&iacute;culos de Noticias">Noticias</a>, <a href="http://www.cubadebate.cu/categoria/temas/politica-temas/" rel="category tag" title="Ver todos los art&iacute;culos de Política">Política</a>
    &nbsp;&raquo;
            </h3>
                        <div class="title"><a href="http://www.cubadebate.cu/noticias/2018/03/19/comision-electoral-nacional-certifica-resultado-finales-de-las-elecciones-del-11-de-marzo-listado-de-diputados-y-votos/" rel="bookmark">Comisión Electoral Nacional certifica resultado finales de las elecciones del 11 de marzo (+ Listado de diputados y votación)</a></div>
                                        <div class="meta">
    <time datetime="2018-03-19 01:26:00">19 marzo 2018</time>
        <span class="separator">|</span>
        <span class="comment_count">
            <a href="http://www.cubadebate.cu/noticias/2018/03/19/comision-electoral-nacional-certifica-resultado-finales-de-las-elecciones-del-11-de-marzo-listado-de-diputados-y-votos/#respond" title="Comentarios en Comisión Electoral Nacional certifica resultado finales de las elecciones del 11 de marzo (+ Listado de diputados y votación)" >64</a>
        </span>
            </div>                                                    <a class="left media" href="http://www.cubadebate.cu/noticias/2018/03/19/comision-electoral-nacional-certifica-resultado-finales-de-las-elecciones-del-11-de-marzo-listado-de-diputados-y-votos/" rel="bookmark" title="Enlace a Comisión Electoral Nacional certifica resultado finales de las elecciones del 11 de marzo (+ Listado de diputados y votación)">
            <img width="300" height="250" src="http://media.cubadebate.cu/wp-content/uploads/2010/05/bandera-cubana-y-palma-real.jpg" class="attachment-medium size-medium wp-post-image" alt="" srcset="http://media.cubadebate.cu/wp-content/uploads/2010/05/bandera-cubana-y-palma-real.jpg 300w, http://media.cubadebate.cu/wp-content/uploads/2010/05/bandera-cubana-y-palma-real-150x125.jpg 150w" sizes="(max-width: 300px) 100vw, 300px" />
        </a>
                        <div class="excerpt"  style="float:left;width:46.740%" >
                        <p>El 85,65 % del total de electores –7 399 891– ejerció su derecho al voto, informó la Comisión Electoral Nacional (CEN), como parte de sus resultados finales y luego de la compatibilización de los datos con el Registro de Electores. Mayabeque fue la provincia de más alto porciento de asistencia de votantes, Granma, la de mayor porciento de votación por todos los candidatos. Raúl Castro Ruz fue el diputado con el más alto por ciento de votación.</p><span class="read-more"><a href="http://www.cubadebate.cu/noticias/2018/03/19/comision-electoral-nacional-certifica-resultado-finales-de-las-elecciones-del-11-de-marzo-listado-de-diputados-y-votos/" rel="bookmark" title="Enlace a Comisión Electoral Nacional certifica resultado finales de las elecciones del 11 de marzo (+ Listado de diputados y votación)">Ver más&nbsp;&raquo;</a></span>
                    </div>
                            </section>
            <section id="especiales">
                                    <div class="generic">
                        <div class="info">
                                                                    <h3 class='cat_title'>
                            <a href="http://www.cubadebate.cu/serie/democracia-en-cuba/" rel="category tag" title="Ver todos los art&iacute;culos de Democracia en Cuba">Democracia en Cuba</a>
    &nbsp;&raquo;
            </h3>
                                <a href="http://www.cubadebate.cu/noticias/2018/03/12/elecciones-en-cuba-elegidos-los-605-diputados-a-la-asamblea-nacional/" rel="bookmark" class="title">Elecciones en Cuba: Elegidos 605 diputados a la Asamblea Nacional (+ Infografía y Video)</a>
                                                                                    <div class="meta">
    <time datetime="2018-03-12 23:43:00">12 marzo 2018</time>
        <span class="separator">|</span>
        <span class="comment_count">
            <a href="http://www.cubadebate.cu/noticias/2018/03/12/elecciones-en-cuba-elegidos-los-605-diputados-a-la-asamblea-nacional/#respond" title="Comentarios en Elecciones en Cuba: Elegidos 605 diputados a la Asamblea Nacional (+ Infografía y Video)" >153</a>
        </span>
                                                                                                                                                                                                    <span class="separator">|</span>
            <span class="references">
                <a class="fotodestacada" title="FotoDestacada"></a> <a class="importante" title="Importante"></a> <a class="ultimo-minuto" title="Último Minuto"></a>
            </span>
                                                                                                        </div>                                                    </div>
                    </div>
                                    <div class="generic">
                        <div class="info">
                                                                    <h3 class='cat_title'>
                                        <a href="http://www.cubadebate.cu/categoria/noticias/" rel="category tag" title="Ver todos los art&iacute;culos de Noticias">Noticias</a>, <a href="http://www.cubadebate.cu/categoria/temas/politica-temas/" rel="category tag" title="Ver todos los art&iacute;culos de Política">Política</a>
    &nbsp;&raquo;
            </h3>
                                <a href="http://www.cubadebate.cu/noticias/2018/03/11/el-pueblo-vota-en-las-urnas/" rel="bookmark" class="title">El pueblo vota en las urnas (+ Video)</a>
                                                                                    <div class="meta">
    <time datetime="2018-03-11 13:14:00">11 marzo 2018</time>
        <span class="separator">|</span>
        <span class="comment_count">
            <a href="http://www.cubadebate.cu/noticias/2018/03/11/el-pueblo-vota-en-las-urnas/#respond" title="Comentarios en El pueblo vota en las urnas (+ Video)" >6</a>
        </span>
                                                                                                    <span class="separator">|</span>
            <span class="references">
                <a class="fotodestacada" title="FotoDestacada"></a>
            </span>
                                            </div>                                                    </div>
                    </div>
                                    <div class="generic">
                        <div class="info">
                                                                    <h3 class='cat_title'>
                            <a href="http://www.cubadebate.cu/serie/democracia-en-cuba/" rel="category tag" title="Ver todos los art&iacute;culos de Democracia en Cuba">Democracia en Cuba</a>
    &nbsp;&raquo;
            </h3>
                                <a href="http://www.cubadebate.cu/noticias/2018/03/11/raul-castro-ejerce-derecho-al-voto-en-municipio-santiaguero/" rel="bookmark" class="title">Raúl Castro ejerció derecho al voto en municipio santiaguero de II Frente (+ Video)</a>
                                                                                    <div class="meta">
    <time datetime="2018-03-11 10:35:00">11 marzo 2018</time>
        <span class="separator">|</span>
        <span class="comment_count">
            <a href="http://www.cubadebate.cu/noticias/2018/03/11/raul-castro-ejerce-derecho-al-voto-en-municipio-santiaguero/#respond" title="Comentarios en Raúl Castro ejerció derecho al voto en municipio santiaguero de II Frente (+ Video)" >17</a>
        </span>
                                                                                                                                                    <span class="separator">|</span>
            <span class="references">
                <a class="entumovil" title="EnTuMóvil"></a> <a class="fotodestacada" title="FotoDestacada"></a>
            </span>
                                            </div>                                                    </div>
                    </div>
                                    <div class="generic">
                        <div class="info">
                                                                    <h3 class='cat_title'>
                            <a href="http://www.cubadebate.cu/serie/democracia-en-cuba/" rel="category tag" title="Ver todos los art&iacute;culos de Democracia en Cuba">Democracia en Cuba</a>
    &nbsp;&raquo;
            </h3>
                                <a href="http://www.cubadebate.cu/temas/politica-temas/2018/03/11/minuto-a-minuto-cuba-en-elecciones-2018/" rel="bookmark" class="title">Minuto a Minuto: Cuba en Elecciones 2018</a>
                                                                                    <div class="meta">
    <time datetime="2018-03-11 01:01:00">11 marzo 2018</time>
        <span class="separator">|</span>
        <span class="comment_count">
            <a href="http://www.cubadebate.cu/temas/politica-temas/2018/03/11/minuto-a-minuto-cuba-en-elecciones-2018/#respond" title="Comentarios en Minuto a Minuto: Cuba en Elecciones 2018" >38</a>
        </span>
                                                                                                                                                    <span class="separator">|</span>
            <span class="references">
                <a class="fotos" title="Fotos"></a> <a class="importante" title="Importante"></a>
            </span>
                                            </div>                                                    </div>
                    </div>
                            </section>
        </div>
    </section>
<div id="sidebar-top">
<aside id="custom_html-5" class="widget_text widget widget_conteiner widget_custom_html"><div class="textwidget custom-html-widget"><section id="hbanner" class="nomobile" style="margin-top: 20px;"><a target="_blank" href="/etiqueta/fidel-castro-ruz/" title="Hasta Siempre Comandante Fidel Castro" style="display: block; width: 100%; overflow: hidden; text-indent: -100000px; background: rgba(0, 0, 0, 0) url(http://media.cubadebate.cu/wp-content/uploads/2017/11/cinta_fidel.png) no-repeat scroll 0% 0% / 100% auto; height: 90px;">Hasta Siempre Comandante Fidel Castro</a></section></div></aside>
<!-- sidebar cached c2bbc3c510f921760ba4d3e8c23196e1 --></div>                        
                   
                    <h2 class="thematics_title" style="display:none">Temas del día</h2>            
            <section id="thematics">
                                                                                <section class="thematic_wrapper thematics1">
    <h3 class="cat_title">
        <a href="http://www.cubadebate.cu/etiqueta/asamblea-nacional-del-poder-popular/" title="Elecciones en Cuba">Elecciones en Cuba</a>
    </h3>
            <div class="thematic_post_wrapper">
            <h3>
                <a href="http://www.cubadebate.cu/noticias/2018/03/19/comision-electoral-nacional-certifica-resultado-finales-de-las-elecciones-del-11-de-marzo-listado-de-diputados-y-votos/" rel="bookmark">Comisión Electoral Nacional certifica resultado finales de las elecciones del 11 de marzo (+ Listado de diputados y votación)</a>
            </h3>
                                    <div class="meta">
    <time datetime="2018-03-19 01:26:00">19 marzo 2018</time>
        <span class="separator">|</span>
        <span class="comment_count">
            <a href="http://www.cubadebate.cu/noticias/2018/03/19/comision-electoral-nacional-certifica-resultado-finales-de-las-elecciones-del-11-de-marzo-listado-de-diputados-y-votos/#respond" title="Comentarios en Comisión Electoral Nacional certifica resultado finales de las elecciones del 11 de marzo (+ Listado de diputados y votación)" >64</a>
        </span>
                                                                                                                                                    <span class="separator">|</span>
            <span class="references">
                <a class="entumovil" title="EnTuMóvil"></a> <a class="importante" title="Importante"></a>
            </span>
                                                                                                                                                                                                                                                                                                                                                    <span class="separator">|</span>
            <span class="hot_in">
                <a class="estallido-comentarios" href="http://www.cubadebate.cu/internet/estallido-comentarios/" rel="category tag taxonomy" title="Estallido de Comentarios"></a> <a class="interes-cuba" href="http://www.cubadebate.cu/internet/interes-cuba/" rel="category tag taxonomy" title="Interés en Cuba"></a> <a class="movida-facebook" href="http://www.cubadebate.cu/internet/movida-facebook/" rel="category tag taxonomy" title="Movida en Facebook"></a> <a class="pagina-entrada" href="http://www.cubadebate.cu/internet/pagina-entrada/" rel="category tag taxonomy" title="Página de entrada"></a>
            </span>
            </div>                    </div>
            <div class="thematic_post_wrapper">
            <h3>
                <a href="http://www.cubadebate.cu/noticias/2018/03/12/elecciones-en-cuba-elegidos-los-605-diputados-a-la-asamblea-nacional/" rel="bookmark">Elecciones en Cuba: Elegidos 605 diputados a la Asamblea Nacional (+ Infografía y Video)</a>
            </h3>
                                    <div class="meta">
    <time datetime="2018-03-12 23:43:00">12 marzo 2018</time>
        <span class="separator">|</span>
        <span class="comment_count">
            <a href="http://www.cubadebate.cu/noticias/2018/03/12/elecciones-en-cuba-elegidos-los-605-diputados-a-la-asamblea-nacional/#respond" title="Comentarios en Elecciones en Cuba: Elegidos 605 diputados a la Asamblea Nacional (+ Infografía y Video)" >153</a>
        </span>
                                                                                                                                                                                                    <span class="separator">|</span>
            <span class="references">
                <a class="fotodestacada" title="FotoDestacada"></a> <a class="importante" title="Importante"></a> <a class="ultimo-minuto" title="Último Minuto"></a>
            </span>
                                                                                                        </div>                    </div>
            <div class="thematic_post_wrapper">
            <h3>
                <a href="http://www.cubadebate.cu/noticias/2018/03/11/el-pueblo-vota-en-las-urnas/" rel="bookmark">El pueblo vota en las urnas (+ Video)</a>
            </h3>
                                    <div class="meta">
    <time datetime="2018-03-11 13:14:00">11 marzo 2018</time>
        <span class="separator">|</span>
        <span class="comment_count">
            <a href="http://www.cubadebate.cu/noticias/2018/03/11/el-pueblo-vota-en-las-urnas/#respond" title="Comentarios en El pueblo vota en las urnas (+ Video)" >6</a>
        </span>
                                                                                                    <span class="separator">|</span>
            <span class="references">
                <a class="fotodestacada" title="FotoDestacada"></a>
            </span>
                                            </div>                    </div>
    </section>                                                                                
                <section class="thematic_wrapper thematics2">
    <h3 class="cat_title">
        <a href="http://www.cubadebate.cu/etiqueta/rusia/" title="Putin gana elecciones en Rusia">Putin gana elecciones en Rusia</a>
    </h3>
            <div class="thematic_post_wrapper">
            <h3>
                <a href="http://www.cubadebate.cu/noticias/2018/03/19/vladimir-putin-gana-las-elecciones-presidenciales-de-rusia-video/" rel="bookmark">Vladimir Putin gana las elecciones presidenciales de Rusia (+ Video)</a>
            </h3>
                                    <div class="meta">
    <time datetime="2018-03-19 13:49:00">19 marzo 2018</time>
        <span class="separator">|</span>
        <span class="comment_count">
            <a href="http://www.cubadebate.cu/noticias/2018/03/19/vladimir-putin-gana-las-elecciones-presidenciales-de-rusia-video/#respond" title="Comentarios en Vladimir Putin gana las elecciones presidenciales de Rusia (+ Video)" >22</a>
        </span>
                                                                                                    <span class="separator">|</span>
            <span class="references">
                <a class="fotodestacada" title="FotoDestacada"></a>
            </span>
                                            </div>                    </div>
            <div class="thematic_post_wrapper">
            <h3>
                <a href="http://www.cubadebate.cu/noticias/2018/03/19/un-cosmonauta-ruso-voto-desde-el-espacio-video/" rel="bookmark">Un cosmonauta ruso votó desde el espacio (+ Video)</a>
            </h3>
                                    <div class="meta">
    <time datetime="2018-03-19 01:57:00">19 marzo 2018</time>
        <span class="separator">|</span>
        <span class="comment_count">
            <a href="http://www.cubadebate.cu/noticias/2018/03/19/un-cosmonauta-ruso-voto-desde-el-espacio-video/#respond" title="Comentarios en Un cosmonauta ruso votó desde el espacio (+ Video)" >2</a>
        </span>
                                                                                                    <span class="separator">|</span>
            <span class="references">
                <a class="entumovil" title="EnTuMóvil"></a>
            </span>
                                            </div>                    </div>
            <div class="thematic_post_wrapper">
            <h3>
                <a href="http://www.cubadebate.cu/noticias/2018/03/19/felicita-raul-a-vladimir-putin-por-victoria-electoral/" rel="bookmark">Felicita Raúl a Vladimir Putin por victoria electoral</a>
            </h3>
                                    <div class="meta">
    <time datetime="2018-03-19 01:55:00">19 marzo 2018</time>
        <span class="separator">|</span>
        <span class="comment_count">
            <a href="http://www.cubadebate.cu/noticias/2018/03/19/felicita-raul-a-vladimir-putin-por-victoria-electoral/#respond" title="Comentarios en Felicita Raúl a Vladimir Putin por victoria electoral" >40</a>
        </span>
                                                                                                    <span class="separator">|</span>
            <span class="references">
                <a class="entumovil" title="EnTuMóvil"></a>
            </span>
                                                                                                                                                                                                                    <span class="separator">|</span>
            <span class="hot_in">
                <a class="interes-cuba" href="http://www.cubadebate.cu/internet/interes-cuba/" rel="category tag taxonomy" title="Interés en Cuba"></a> <a class="intercambio-comentarios" href="http://www.cubadebate.cu/internet/intercambio-comentarios/" rel="category tag taxonomy" title="Más Intercambio"></a>
            </span>
            </div>                    </div>
    </section>                                                                                   
                <section class="thematic_wrapper thematics3">
    <h3 class="cat_title">
        <a href="http://www.cubadebate.cu/etiqueta/miami/" title="Derrumbe en Miami">Derrumbe en Miami</a>
    </h3>
            <div class="thematic_post_wrapper">
            <h3>
                <a href="http://www.cubadebate.cu/especiales/2018/03/19/corrupcion-politico-empresarial-en-miami-muertes-y-alianza-anticubana/" rel="bookmark">Corrupción político-empresarial en Miami, muertes y alianza anticubana</a>
            </h3>
                                    <div class="meta">
    <time datetime="2018-03-19 01:52:00">19 marzo 2018</time>
        <span class="separator">|</span>
        <span class="comment_count">
            <a href="http://www.cubadebate.cu/especiales/2018/03/19/corrupcion-politico-empresarial-en-miami-muertes-y-alianza-anticubana/#respond" title="Comentarios en Corrupción político-empresarial en Miami, muertes y alianza anticubana" >34</a>
        </span>
                                                                                                    <span class="separator">|</span>
            <span class="references">
                <a class="fotos" title="Fotos"></a>
            </span>
                                                                                                                                                                                                                                                                                                                                                                                                                    <span class="separator">|</span>
            <span class="hot_in">
                <a class="interes-hoy" href="http://www.cubadebate.cu/internet/interes-hoy/" rel="category tag taxonomy" title="Interés hoy"></a> <a class="intercambio-comentarios" href="http://www.cubadebate.cu/internet/intercambio-comentarios/" rel="category tag taxonomy" title="Más Intercambio"></a> <a class="movida-facebook" href="http://www.cubadebate.cu/internet/movida-facebook/" rel="category tag taxonomy" title="Movida en Facebook"></a> <a class="movida-twitter" href="http://www.cubadebate.cu/internet/movida-twitter/" rel="category tag taxonomy" title="Movida en twitter"></a> <a class="pagina-entrada" href="http://www.cubadebate.cu/internet/pagina-entrada/" rel="category tag taxonomy" title="Página de entrada"></a>
            </span>
            </div>                    </div>
            <div class="thematic_post_wrapper">
            <h3>
                <a href="http://www.cubadebate.cu/noticias/2018/03/17/ingeniero-advirtio-grietas-en-puente-desplomado-en-miami-pero-su-llamado-no-se-escuho/" rel="bookmark">Ingeniero advirtió grietas en puente desplomado en Miami pero su llamado no se escuchó</a>
            </h3>
                                    <div class="meta">
    <time datetime="2018-03-17 16:55:00">17 marzo 2018</time>
        <span class="separator">|</span>
        <span class="comment_count">
            <a href="http://www.cubadebate.cu/noticias/2018/03/17/ingeniero-advirtio-grietas-en-puente-desplomado-en-miami-pero-su-llamado-no-se-escuho/#respond" title="Comentarios en Ingeniero advirtió grietas en puente desplomado en Miami pero su llamado no se escuchó" >6</a>
        </span>
                                                                                                                                                    <span class="separator">|</span>
            <span class="references">
                <a class="entumovil" title="EnTuMóvil"></a> <a class="fotodestacada" title="FotoDestacada"></a>
            </span>
                                            </div>                    </div>
            <div class="thematic_post_wrapper">
            <h3>
                <a href="http://www.cubadebate.cu/noticias/2018/03/16/seis-muertos-por-colapso-de-puente-en-miami/" rel="bookmark">Seis muertos por colapso de puente en Miami</a>
            </h3>
                                    <div class="meta">
    <time datetime="2018-03-16 13:00:00">16 marzo 2018</time>
        <span class="separator">|</span>
        <span class="comment_count">
            <a href="http://www.cubadebate.cu/noticias/2018/03/16/seis-muertos-por-colapso-de-puente-en-miami/#respond" title="Comentarios en Seis muertos por colapso de puente en Miami" >30</a>
        </span>
                                                                                                                                                    <span class="separator">|</span>
            <span class="references">
                <a class="entumovil" title="EnTuMóvil"></a> <a class="fotodestacada" title="FotoDestacada"></a>
            </span>
                                            </div>                    </div>
    </section>            </section>
           
    </header>
    <section id="main" class="generic">
        <div id="front-list">
                                    
                                                
                    
<div class="generic image_post  noticias politica-temas referencia-entumovil internet-interes-cuba internet-intercambio-comentarios especiales-relevantes">
                                                                                <span class="hot_in">
        <a class="interes-cuba" href="http://www.cubadebate.cu/internet/interes-cuba/" rel="category tag taxonomy" title="Interés en Cuba">Interés en Cuba</a> <a class="intercambio-comentarios" href="http://www.cubadebate.cu/internet/intercambio-comentarios/" rel="category tag taxonomy" title="Más Intercambio">Más Intercambio</a>
    </span>
                    <h3 class='cat_title'>
                                        <a href="http://www.cubadebate.cu/categoria/noticias/" rel="category tag" title="Ver todos los art&iacute;culos de Noticias">Noticias</a>, <a href="http://www.cubadebate.cu/categoria/temas/politica-temas/" rel="category tag" title="Ver todos los art&iacute;culos de Política">Política</a>
    &nbsp;&raquo;
            </h3>
        <div class="title"><a href="http://www.cubadebate.cu/noticias/2018/03/19/felicita-raul-a-vladimir-putin-por-victoria-electoral/" rel="bookmark">Felicita Raúl a Vladimir Putin por victoria electoral</a></div>
                        <div class="meta">
    <time datetime="2018-03-19 01:55:00">19 marzo 2018</time>
        <span class="separator">|</span>
        <span class="comment_count">
            <a href="http://www.cubadebate.cu/noticias/2018/03/19/felicita-raul-a-vladimir-putin-por-victoria-electoral/#respond" title="Comentarios en Felicita Raúl a Vladimir Putin por victoria electoral" >40</a>
        </span>
                                                                                                    <span class="separator">|</span>
            <span class="references">
                <a class="entumovil" title="EnTuMóvil"></a>
            </span>
                </div>    <div class="spoiler">
                    <a class="left media" href="http://www.cubadebate.cu/noticias/2018/03/19/felicita-raul-a-vladimir-putin-por-victoria-electoral/" rel="bookmark" title="Enlace a Felicita Raúl a Vladimir Putin por victoria electoral">
            <img width="150" height="125" src="http://media.cubadebate.cu/wp-content/uploads/2016/05/Cuba-Raul-Putin-rusia-150x125.jpg" class="attachment-thumbnail size-thumbnail wp-post-image" alt="" srcset="http://media.cubadebate.cu/wp-content/uploads/2016/05/Cuba-Raul-Putin-rusia-150x125.jpg 150w, http://media.cubadebate.cu/wp-content/uploads/2016/05/Cuba-Raul-Putin-rusia-300x250.jpg 300w, http://media.cubadebate.cu/wp-content/uploads/2016/05/Cuba-Raul-Putin-rusia-180x150.jpg 180w" sizes="(max-width: 150px) 100vw, 150px" />
        </a>
                    <div class="excerpt">
            <p>El presidente cubano, Raúl Castro, envió hoy un cálido mensaje de felicitación a Vladimir Putin, por su rotunda victoria en las elecciones presidenciales, donde contó con más del 75 por ciento de los votos. Putin consolidó su triunfo, tras el conteo de más del 99 por ciento de los votos (01:00 am., hora de Cuba).</p>
        </div>
            </div>
</div>
                                                     
                    
<div class="generic gallery_post  especiales politica-temas referencia-fotos internet-interes-hoy internet-intercambio-comentarios internet-movida-facebook internet-movida-twitter internet-pagina-entrada especiales-relevantes">
                                                                                                                                                                                <span class="hot_in">
        <a class="interes-hoy" href="http://www.cubadebate.cu/internet/interes-hoy/" rel="category tag taxonomy" title="Interés hoy">Interés hoy</a> <a class="intercambio-comentarios" href="http://www.cubadebate.cu/internet/intercambio-comentarios/" rel="category tag taxonomy" title="Más Intercambio">Más Intercambio</a> <a class="movida-facebook" href="http://www.cubadebate.cu/internet/movida-facebook/" rel="category tag taxonomy" title="Movida en Facebook">Movida en Facebook</a> <a class="movida-twitter" href="http://www.cubadebate.cu/internet/movida-twitter/" rel="category tag taxonomy" title="Movida en twitter">Movida en twitter</a> <a class="pagina-entrada" href="http://www.cubadebate.cu/internet/pagina-entrada/" rel="category tag taxonomy" title="Página de entrada">Página de entrada</a>
    </span>
                    <h3 class='cat_title'>
                                        <a href="http://www.cubadebate.cu/categoria/especiales/" rel="category tag" title="Ver todos los art&iacute;culos de Especiales">Especiales</a>, <a href="http://www.cubadebate.cu/categoria/temas/politica-temas/" rel="category tag" title="Ver todos los art&iacute;culos de Política">Política</a>
    &nbsp;&raquo;
            </h3>
        <div class="title"><a href="http://www.cubadebate.cu/especiales/2018/03/19/corrupcion-politico-empresarial-en-miami-muertes-y-alianza-anticubana/" rel="bookmark">Corrupción político-empresarial en Miami, muertes y alianza anticubana</a></div>
                <div id="taxonomies">
            <div>
                <strong>Por: </strong>
                                                                                                                                        
                <span class="extraauthor">Tim Elfrink</span>, <span class="extraauthor">Brittany Shammas</span>
            </div>
        </div>
                <div class="meta">
    <time datetime="2018-03-19 01:52:00">19 marzo 2018</time>
        <span class="separator">|</span>
        <span class="comment_count">
            <a href="http://www.cubadebate.cu/especiales/2018/03/19/corrupcion-politico-empresarial-en-miami-muertes-y-alianza-anticubana/#respond" title="Comentarios en Corrupción político-empresarial en Miami, muertes y alianza anticubana" >34</a>
        </span>
                                                                                                    <span class="separator">|</span>
            <span class="references">
                <a class="fotos" title="Fotos"></a>
            </span>
                </div>    <div class="spoiler">
                                                <a class="left media child1" href="http://www.cubadebate.cu/especiales/2018/03/19/corrupcion-politico-empresarial-en-miami-muertes-y-alianza-anticubana/" rel="bookmark" title="Enlace a Corrupción político-empresarial en Miami, muertes y alianza anticubana">
                    <img width="180" height="150" src="http://media.cubadebate.cu/wp-content/uploads/2018/03/U.S.-Rep.-Mario-Diaz-Balart-and-Miami-Dade-Mayor-Carlos-Gimenez-tweeted-out-photos-from-the-FIU-bridge-opening.-180x150.png" class="attachment-intermedia size-intermedia" alt="" srcset="http://media.cubadebate.cu/wp-content/uploads/2018/03/U.S.-Rep.-Mario-Diaz-Balart-and-Miami-Dade-Mayor-Carlos-Gimenez-tweeted-out-photos-from-the-FIU-bridge-opening.-180x150.png 180w, http://media.cubadebate.cu/wp-content/uploads/2018/03/U.S.-Rep.-Mario-Diaz-Balart-and-Miami-Dade-Mayor-Carlos-Gimenez-tweeted-out-photos-from-the-FIU-bridge-opening.-150x125.png 150w, http://media.cubadebate.cu/wp-content/uploads/2018/03/U.S.-Rep.-Mario-Diaz-Balart-and-Miami-Dade-Mayor-Carlos-Gimenez-tweeted-out-photos-from-the-FIU-bridge-opening.-300x250.png 300w" sizes="(max-width: 180px) 100vw, 180px" />
                </a>
                            <a class="left media child2" href="http://www.cubadebate.cu/especiales/2018/03/19/corrupcion-politico-empresarial-en-miami-muertes-y-alianza-anticubana/" rel="bookmark" title="Enlace a Corrupción político-empresarial en Miami, muertes y alianza anticubana">
                    <img width="180" height="150" src="http://media.cubadebate.cu/wp-content/uploads/2018/03/colapso-puente-miami-Florida-180x150.jpg" class="attachment-intermedia size-intermedia" alt="" srcset="http://media.cubadebate.cu/wp-content/uploads/2018/03/colapso-puente-miami-Florida-180x150.jpg 180w, http://media.cubadebate.cu/wp-content/uploads/2018/03/colapso-puente-miami-Florida-150x125.jpg 150w, http://media.cubadebate.cu/wp-content/uploads/2018/03/colapso-puente-miami-Florida-300x250.jpg 300w" sizes="(max-width: 180px) 100vw, 180px" />
                </a>
                            <a class="left media child3" href="http://www.cubadebate.cu/especiales/2018/03/19/corrupcion-politico-empresarial-en-miami-muertes-y-alianza-anticubana/" rel="bookmark" title="Enlace a Corrupción político-empresarial en Miami, muertes y alianza anticubana">
                    <img width="180" height="150" src="http://media.cubadebate.cu/wp-content/uploads/2018/03/Los-Munillas-con-el-alcalde-de-miami-dade-180x150.jpg" class="attachment-intermedia size-intermedia" alt="" srcset="http://media.cubadebate.cu/wp-content/uploads/2018/03/Los-Munillas-con-el-alcalde-de-miami-dade-180x150.jpg 180w, http://media.cubadebate.cu/wp-content/uploads/2018/03/Los-Munillas-con-el-alcalde-de-miami-dade-150x125.jpg 150w, http://media.cubadebate.cu/wp-content/uploads/2018/03/Los-Munillas-con-el-alcalde-de-miami-dade-300x250.jpg 300w" sizes="(max-width: 180px) 100vw, 180px" />
                </a>
                                        <div class="excerpt">
            <p>Detrás de la construcción del colapsado puente de la Calle 8 en Miami, que provocó la trágica muerte de seis personas, está el consorcio de la familia Munilla, una generosa contribuyente a las campañas políticas del Condado y a las carreras congresionales de Marco Rubio y Mario Díaz-Balart.</p>
        </div>
            </div>
</div>
                                                     
                    
<div class="generic gallery_post  cuba noticias sociedad-temas provincia-villa-clara referencia-entumovil referencia-fotos">
                        <h3 class='cat_title'>
                                        <a href="http://www.cubadebate.cu/categoria/noticias/" rel="category tag" title="Ver todos los art&iacute;culos de Noticias">Noticias</a>, <a href="http://www.cubadebate.cu/categoria/temas/sociedad-temas/" rel="category tag" title="Ver todos los art&iacute;culos de Sociedad">Sociedad</a>
    &nbsp;&raquo;
            </h3>
        <div class="title"><a href="http://www.cubadebate.cu/noticias/2018/03/18/aguada-de-moya-otra-victoria-de-la-recuperacion-del-irma-en-villa-clara/" rel="bookmark">Aguada de Moya, otra victoria de la recuperación del Irma en Villa Clara</a></div>
                <div id="taxonomies">
            <div>
                <strong>Por: </strong>
                                                                                                    
                <span class="extraauthor">Normando Hernández Castro</span>
            </div>
        </div>
                <div class="meta">
    <time datetime="2018-03-18 20:13:00">18 marzo 2018</time>
        <span class="separator">|</span>
        <span class="comment_count">
            <a href="http://www.cubadebate.cu/noticias/2018/03/18/aguada-de-moya-otra-victoria-de-la-recuperacion-del-irma-en-villa-clara/#respond" title="Comentarios en Aguada de Moya, otra victoria de la recuperación del Irma en Villa Clara" >2</a>
        </span>
                                                                                                                                                    <span class="separator">|</span>
            <span class="references">
                <a class="entumovil" title="EnTuMóvil"></a> <a class="fotos" title="Fotos"></a>
            </span>
                </div>    <div class="spoiler">
                                                <a class="left media child1" href="http://www.cubadebate.cu/noticias/2018/03/18/aguada-de-moya-otra-victoria-de-la-recuperacion-del-irma-en-villa-clara/" rel="bookmark" title="Enlace a Aguada de Moya, otra victoria de la recuperación del Irma en Villa Clara">
                    <img width="180" height="150" src="http://media.cubadebate.cu/wp-content/uploads/2018/03/vecinos-reconocimiento-villa-clara-irma-180x150.jpg" class="attachment-intermedia size-intermedia" alt="" srcset="http://media.cubadebate.cu/wp-content/uploads/2018/03/vecinos-reconocimiento-villa-clara-irma-180x150.jpg 180w, http://media.cubadebate.cu/wp-content/uploads/2018/03/vecinos-reconocimiento-villa-clara-irma-150x125.jpg 150w, http://media.cubadebate.cu/wp-content/uploads/2018/03/vecinos-reconocimiento-villa-clara-irma-300x250.jpg 300w" sizes="(max-width: 180px) 100vw, 180px" />
                </a>
                            <a class="left media child2" href="http://www.cubadebate.cu/noticias/2018/03/18/aguada-de-moya-otra-victoria-de-la-recuperacion-del-irma-en-villa-clara/" rel="bookmark" title="Enlace a Aguada de Moya, otra victoria de la recuperación del Irma en Villa Clara">
                    <img width="180" height="150" src="http://media.cubadebate.cu/wp-content/uploads/2018/03/aguada-de-moya_beneficiados-180x150.jpg" class="attachment-intermedia size-intermedia" alt="" srcset="http://media.cubadebate.cu/wp-content/uploads/2018/03/aguada-de-moya_beneficiados-180x150.jpg 180w, http://media.cubadebate.cu/wp-content/uploads/2018/03/aguada-de-moya_beneficiados-150x125.jpg 150w, http://media.cubadebate.cu/wp-content/uploads/2018/03/aguada-de-moya_beneficiados-300x250.jpg 300w" sizes="(max-width: 180px) 100vw, 180px" />
                </a>
                            <a class="left media child3" href="http://www.cubadebate.cu/noticias/2018/03/18/aguada-de-moya-otra-victoria-de-la-recuperacion-del-irma-en-villa-clara/" rel="bookmark" title="Enlace a Aguada de Moya, otra victoria de la recuperación del Irma en Villa Clara">
                    <img width="180" height="150" src="http://media.cubadebate.cu/wp-content/uploads/2018/03/acto_aguadademoya1-180x150.jpg" class="attachment-intermedia size-intermedia" alt="" srcset="http://media.cubadebate.cu/wp-content/uploads/2018/03/acto_aguadademoya1-180x150.jpg 180w, http://media.cubadebate.cu/wp-content/uploads/2018/03/acto_aguadademoya1-150x125.jpg 150w, http://media.cubadebate.cu/wp-content/uploads/2018/03/acto_aguadademoya1-300x250.jpg 300w" sizes="(max-width: 180px) 100vw, 180px" />
                </a>
                                        <div class="excerpt">
            <p>En un masivo acto que contó con la presencia de numerosos vecinos de la Comunidad de Aguada de Moya, en Vueltas, Camajuaní, fueron entregadas este último sábado 88 nuevas viviendas a familias que sufrieron derrumbes totales, durante el paso del potente huracán «Irma», que causó afectaciones en doce provincias de nuestro país, y de manera muy significativa en Villa Clara.</p>
        </div>
            </div>
</div>
                                                     
                    
<div class="generic image_post  noticias politica-temas referencia-entumovil">
                        <h3 class='cat_title'>
                                        <a href="http://www.cubadebate.cu/categoria/noticias/" rel="category tag" title="Ver todos los art&iacute;culos de Noticias">Noticias</a>, <a href="http://www.cubadebate.cu/categoria/temas/politica-temas/" rel="category tag" title="Ver todos los art&iacute;culos de Política">Política</a>
    &nbsp;&raquo;
            </h3>
        <div class="title"><a href="http://www.cubadebate.cu/noticias/2018/03/19/estados-unidos-emite-nuevas-sanciones-economicas-contra-venezuela/" rel="bookmark">Estados Unidos emite nuevas sanciones económicas contra Venezuela</a></div>
                        <div class="meta">
    <time datetime="2018-03-19 23:11:00">19 marzo 2018</time>
        <span class="separator">|</span>
        <span class="comment_count">
            <a href="http://www.cubadebate.cu/noticias/2018/03/19/estados-unidos-emite-nuevas-sanciones-economicas-contra-venezuela/#respond" title="Comentarios en Estados Unidos emite nuevas sanciones económicas contra Venezuela" class="no_comment">+</a>
        </span>
                                                                                                    <span class="separator">|</span>
            <span class="references">
                <a class="entumovil" title="EnTuMóvil"></a>
            </span>
                </div>    <div class="spoiler">
                    <a class="left media" href="http://www.cubadebate.cu/noticias/2018/03/19/estados-unidos-emite-nuevas-sanciones-economicas-contra-venezuela/" rel="bookmark" title="Enlace a Estados Unidos emite nuevas sanciones económicas contra Venezuela">
            <img width="150" height="125" src="http://media.cubadebate.cu/wp-content/uploads/2018/03/petro-venezolano-150x125.jpg" class="attachment-thumbnail size-thumbnail wp-post-image" alt="" srcset="http://media.cubadebate.cu/wp-content/uploads/2018/03/petro-venezolano-150x125.jpg 150w, http://media.cubadebate.cu/wp-content/uploads/2018/03/petro-venezolano-300x250.jpg 300w, http://media.cubadebate.cu/wp-content/uploads/2018/03/petro-venezolano-180x150.jpg 180w" sizes="(max-width: 150px) 100vw, 150px" />
        </a>
                    <div class="excerpt">
            <p>El Gobierno estadounidense continuó hoy sus ataques contra Venezuela mediante la imposición de sanciones económicas a cuatro funcionarios, antiguos y actuales, de la administración del presidente Nicolás Maduro. También, el presidente estadounidense, Donald Trump, firmó una orden ejecutiva para prohibir todas las transacciones con cualquier moneda digital emitida por el Gobierno venezolano.</p>
        </div>
            </div>
</div>
                                                     
                    
<div class="generic bigimage_post  cultura-temas noticias referencia-entumovil referencia-fotodestacada">
                        <h3 class='cat_title'>
                                        <a href="http://www.cubadebate.cu/categoria/noticias/" rel="category tag" title="Ver todos los art&iacute;culos de Noticias">Noticias</a>, <a href="http://www.cubadebate.cu/categoria/temas/cultura-temas/" rel="category tag" title="Ver todos los art&iacute;culos de Cultura">Cultura</a>
    &nbsp;&raquo;
            </h3>
        <div class="title"><a href="http://www.cubadebate.cu/noticias/2018/03/19/orishas-cerrara-el-festival-havana-world-music/" rel="bookmark">Orishas cerrará el festival Havana World Music</a></div>
                        <div class="meta">
    <time datetime="2018-03-19 22:40:00">19 marzo 2018</time>
        <span class="separator">|</span>
        <span class="comment_count">
            <a href="http://www.cubadebate.cu/noticias/2018/03/19/orishas-cerrara-el-festival-havana-world-music/#respond" title="Comentarios en Orishas cerrará el festival Havana World Music" class="no_comment">+</a>
        </span>
                                                                                                                                                    <span class="separator">|</span>
            <span class="references">
                <a class="entumovil" title="EnTuMóvil"></a> <a class="fotodestacada" title="FotoDestacada"></a>
            </span>
                </div>    <div class="spoiler">
                    <a class="left media" href="http://www.cubadebate.cu/noticias/2018/03/19/orishas-cerrara-el-festival-havana-world-music/" rel="bookmark" title="Enlace a Orishas cerrará el festival Havana World Music">
            <img width="580" height="330" src="http://media.cubadebate.cu/wp-content/uploads/2018/03/Orishas.jpg" class="attachment-full size-full wp-post-image" alt="" />
        </a>
                    <div class="excerpt">
            <p>Orishas, la banda más representativa del hip hop cubano, interpretará en vivo, por primera vez en el mundo, su tema Cuba Isla Bella, el 24 de marzo próximo en el concierto de cierre del festival Havana World Music, presidido por la cantautora cubana, Eme Alfonso.</p>
        </div>
            </div>
</div>
                                                     
                    
<div class="generic image_post  especiales politica-temas referencia-entumovil">
                        <h3 class='cat_title'>
                                        <a href="http://www.cubadebate.cu/categoria/especiales/" rel="category tag" title="Ver todos los art&iacute;culos de Especiales">Especiales</a>, <a href="http://www.cubadebate.cu/categoria/temas/politica-temas/" rel="category tag" title="Ver todos los art&iacute;culos de Política">Política</a>
    &nbsp;&raquo;
            </h3>
        <div class="title"><a href="http://www.cubadebate.cu/especiales/2018/03/19/levanta-donald-trump-un-nuevo-muro-migratorio-en-cuba/" rel="bookmark">¿Levanta Donald Trump un nuevo muro migratorio… en Cuba? (+ Video)</a></div>
                        <div class="meta">
    <time datetime="2018-03-19 20:34:00">19 marzo 2018</time>
        <span class="separator">|</span>
        <span class="comment_count">
            <a href="http://www.cubadebate.cu/especiales/2018/03/19/levanta-donald-trump-un-nuevo-muro-migratorio-en-cuba/#respond" title="Comentarios en ¿Levanta Donald Trump un nuevo muro migratorio… en Cuba? (+ Video)" class="no_comment">+</a>
        </span>
                                                                                                    <span class="separator">|</span>
            <span class="references">
                <a class="entumovil" title="EnTuMóvil"></a>
            </span>
                </div>    <div class="spoiler">
                    <a class="left media" href="http://www.cubadebate.cu/especiales/2018/03/19/levanta-donald-trump-un-nuevo-muro-migratorio-en-cuba/" rel="bookmark" title="Enlace a ¿Levanta Donald Trump un nuevo muro migratorio… en Cuba? (+ Video)">
            <img width="150" height="125" src="http://media.cubadebate.cu/wp-content/uploads/2017/09/embajada-eeuu-en-cuba-150x125.jpg" class="attachment-thumbnail size-thumbnail wp-post-image" alt="" srcset="http://media.cubadebate.cu/wp-content/uploads/2017/09/embajada-eeuu-en-cuba-150x125.jpg 150w, http://media.cubadebate.cu/wp-content/uploads/2017/09/embajada-eeuu-en-cuba-300x250.jpg 300w, http://media.cubadebate.cu/wp-content/uploads/2017/09/embajada-eeuu-en-cuba-180x150.jpg 180w" sizes="(max-width: 150px) 100vw, 150px" />
        </a>
                    <div class="excerpt">
            <p>Con la excusa de unos supuestos "ataques acústicos" a sus diplomáticos, que nadie ha probado, la Casa Blanca decidió reducir el 60 % del personal en su Embajada en Cuba y paralizar la entrega de visas. Ahora, para solicitar visado a EEUU, cubanas y cubanos deben primero viajar a otro país. Si la visa es migratoria, deben ir a Colombia.</p>
        </div>
            </div>
</div>
                                                     
                    <div class="generic external mesa-redonda image_post">
    <h3 class="cat_title">
    <a href="http://mesaredonda.cubadebate.cu" target="_blank" title="Ir a Mesa Redonda">Mesa Redonda</a>
    &nbsp;»
    </h3>    
    <div class="title"><a target="_blank" title="Ver esta nota en Mesa Redonda" href="http://mesaredonda.cubadebate.cu/noticias/2018/03/19/%c2%bfpor-que-gano-putin/" rel="bookmark">¿Por qué ganó Putin?</a></div>
    <div class="meta">
    19 marzo 2018
    <span class="separator">|</span>
    <span class="comment_count">
        <a target="_blank" href="http://mesaredonda.cubadebate.cu/noticias/2018/03/19/%c2%bfpor-que-gano-putin/#respond" title="Comentarios en ¿Por qué ganó Putin?" class="no_comment">+</a>
    </span>
</div>
    <div class="spoiler">
                <a class="left media" href="http://mesaredonda.cubadebate.cu/noticias/2018/03/19/%c2%bfpor-que-gano-putin/" rel="bookmark" title="Enlace a ¿Por qué ganó Putin?">
            <img src="http://mesaredonda.cubadebate.cu/wp-content/uploads/2018/03/putin-victoria-electoral-2-200x112.jpg" width="150px">
        </a>        
                <div class="excerpt">
            <p>En medio de una campaña marcada por la llamada "rusofobia", intentos de ataques cibernéticos y acusaciones como la realizada por Londres sobre el caso Skripal, Vladimir Putin fue reelegido presidente este domingo. ¿Por qué ganó?&nbsp;&raquo;</p>
        </div>   
    </div>
</div>                                                                                                             
                        
<div class="generic image_post  opinion politica-temas autor-elier-ramirez-canedo mobile">
                        <h3 class='cat_title'>
                                        <a href="http://www.cubadebate.cu/categoria/opinion/" rel="category tag" title="Ver todos los art&iacute;culos de Opinión">Opinión</a>, <a href="http://www.cubadebate.cu/categoria/temas/politica-temas/" rel="category tag" title="Ver todos los art&iacute;culos de Política">Política</a>
    &nbsp;&raquo;
            </h3>
        <div class="title"><a href="http://www.cubadebate.cu/opinion/2018/03/19/la-doctrina-monroe-un-poco-de-historia-a-la-luz-de-las-declaraciones-de-tillerson/" rel="bookmark">La doctrina Monroe: Un poco de historia a la luz de las declaraciones de Tillerson</a></div>
                <div id="taxonomies">
            <div>
                <strong>Por: </strong>
                                                                                                    
                <a href="http://www.cubadebate.cu/autor/elier-ramirez-canedo/" rel="category tag taxonomy" title="Ver todos los art&iacute;culos de Elier Ramírez Cañedo">Elier Ramírez Cañedo</a>
            </div>
        </div>
                <div class="meta">
    <time datetime="2018-03-19 19:18:00">19 marzo 2018</time>
        <span class="separator">|</span>
        <span class="comment_count">
            <a href="http://www.cubadebate.cu/opinion/2018/03/19/la-doctrina-monroe-un-poco-de-historia-a-la-luz-de-las-declaraciones-de-tillerson/#respond" title="Comentarios en La doctrina Monroe: Un poco de historia a la luz de las declaraciones de Tillerson" class="no_comment">+</a>
        </span>
                                                </div>    <div class="spoiler">
                    <a class="left media" href="http://www.cubadebate.cu/opinion/2018/03/19/la-doctrina-monroe-un-poco-de-historia-a-la-luz-de-las-declaraciones-de-tillerson/" rel="bookmark" title="Enlace a La doctrina Monroe: Un poco de historia a la luz de las declaraciones de Tillerson">
            <img src="http://media.cubadebate.cu/wp-content/gallery/autores/elier-ramirez-canedo.jpg" alt="Elier Ramírez Cañedo" title="Opini&oacute;n de Elier Ramírez Cañedo" />
        </a>
                    <div class="excerpt">
            <p>Ante los voraces apetitos de las potencias europeas sobre los territorios americanos, enfrentados a los intereses expansionistas de los Estados Unidos, a fines de 1823, mediante un mensaje al Congreso, el presidente James Monroe proclamó lo que se conocería como la Doctrina Monroe.</p>
        </div>
            </div>
</div>
                                                                 
                    
<div class="generic image_post  cultura-temas especiales autor-silvio-rodriguez">
                        <h3 class='cat_title'>
                                        <a href="http://www.cubadebate.cu/categoria/especiales/" rel="category tag" title="Ver todos los art&iacute;culos de Especiales">Especiales</a>, <a href="http://www.cubadebate.cu/categoria/temas/cultura-temas/" rel="category tag" title="Ver todos los art&iacute;culos de Cultura">Cultura</a>
    &nbsp;&raquo;
            </h3>
        <div class="title"><a href="http://www.cubadebate.cu/especiales/2018/03/19/palabras-de-silvio-rodriguez-para-despedir-a-eduardo-ramos/" rel="bookmark">Palabras de Silvio Rodríguez para despedir a Eduardo Ramos</a></div>
                <div id="taxonomies">
            <div>
                <strong>Por: </strong>
                                                                                                    
                <a href="http://www.cubadebate.cu/autor/silvio-rodriguez/" rel="category tag taxonomy" title="Ver todos los art&iacute;culos de Silvio Rodríguez">Silvio Rodríguez</a>
            </div>
        </div>
                <div class="meta">
    <time datetime="2018-03-19 18:55:00">19 marzo 2018</time>
        <span class="separator">|</span>
        <span class="comment_count">
            <a href="http://www.cubadebate.cu/especiales/2018/03/19/palabras-de-silvio-rodriguez-para-despedir-a-eduardo-ramos/#respond" title="Comentarios en Palabras de Silvio Rodríguez para despedir a Eduardo Ramos" class="no_comment">+</a>
        </span>
                                                </div>    <div class="spoiler">
                    <a class="left media" href="http://www.cubadebate.cu/especiales/2018/03/19/palabras-de-silvio-rodriguez-para-despedir-a-eduardo-ramos/" rel="bookmark" title="Enlace a Palabras de Silvio Rodríguez para despedir a Eduardo Ramos">
            <img width="150" height="125" src="http://media.cubadebate.cu/wp-content/uploads/2018/03/eduardo-ramos-1-150x125.jpg" class="attachment-thumbnail size-thumbnail" alt="" srcset="http://media.cubadebate.cu/wp-content/uploads/2018/03/eduardo-ramos-1-150x125.jpg 150w, http://media.cubadebate.cu/wp-content/uploads/2018/03/eduardo-ramos-1-300x250.jpg 300w, http://media.cubadebate.cu/wp-content/uploads/2018/03/eduardo-ramos-1-180x150.jpg 180w" sizes="(max-width: 150px) 100vw, 150px" />
        </a>
                    <div class="excerpt">
            <p>Cuando le conocí, Eduardo Ramos tocaba la guitarra y por aquellos mismos días, porque era necesario, se convirtió en bajista. Al principio, como no tenía el instrumento, lo imitaba con su guitarra eléctrica, ecualizada en bajas frecuencias. Acaso por eso se acostumbró a mover los graves con un sentido acompañante, casi libre, haciendo a veces segundas voces, como en la trova.</p>
        </div>
            </div>
</div>
                                                                                                             
                        
<div class="generic image_post  opinion politica-temas autor-jesus-arboleya-cervera mobile">
                        <h3 class='cat_title'>
                                        <a href="http://www.cubadebate.cu/categoria/opinion/" rel="category tag" title="Ver todos los art&iacute;culos de Opinión">Opinión</a>, <a href="http://www.cubadebate.cu/categoria/temas/politica-temas/" rel="category tag" title="Ver todos los art&iacute;culos de Política">Política</a>
    &nbsp;&raquo;
            </h3>
        <div class="title"><a href="http://www.cubadebate.cu/opinion/2018/03/19/y-ahora-para-que-sirve-la-embajada-de-estados-unidos-en-la-habana/" rel="bookmark">¿Y ahora para qué sirve la embajada de Estados Unidos en La Habana?</a></div>
                <div id="taxonomies">
            <div>
                <strong>Por: </strong>
                                                                                                    
                <a href="http://www.cubadebate.cu/autor/jesus-arboleya-cervera/" rel="category tag taxonomy" title="Ver todos los art&iacute;culos de Jesús Arboleya Cervera">Jesús Arboleya Cervera</a>
            </div>
        </div>
                <div class="meta">
    <time datetime="2018-03-19 19:00:00">19 marzo 2018</time>
        <span class="separator">|</span>
        <span class="comment_count">
            <a href="http://www.cubadebate.cu/opinion/2018/03/19/y-ahora-para-que-sirve-la-embajada-de-estados-unidos-en-la-habana/#respond" title="Comentarios en ¿Y ahora para qué sirve la embajada de Estados Unidos en La Habana?" class="no_comment">+</a>
        </span>
                                                </div>    <div class="spoiler">
                    <a class="left media" href="http://www.cubadebate.cu/opinion/2018/03/19/y-ahora-para-que-sirve-la-embajada-de-estados-unidos-en-la-habana/" rel="bookmark" title="Enlace a ¿Y ahora para qué sirve la embajada de Estados Unidos en La Habana?">
            <img src="http://media.cubadebate.cu/wp-content/gallery/autores/jesus-arboleya-cervera.jpg" alt="Jesús Arboleya Cervera" title="Opini&oacute;n de Jesús Arboleya Cervera" />
        </a>
                    <div class="excerpt">
            <p>Por lo general, las embajadas norteamericanas en el mundo semejan grandes fortalezas. Cinturones de seguridad con vallas alambradas, obstáculos al tránsito, detectores de armas y explosivos, así como garitas con marines portando sofisticados fusiles de guerra, sirven de antesala al ingreso en estos recintos diplomáticos.</p>
        </div>
            </div>
</div>
                                                                 
                    
<div class="generic gallery_post  tecnologia-temas noticias autor-ismael-francisco autor-oscar-figueredo-reinaldo autor-yunier-javier-sifonte-diaz referencia-fotos">
                        <h3 class='cat_title'>
                                        <a href="http://www.cubadebate.cu/categoria/noticias/" rel="category tag" title="Ver todos los art&iacute;culos de Noticias">Noticias</a>, <a href="http://www.cubadebate.cu/categoria/temas/tecnologia-temas/" rel="category tag" title="Ver todos los art&iacute;culos de Ciencia y Tecnología">Ciencia y Tecnología</a>
    &nbsp;&raquo;
            </h3>
        <div class="title"><a href="http://www.cubadebate.cu/noticias/2018/03/19/como-marcha-el-proceso-de-informatizacion-de-la-sociedad-cubana/" rel="bookmark">¿Cómo marcha el proceso de informatización de la sociedad cubana?</a></div>
                <div id="taxonomies">
            <div>
                <strong>Por: </strong>
                                                                                                                                                                            
                <a href="http://www.cubadebate.cu/autor/oscar-figueredo-reinaldo/" rel="category tag taxonomy" title="Ver todos los art&iacute;culos de Oscar Figueredo Reinaldo">Oscar Figueredo Reinaldo</a>, <a href="http://www.cubadebate.cu/autor/yunier-javier-sifonte-diaz/" rel="category tag taxonomy" title="Ver todos los art&iacute;culos de Yunier Javier Sifonte Díaz">Yunier Javier Sifonte Díaz</a>, <a href="http://www.cubadebate.cu/autor/ismael-francisco/" rel="category tag taxonomy" title="Ver todos los art&iacute;culos de Ismael Francisco">Ismael Francisco</a>
            </div>
        </div>
                <div class="meta">
    <time datetime="2018-03-19 18:11:00">19 marzo 2018</time>
        <span class="separator">|</span>
        <span class="comment_count">
            <a href="http://www.cubadebate.cu/noticias/2018/03/19/como-marcha-el-proceso-de-informatizacion-de-la-sociedad-cubana/#respond" title="Comentarios en ¿Cómo marcha el proceso de informatización de la sociedad cubana?" class="no_comment">+</a>
        </span>
                                                                                                    <span class="separator">|</span>
            <span class="references">
                <a class="fotos" title="Fotos"></a>
            </span>
                </div>    <div class="spoiler">
                                                <a class="left media child1" href="http://www.cubadebate.cu/noticias/2018/03/19/como-marcha-el-proceso-de-informatizacion-de-la-sociedad-cubana/" rel="bookmark" title="Enlace a ¿Cómo marcha el proceso de informatización de la sociedad cubana?">
                    <img width="180" height="150" src="http://media.cubadebate.cu/wp-content/uploads/2018/03/mayra-arevich-directora-de-etecsa-180x150.jpg" class="attachment-intermedia size-intermedia" alt="" srcset="http://media.cubadebate.cu/wp-content/uploads/2018/03/mayra-arevich-directora-de-etecsa-180x150.jpg 180w, http://media.cubadebate.cu/wp-content/uploads/2018/03/mayra-arevich-directora-de-etecsa-150x125.jpg 150w, http://media.cubadebate.cu/wp-content/uploads/2018/03/mayra-arevich-directora-de-etecsa-300x250.jpg 300w" sizes="(max-width: 180px) 100vw, 180px" />
                </a>
                            <a class="left media child2" href="http://www.cubadebate.cu/noticias/2018/03/19/como-marcha-el-proceso-de-informatizacion-de-la-sociedad-cubana/" rel="bookmark" title="Enlace a ¿Cómo marcha el proceso de informatización de la sociedad cubana?">
                    <img width="180" height="150" src="http://media.cubadebate.cu/wp-content/uploads/2018/03/informatica-2018-180x150.jpg" class="attachment-intermedia size-intermedia" alt="" srcset="http://media.cubadebate.cu/wp-content/uploads/2018/03/informatica-2018-180x150.jpg 180w, http://media.cubadebate.cu/wp-content/uploads/2018/03/informatica-2018-150x125.jpg 150w, http://media.cubadebate.cu/wp-content/uploads/2018/03/informatica-2018-300x250.jpg 300w" sizes="(max-width: 180px) 100vw, 180px" />
                </a>
                            <a class="left media child3" href="http://www.cubadebate.cu/noticias/2018/03/19/como-marcha-el-proceso-de-informatizacion-de-la-sociedad-cubana/" rel="bookmark" title="Enlace a ¿Cómo marcha el proceso de informatización de la sociedad cubana?">
                    <img width="180" height="150" src="http://media.cubadebate.cu/wp-content/uploads/2018/03/wilfredo-gonzalez-viceministro-de-comunicaciones-de-cuba-180x150.jpg" class="attachment-intermedia size-intermedia" alt="" srcset="http://media.cubadebate.cu/wp-content/uploads/2018/03/wilfredo-gonzalez-viceministro-de-comunicaciones-de-cuba-180x150.jpg 180w, http://media.cubadebate.cu/wp-content/uploads/2018/03/wilfredo-gonzalez-viceministro-de-comunicaciones-de-cuba-150x125.jpg 150w, http://media.cubadebate.cu/wp-content/uploads/2018/03/wilfredo-gonzalez-viceministro-de-comunicaciones-de-cuba-300x250.jpg 300w" sizes="(max-width: 180px) 100vw, 180px" />
                </a>
                                        <div class="excerpt">
            <p>¿Qué novedades hay en el proceso de informatización de la sociedad cubana? ¿Cuánto se logró avanzar en el año 2017? ¿Cuáles son los planes a corto plazo de los diferentes sectores en Cuba? Estas y otras preguntas encontraron respuesta durante la Mesa Redonda realizada en la tarde de este lunes, durante la sesión inaugural de XVII Convención y Feria Informática 2018.</div><div><ul class="anexos_excerpt"><li><a href="http://www.cubadebate.cu/noticias/2018/03/19/como-marcha-el-proceso-de-informatizacion-de-la-sociedad-cubana/#anexo-1034787" title="Ver anexos">Informática 2018 abre sus puertas al desarrollo sostenible</a></li></ul></p>
        </div>
            </div>
</div>
                                                     
                    
<div class="generic image_post  especiales historia-temas ">
                        <h3 class='cat_title'>
                                        <a href="http://www.cubadebate.cu/categoria/especiales/" rel="category tag" title="Ver todos los art&iacute;culos de Especiales">Especiales</a>, <a href="http://www.cubadebate.cu/categoria/temas/historia-temas/" rel="category tag" title="Ver todos los art&iacute;culos de Historia">Historia</a>
    &nbsp;&raquo;
            </h3>
        <div class="title"><a href="http://www.cubadebate.cu/especiales/2018/03/19/a-60-anos-del-asesinato-de-sergio-el-curita/" rel="bookmark">A 60 años del asesinato de Sergio El Curita</a></div>
                <div id="taxonomies">
            <div>
                <strong>Por: </strong>
                                                                                                    
                <span class="extraauthor">Martha Gómez Ferrals</span>
            </div>
        </div>
                <div class="meta">
    <time datetime="2018-03-19 17:07:00">19 marzo 2018</time>
        <span class="separator">|</span>
        <span class="comment_count">
            <a href="http://www.cubadebate.cu/especiales/2018/03/19/a-60-anos-del-asesinato-de-sergio-el-curita/#respond" title="Comentarios en A 60 años del asesinato de Sergio El Curita" >1</a>
        </span>
                                                </div>    <div class="spoiler">
                    <a class="left media" href="http://www.cubadebate.cu/especiales/2018/03/19/a-60-anos-del-asesinato-de-sergio-el-curita/" rel="bookmark" title="Enlace a A 60 años del asesinato de Sergio El Curita">
            <img width="150" height="125" src="http://media.cubadebate.cu/wp-content/uploads/2018/03/Sergio-González-El-Curita-150x125.jpg" class="attachment-thumbnail size-thumbnail wp-post-image" alt="" srcset="http://media.cubadebate.cu/wp-content/uploads/2018/03/Sergio-González-El-Curita-150x125.jpg 150w, http://media.cubadebate.cu/wp-content/uploads/2018/03/Sergio-González-El-Curita-180x150.jpg 180w, http://media.cubadebate.cu/wp-content/uploads/2018/03/Sergio-González-El-Curita.jpg 300w" sizes="(max-width: 150px) 100vw, 150px" />
        </a>
                    <div class="excerpt">
            <p>El 19 de marzo de 1958 apareció en una calle de La Habana el cuerpo sin vida y cruelmente ultrajado del revolucionario Sergio González López, también llamado cariñosamente El Curita, jefe de Acción y Sabotaje del Movimiento 26 de Julio en la capital. Junto a él, también fueron encontrados los cadáveres de sus compañeros de lucha Juan Borrel y Bernardino García Santos.</p>
        </div>
            </div>
</div>
                                                     
                    
<div class="generic gallery_post  noticias sociedad-temas referencia-entumovil referencia-fotos internet-estallido-comentarios internet-interes-hoy internet-movida-facebook internet-pagina-entrada especiales-relevantes">
                                                                                                                                                <span class="hot_in">
        <a class="estallido-comentarios" href="http://www.cubadebate.cu/internet/estallido-comentarios/" rel="category tag taxonomy" title="Estallido de Comentarios">Estallido de Comentarios</a> <a class="interes-hoy" href="http://www.cubadebate.cu/internet/interes-hoy/" rel="category tag taxonomy" title="Interés hoy">Interés hoy</a> <a class="movida-facebook" href="http://www.cubadebate.cu/internet/movida-facebook/" rel="category tag taxonomy" title="Movida en Facebook">Movida en Facebook</a> <a class="pagina-entrada" href="http://www.cubadebate.cu/internet/pagina-entrada/" rel="category tag taxonomy" title="Página de entrada">Página de entrada</a>
    </span>
                    <h3 class='cat_title'>
                                        <a href="http://www.cubadebate.cu/categoria/noticias/" rel="category tag" title="Ver todos los art&iacute;culos de Noticias">Noticias</a>, <a href="http://www.cubadebate.cu/categoria/temas/sociedad-temas/" rel="category tag" title="Ver todos los art&iacute;culos de Sociedad">Sociedad</a>
    &nbsp;&raquo;
            </h3>
        <div class="title"><a href="http://www.cubadebate.cu/noticias/2018/03/19/un-fallecido-y-34-lesionados-por-accidente-masivo-en-oriente-cubano/" rel="bookmark">Un fallecido y 34 lesionados por accidente masivo en oriente cubano (+ Fotos y Video)</a></div>
                        <div class="meta">
    <time datetime="2018-03-19 16:00:00">19 marzo 2018</time>
        <span class="separator">|</span>
        <span class="comment_count">
            <a href="http://www.cubadebate.cu/noticias/2018/03/19/un-fallecido-y-34-lesionados-por-accidente-masivo-en-oriente-cubano/#respond" title="Comentarios en Un fallecido y 34 lesionados por accidente masivo en oriente cubano (+ Fotos y Video)" >60</a>
        </span>
                                                                                                                                                    <span class="separator">|</span>
            <span class="references">
                <a class="entumovil" title="EnTuMóvil"></a> <a class="fotos" title="Fotos"></a>
            </span>
                </div>    <div class="spoiler">
                                                <a class="left media child1" href="http://www.cubadebate.cu/noticias/2018/03/19/un-fallecido-y-34-lesionados-por-accidente-masivo-en-oriente-cubano/" rel="bookmark" title="Enlace a Un fallecido y 34 lesionados por accidente masivo en oriente cubano (+ Fotos y Video)">
                    <img width="180" height="150" src="http://media.cubadebate.cu/wp-content/uploads/2018/03/accidente-pilon-granma-1-180x150.jpg" class="attachment-intermedia size-intermedia" alt="" srcset="http://media.cubadebate.cu/wp-content/uploads/2018/03/accidente-pilon-granma-1-180x150.jpg 180w, http://media.cubadebate.cu/wp-content/uploads/2018/03/accidente-pilon-granma-1-150x125.jpg 150w, http://media.cubadebate.cu/wp-content/uploads/2018/03/accidente-pilon-granma-1-300x250.jpg 300w" sizes="(max-width: 180px) 100vw, 180px" />
                </a>
                            <a class="left media child2" href="http://www.cubadebate.cu/noticias/2018/03/19/un-fallecido-y-34-lesionados-por-accidente-masivo-en-oriente-cubano/" rel="bookmark" title="Enlace a Un fallecido y 34 lesionados por accidente masivo en oriente cubano (+ Fotos y Video)">
                    <img width="180" height="150" src="http://media.cubadebate.cu/wp-content/uploads/2018/03/heridos-accidente-granma-180x150.jpg" class="attachment-intermedia size-intermedia" alt="" srcset="http://media.cubadebate.cu/wp-content/uploads/2018/03/heridos-accidente-granma-180x150.jpg 180w, http://media.cubadebate.cu/wp-content/uploads/2018/03/heridos-accidente-granma-150x125.jpg 150w, http://media.cubadebate.cu/wp-content/uploads/2018/03/heridos-accidente-granma-300x250.jpg 300w" sizes="(max-width: 180px) 100vw, 180px" />
                </a>
                            <a class="left media child3" href="http://www.cubadebate.cu/noticias/2018/03/19/un-fallecido-y-34-lesionados-por-accidente-masivo-en-oriente-cubano/" rel="bookmark" title="Enlace a Un fallecido y 34 lesionados por accidente masivo en oriente cubano (+ Fotos y Video)">
                    <img width="180" height="150" src="http://media.cubadebate.cu/wp-content/uploads/2018/03/accidente-camion-gugua-180x150.jpg" class="attachment-intermedia size-intermedia" alt="" srcset="http://media.cubadebate.cu/wp-content/uploads/2018/03/accidente-camion-gugua-180x150.jpg 180w, http://media.cubadebate.cu/wp-content/uploads/2018/03/accidente-camion-gugua-150x125.jpg 150w, http://media.cubadebate.cu/wp-content/uploads/2018/03/accidente-camion-gugua-300x250.jpg 300w" sizes="(max-width: 180px) 100vw, 180px" />
                </a>
                                        <div class="excerpt">
            <p>Una persona fallecida y 34 lesionadas fue el saldo preliminar de un accidente masivo de tránsito, ocurrido esta mañana en el tramo final de la carretera sobre la loma de La Herradura, en la localidad costera de Pilón, provincia de Granma. El fallecido se nombra Eduardo Rosario Verdecia, de 54 años de edad, y metodólogo de Educación.</p>
        </div>
            </div>
</div>
                                                     
                    
<div class="generic bigimage_post  deporte-temas especiales referencia-entumovil referencia-fotodestacada">
                        <h3 class='cat_title'>
                                        <a href="http://www.cubadebate.cu/categoria/especiales/" rel="category tag" title="Ver todos los art&iacute;culos de Especiales">Especiales</a>, <a href="http://www.cubadebate.cu/categoria/temas/deporte-temas/" rel="category tag" title="Ver todos los art&iacute;culos de Deportes">Deportes</a>
    &nbsp;&raquo;
            </h3>
        <div class="title"><a href="http://www.cubadebate.cu/especiales/2018/03/19/los-peloteros-correa-y-beltran-reavivan-la-discusion-sobre-puerto-rico-en-eeuu/" rel="bookmark">Los peloteros Correa y Beltrán reavivan la discusión sobre Puerto Rico en EEUU</a></div>
                <div id="taxonomies">
            <div>
                <strong>Por: </strong>
                                                                                                    
                <span class="extraauthor">Noel Algarín Martínez</span>
            </div>
        </div>
                <div class="meta">
    <time datetime="2018-03-19 15:22:00">19 marzo 2018</time>
        <span class="separator">|</span>
        <span class="comment_count">
            <a href="http://www.cubadebate.cu/especiales/2018/03/19/los-peloteros-correa-y-beltran-reavivan-la-discusion-sobre-puerto-rico-en-eeuu/#respond" title="Comentarios en Los peloteros Correa y Beltrán reavivan la discusión sobre Puerto Rico en EEUU" >4</a>
        </span>
                                                                                                                                                    <span class="separator">|</span>
            <span class="references">
                <a class="entumovil" title="EnTuMóvil"></a> <a class="fotodestacada" title="FotoDestacada"></a>
            </span>
                </div>    <div class="spoiler">
                    <a class="left media" href="http://www.cubadebate.cu/especiales/2018/03/19/los-peloteros-correa-y-beltran-reavivan-la-discusion-sobre-puerto-rico-en-eeuu/" rel="bookmark" title="Enlace a Los peloteros Correa y Beltrán reavivan la discusión sobre Puerto Rico en EEUU">
            <img src="http://media.cubadebate.cu/wp-content/uploads/2017/10/altuve-correa-astros-de-houston.jpg" width="580" height="330" alt="Los peloteros Correa y Beltrán reavivan la discusión sobre Puerto Rico en EEUU" title="Los peloteros Correa y Beltrán reavivan la discusión sobre Puerto Rico en EEUU" />
        </a>
                    <div class="excerpt">
            <p>Ninguno lo verbalizó. No hizo falta. Estaba claro que el pasado lunes 13 de marzo, los peloteros puertorriqueños Carlos Beltrán y Carlos Correa preferían estar en otro lugar que no fuese la Casa Blanca en Washington. Ese día, los campeones de las Grandes Ligas, Astros de Houston, viajaron a la residencia del presidente de Estados Unidos, Donald Trump.</p>
        </div>
            </div>
</div>
                                                     
                    
<div class="generic image_post  noticias sociedad-temas referencia-entumovil internet-estallido-comentarios especiales-relevantes">
                                                <span class="hot_in">
        <a class="estallido-comentarios" href="http://www.cubadebate.cu/internet/estallido-comentarios/" rel="category tag taxonomy" title="Estallido de Comentarios">Estallido de Comentarios</a>
    </span>
                    <h3 class='cat_title'>
                                        <a href="http://www.cubadebate.cu/categoria/noticias/" rel="category tag" title="Ver todos los art&iacute;culos de Noticias">Noticias</a>, <a href="http://www.cubadebate.cu/categoria/temas/sociedad-temas/" rel="category tag" title="Ver todos los art&iacute;culos de Sociedad">Sociedad</a>
    &nbsp;&raquo;
            </h3>
        <div class="title"><a href="http://www.cubadebate.cu/noticias/2018/03/19/nace-bebe-en-pleno-vuelo-la-habana-bogota/" rel="bookmark">Nace bebé en pleno vuelo La Habana-Bogotá</a></div>
                        <div class="meta">
    <time datetime="2018-03-19 15:06:00">19 marzo 2018</time>
        <span class="separator">|</span>
        <span class="comment_count">
            <a href="http://www.cubadebate.cu/noticias/2018/03/19/nace-bebe-en-pleno-vuelo-la-habana-bogota/#respond" title="Comentarios en Nace bebé en pleno vuelo La Habana-Bogotá" >25</a>
        </span>
                                                                                                    <span class="separator">|</span>
            <span class="references">
                <a class="entumovil" title="EnTuMóvil"></a>
            </span>
                </div>    <div class="spoiler">
                    <a class="left media" href="http://www.cubadebate.cu/noticias/2018/03/19/nace-bebe-en-pleno-vuelo-la-habana-bogota/" rel="bookmark" title="Enlace a Nace bebé en pleno vuelo La Habana-Bogotá">
            <img width="150" height="125" src="http://media.cubadebate.cu/wp-content/uploads/2018/03/avianca-150x125.jpg" class="attachment-thumbnail size-thumbnail wp-post-image" alt="" srcset="http://media.cubadebate.cu/wp-content/uploads/2018/03/avianca-150x125.jpg 150w, http://media.cubadebate.cu/wp-content/uploads/2018/03/avianca-300x250.jpg 300w, http://media.cubadebate.cu/wp-content/uploads/2018/03/avianca-180x150.jpg 180w" sizes="(max-width: 150px) 100vw, 150px" />
        </a>
                    <div class="excerpt">
            <p>Una mujer de origen angolano que viajaba en un vuelo comercial procedente de La Habana hacia Bogotá, dio a luz a 37.000 pies de altura, lo que obligó al avión a desviarse de la ruta y aterrizar en Cartagena. El parto fue asistido por uno de los pasajeros del vuelo AV255 de la aerolínea colombiana Avianca, informó la empresa en un comunicado este domingo .</p>
        </div>
            </div>
</div>
                                                     
                    
<div class="generic gallery_post  noticias politica-temas referencia-fotos">
                        <h3 class='cat_title'>
                                        <a href="http://www.cubadebate.cu/categoria/noticias/" rel="category tag" title="Ver todos los art&iacute;culos de Noticias">Noticias</a>, <a href="http://www.cubadebate.cu/categoria/temas/politica-temas/" rel="category tag" title="Ver todos los art&iacute;culos de Política">Política</a>
    &nbsp;&raquo;
            </h3>
        <div class="title"><a href="http://www.cubadebate.cu/noticias/2018/03/19/comediante-estadounidense-caricaturiza-a-politicos-en-twitter/" rel="bookmark">Las caricaturas de Jim Carrey sobre políticos que son tendencia en Twitter</a></div>
                        <div class="meta">
    <time datetime="2018-03-19 14:44:00">19 marzo 2018</time>
        <span class="separator">|</span>
        <span class="comment_count">
            <a href="http://www.cubadebate.cu/noticias/2018/03/19/comediante-estadounidense-caricaturiza-a-politicos-en-twitter/#respond" title="Comentarios en Las caricaturas de Jim Carrey sobre políticos que son tendencia en Twitter" >2</a>
        </span>
                                                                                                    <span class="separator">|</span>
            <span class="references">
                <a class="fotos" title="Fotos"></a>
            </span>
                </div>    <div class="spoiler">
                                                <a class="left media child1" href="http://www.cubadebate.cu/noticias/2018/03/19/comediante-estadounidense-caricaturiza-a-politicos-en-twitter/" rel="bookmark" title="Enlace a Las caricaturas de Jim Carrey sobre políticos que son tendencia en Twitter">
                    <img width="180" height="150" src="http://media.cubadebate.cu/wp-content/uploads/2018/03/caricatura-carrey-180x150.jpg" class="attachment-intermedia size-intermedia" alt="" srcset="http://media.cubadebate.cu/wp-content/uploads/2018/03/caricatura-carrey-180x150.jpg 180w, http://media.cubadebate.cu/wp-content/uploads/2018/03/caricatura-carrey-150x125.jpg 150w, http://media.cubadebate.cu/wp-content/uploads/2018/03/caricatura-carrey-300x250.jpg 300w" sizes="(max-width: 180px) 100vw, 180px" />
                </a>
                            <a class="left media child2" href="http://www.cubadebate.cu/noticias/2018/03/19/comediante-estadounidense-caricaturiza-a-politicos-en-twitter/" rel="bookmark" title="Enlace a Las caricaturas de Jim Carrey sobre políticos que son tendencia en Twitter">
                    <img width="180" height="150" src="http://media.cubadebate.cu/wp-content/uploads/2018/03/caricatura-Jim-Carrey-180x150.jpg" class="attachment-intermedia size-intermedia" alt="" srcset="http://media.cubadebate.cu/wp-content/uploads/2018/03/caricatura-Jim-Carrey-180x150.jpg 180w, http://media.cubadebate.cu/wp-content/uploads/2018/03/caricatura-Jim-Carrey-150x125.jpg 150w, http://media.cubadebate.cu/wp-content/uploads/2018/03/caricatura-Jim-Carrey-300x250.jpg 300w" sizes="(max-width: 180px) 100vw, 180px" />
                </a>
                            <a class="left media child3" href="http://www.cubadebate.cu/noticias/2018/03/19/comediante-estadounidense-caricaturiza-a-politicos-en-twitter/" rel="bookmark" title="Enlace a Las caricaturas de Jim Carrey sobre políticos que son tendencia en Twitter">
                    <img width="180" height="150" src="http://media.cubadebate.cu/wp-content/uploads/2018/03/caricatura-sahara-180x150.jpg" class="attachment-intermedia size-intermedia" alt="" srcset="http://media.cubadebate.cu/wp-content/uploads/2018/03/caricatura-sahara-180x150.jpg 180w, http://media.cubadebate.cu/wp-content/uploads/2018/03/caricatura-sahara-150x125.jpg 150w, http://media.cubadebate.cu/wp-content/uploads/2018/03/caricatura-sahara-300x250.jpg 300w" sizes="(max-width: 180px) 100vw, 180px" />
                </a>
                                        <div class="excerpt">
            <p>Desde el pasado fin de semana, un dibujo caricaturizando a Sarah Huckabee Sanders, secretaria de prensa de la Casa Blanca, realizado por Jim Carrey, se convirtió en tendencia en Twitter. Sin embargo, Sanders no ha sido la única "víctima" del comediante; otros políticos y situaciones del ámbito estadounidense han sido su blanco de críticas.</p>
        </div>
            </div>
</div>
                                                     
                    
<div class="generic image_post  cultura-temas noticias columna-podcast autor-reinaldo-taladrid-herrero internet-estallido-comentarios especiales-relevantes">
                                                <span class="hot_in">
        <a class="estallido-comentarios" href="http://www.cubadebate.cu/internet/estallido-comentarios/" rel="category tag taxonomy" title="Estallido de Comentarios">Estallido de Comentarios</a>
    </span>
                    <h3 class='cat_title'>
                            <a href="http://www.cubadebate.cu/columna/podcast/" rel="category tag" title="Ver todos los art&iacute;culos de Podcast">Podcast</a>
    &nbsp;&raquo;
            </h3>
        <div class="title"><a href="http://www.cubadebate.cu/noticias/2018/03/19/pasaje-a-lo-desconocido-que-pizza-prefieren-los-cubanos-la-cubana-o-la-italiana-podcast/" rel="bookmark">Pasaje a lo desconocido: ¿Qué pizza prefieren los cubanos, la cubana o la italiana? (+ Podcast)</a></div>
                <div id="taxonomies">
            <div>
                <strong>Por: </strong>
                                                                                                    
                <a href="http://www.cubadebate.cu/autor/reinaldo-taladrid-herrero/" rel="category tag taxonomy" title="Ver todos los art&iacute;culos de Reinaldo Taladrid Herrero">Reinaldo Taladrid Herrero</a>
            </div>
        </div>
                <div class="meta">
    <time datetime="2018-03-19 13:50:00">19 marzo 2018</time>
        <span class="separator">|</span>
        <span class="comment_count">
            <a href="http://www.cubadebate.cu/noticias/2018/03/19/pasaje-a-lo-desconocido-que-pizza-prefieren-los-cubanos-la-cubana-o-la-italiana-podcast/#respond" title="Comentarios en Pasaje a lo desconocido: ¿Qué pizza prefieren los cubanos, la cubana o la italiana? (+ Podcast)" >33</a>
        </span>
                                                </div>    <div class="spoiler">
                    <a class="left media" href="http://www.cubadebate.cu/noticias/2018/03/19/pasaje-a-lo-desconocido-que-pizza-prefieren-los-cubanos-la-cubana-o-la-italiana-podcast/" rel="bookmark" title="Enlace a Pasaje a lo desconocido: ¿Qué pizza prefieren los cubanos, la cubana o la italiana? (+ Podcast)">
            <img width="150" height="125" src="http://media.cubadebate.cu/wp-content/uploads/2018/02/pasaje-a-lo-desconocido-podcast-CD-150x125.jpg" class="attachment-thumbnail size-thumbnail wp-post-image" alt="" srcset="http://media.cubadebate.cu/wp-content/uploads/2018/02/pasaje-a-lo-desconocido-podcast-CD-150x125.jpg 150w, http://media.cubadebate.cu/wp-content/uploads/2018/02/pasaje-a-lo-desconocido-podcast-CD-300x250.jpg 300w, http://media.cubadebate.cu/wp-content/uploads/2018/02/pasaje-a-lo-desconocido-podcast-CD-180x150.jpg 180w" sizes="(max-width: 150px) 100vw, 150px" />
        </a>
                    <div class="excerpt">
            <p>¿Cuál es el tema de hoy? La pizza, ese plato tradicional italiano que es universal. ¿Conoces su historia? ¿Tiene valor nutricional? ¿Qué prefieren los cubanos la pizza italiana o la cubana? ¿En qué se diferencian? En este episodio, el periodista Reinaldo Taladrid dialoga con Luis Garcés García-Espinosa, jefe de Nutrición de la Clínica del Hospital Calixto García.</p>
        </div>
            </div>
</div>
                             <div class="navigation">
                                    <div class='wp-pagenavi'>
<span class='current'>1</span><a class="page larger" title="Página 2" href="http://www.cubadebate.cu/page/2/">2</a><a class="page larger" title="Página 3" href="http://www.cubadebate.cu/page/3/">3</a><a class="page larger" title="Página 4" href="http://www.cubadebate.cu/page/4/">4</a><a class="page larger" title="Página 5" href="http://www.cubadebate.cu/page/5/">5</a><a class="page larger" title="Página 6" href="http://www.cubadebate.cu/page/6/">6</a><a class="page larger" title="Página 7" href="http://www.cubadebate.cu/page/7/">7</a><a class="page larger" title="Página 8" href="http://www.cubadebate.cu/page/8/">8</a><a class="page larger" title="Página 9" href="http://www.cubadebate.cu/page/9/">9</a><a class="page larger" title="Página 10" href="http://www.cubadebate.cu/page/10/">10</a><span class='extend'>...</span><a class="nextpostslink" rel="next" href="http://www.cubadebate.cu/page/2/">»</a><a class="last" href="http://www.cubadebate.cu/page/7067/">Última »</a>
</div>
                            </div>
        </div>
        <section id="relates" class="nomobile">
            <aside id="op_post" class="op_posts">
    <header class="title_wrapper">
        <span class="icon_class opinion_red"></span>
        <h2 class="widget-title seccion_title">Opiniones</h2>
    </header>
    <ul>
                    <li class="op-posts-item-1034775">
                <div class="header">
                    <div class="image_wrapper cycle-slideshow" data-timeout="8000" data-speed="1000">
                                                                            <img class="authorphoto" align="left" src="http://media.cubadebate.cu/wp-content/gallery/autores/elier-ramirez-canedo.jpg" alt="Elier Ramírez Cañedo" title="Articulos de Elier Ramírez Cañedo publicados en Cubadebate" width="60" height="80" />
                                            </div>
                    <div class="op_wrapper" style="margin-left: 5px;">
                                                                                            <h3 class='cat_title'>
                                        <a href="http://www.cubadebate.cu/categoria/opinion/" rel="category tag" title="Ver todos los art&iacute;culos de Opinión">Opinión</a>, <a href="http://www.cubadebate.cu/categoria/temas/politica-temas/" rel="category tag" title="Ver todos los art&iacute;culos de Política">Política</a>
    &nbsp;&raquo;
            </h3>
                                                    <h3 class="post_title"><a href="http://www.cubadebate.cu/opinion/2018/03/19/la-doctrina-monroe-un-poco-de-historia-a-la-luz-de-las-declaraciones-de-tillerson/">La doctrina Monroe: Un poco de historia a la luz de las declaraciones de Tillerson</a></h3>
                            <div id="taxonomies">
            <div>
                <strong>Por: </strong>
                                                                                                    
                <a href="http://www.cubadebate.cu/autor/elier-ramirez-canedo/" rel="category tag taxonomy" title="Ver todos los art&iacute;culos de Elier Ramírez Cañedo">Elier Ramírez Cañedo</a>
            </div>
        </div>
                            <div class="meta">
    <time datetime="2018-03-19 19:18:00">19 marzo 2018</time>
        <span class="separator">|</span>
        <span class="comment_count">
            <a href="http://www.cubadebate.cu/opinion/2018/03/19/la-doctrina-monroe-un-poco-de-historia-a-la-luz-de-las-declaraciones-de-tillerson/#respond" title="Comentarios en La doctrina Monroe: Un poco de historia a la luz de las declaraciones de Tillerson" class="no_comment">+</a>
        </span>
            </div>                    </div>
                </div>
                                    <br/>
                    <div class="op_excerpt">
                        <p>Ante los voraces apetitos de las potencias europeas sobre los territorios americanos, enfrentados a los intereses expansionistas de los Estados Unidos, a fines de 1823, mediante un mensaje al Congreso, el presidente James Monroe proclamó lo que se conocería como la Doctrina Monroe.</p>
                    </div>
                            </li>
                    <li class="op-posts-item-1034797">
                <div class="header">
                    <div class="image_wrapper cycle-slideshow" data-timeout="8000" data-speed="1000">
                                                                            <img class="authorphoto" align="left" src="http://media.cubadebate.cu/wp-content/gallery/autores/jesus-arboleya-cervera.jpg" alt="Jesús Arboleya Cervera" title="Articulos de Jesús Arboleya Cervera publicados en Cubadebate" width="60" height="80" />
                                            </div>
                    <div class="op_wrapper" style="margin-left: 5px;">
                                                                                            <h3 class='cat_title'>
                                        <a href="http://www.cubadebate.cu/categoria/opinion/" rel="category tag" title="Ver todos los art&iacute;culos de Opinión">Opinión</a>, <a href="http://www.cubadebate.cu/categoria/temas/politica-temas/" rel="category tag" title="Ver todos los art&iacute;culos de Política">Política</a>
    &nbsp;&raquo;
            </h3>
                                                    <h3 class="post_title"><a href="http://www.cubadebate.cu/opinion/2018/03/19/y-ahora-para-que-sirve-la-embajada-de-estados-unidos-en-la-habana/">¿Y ahora para qué sirve la embajada de Estados Unidos en La Habana?</a></h3>
                            <div id="taxonomies">
            <div>
                <strong>Por: </strong>
                                                                                                    
                <a href="http://www.cubadebate.cu/autor/jesus-arboleya-cervera/" rel="category tag taxonomy" title="Ver todos los art&iacute;culos de Jesús Arboleya Cervera">Jesús Arboleya Cervera</a>
            </div>
        </div>
                            <div class="meta">
    <time datetime="2018-03-19 19:00:00">19 marzo 2018</time>
        <span class="separator">|</span>
        <span class="comment_count">
            <a href="http://www.cubadebate.cu/opinion/2018/03/19/y-ahora-para-que-sirve-la-embajada-de-estados-unidos-en-la-habana/#respond" title="Comentarios en ¿Y ahora para qué sirve la embajada de Estados Unidos en La Habana?" class="no_comment">+</a>
        </span>
            </div>                    </div>
                </div>
                                    <br/>
                    <div class="op_excerpt">
                        <p>Por lo general, las embajadas norteamericanas en el mundo semejan grandes fortalezas. Cinturones de seguridad con vallas alambradas, obstáculos al tránsito, detectores de armas y explosivos, así como garitas con marines portando sofisticados fusiles de guerra, sirven de antesala al ingreso en estos recintos diplomáticos.</p>
                    </div>
                            </li>
                    <li class="op-posts-item-1034649">
                <div class="header">
                    <div class="image_wrapper cycle-slideshow" data-timeout="8000" data-speed="1000">
                                                                            <img class="authorphoto" align="left" src="http://media.cubadebate.cu/wp-content/gallery/autores/gracziella-pogolotti.jpg" alt="Graziella Pogolotti" title="Articulos de Graziella Pogolotti publicados en Cubadebate" width="60" height="80" />
                                            </div>
                    <div class="op_wrapper" style="margin-left: 5px;">
                                                                                            <h3 class='cat_title'>
                                        <a href="http://www.cubadebate.cu/categoria/opinion/" rel="category tag" title="Ver todos los art&iacute;culos de Opinión">Opinión</a>, <a href="http://www.cubadebate.cu/categoria/temas/medios-temas/" rel="category tag" title="Ver todos los art&iacute;culos de Medios">Medios</a>
    &nbsp;&raquo;
            </h3>
                                                    <h3 class="post_title"><a href="http://www.cubadebate.cu/opinion/2018/03/19/periodismo-y-cultura/">Periodismo y cultura</a></h3>
                            <div id="taxonomies">
            <div>
                <strong>Por: </strong>
                                                                                                    
                <a href="http://www.cubadebate.cu/autor/gracziella-pogolotti/" rel="category tag taxonomy" title="Ver todos los art&iacute;culos de Graziella Pogolotti">Graziella Pogolotti</a>
            </div>
        </div>
                            <div class="meta">
    <time datetime="2018-03-19 13:21:00">19 marzo 2018</time>
        <span class="separator">|</span>
        <span class="comment_count">
            <a href="http://www.cubadebate.cu/opinion/2018/03/19/periodismo-y-cultura/#respond" title="Comentarios en Periodismo y cultura" class="no_comment">+</a>
        </span>
            </div>                    </div>
                </div>
                                    <br/>
                    <div class="op_excerpt">
                        <p>José Antonio Fernández de Castro es el nombre del premio que se concede a los periodistas del sector cultural. Poco se sabe, sin embargo, de la trayectoria fulgurante y efímera de este singular personaje. Emergió en el contexto de la Primera Vanguardia y del Grupo Minorista, fenómenos que contribuyeron a configurar el entorno de los años 20 en la Cuba del pasado siglo. Perteneció a la generación de intelectuales que impulsó, en la práctica concreta, la renovación de los lenguajes artísticos.</p>
                    </div>
                            </li>
                    <li class="op-posts-item-1034193">
                <div class="header">
                    <div class="image_wrapper cycle-slideshow" data-timeout="8000" data-speed="1000">
                                                                            <img class="authorphoto" align="left" src="http://media.cubadebate.cu/wp-content/gallery/autores/aynel-martinez-hernandez.jpg" alt="Aynel Martínez Hernández" title="Articulos de Aynel Martínez Hernández publicados en Cubadebate" width="60" height="80" />
                                            </div>
                    <div class="op_wrapper" style="margin-left: 5px;">
                                                                                            <h3 class='cat_title'>
                            <a href="http://www.cubadebate.cu/columna/goles-son-amores/" rel="category tag" title="Ver todos los art&iacute;culos de Goles son amores">Goles son amores</a>
    &nbsp;&raquo;
            </h3>
                                                    <h3 class="post_title"><a href="http://www.cubadebate.cu/opinion/2018/03/19/goles-son-amores-vicios/">Goles son amores: Vicios</a></h3>
                            <div id="taxonomies">
            <div>
                <strong>Por: </strong>
                                                                                                    
                <a href="http://www.cubadebate.cu/autor/aynel-martinez-hernandez/" rel="category tag taxonomy" title="Ver todos los art&iacute;culos de Aynel Martínez Hernández">Aynel Martínez Hernández</a>
            </div>
        </div>
                            <div class="meta">
    <time datetime="2018-03-19 08:16:00">19 marzo 2018</time>
        <span class="separator">|</span>
        <span class="comment_count">
            <a href="http://www.cubadebate.cu/opinion/2018/03/19/goles-son-amores-vicios/#respond" title="Comentarios en Goles son amores: Vicios" >28</a>
        </span>
            </div>                    </div>
                </div>
                                    <br/>
                    <div class="op_excerpt">
                        <p>Los equipos de 'Mou' se definen a partir de una relación causal entre vicios propios y racionalidades, a priori, obstinadas. Eso no debería estar mal del todo. El fútbol también obedece, en ocasiones, a esas cuestiones casi extravagantes a partir de las que, por ejemplo, Pogba no puede jugar.</p>
                    </div>
                            </li>
                    <li class="op-posts-item-1034019">
                <div class="header">
                    <div class="image_wrapper cycle-slideshow" data-timeout="8000" data-speed="1000">
                                                                            <img class="authorphoto" align="left" src="http://media.cubadebate.cu/wp-content/gallery/autores/angeles-maestro.jpg" alt="Angeles Maestro" title="Articulos de Angeles Maestro publicados en Cubadebate" width="60" height="80" />
                                            </div>
                    <div class="op_wrapper" style="margin-left: 5px;">
                                                                                            <h3 class='cat_title'>
                                        <a href="http://www.cubadebate.cu/categoria/opinion/" rel="category tag" title="Ver todos los art&iacute;culos de Opinión">Opinión</a>, <a href="http://www.cubadebate.cu/categoria/temas/politica-temas/" rel="category tag" title="Ver todos los art&iacute;culos de Política">Política</a>
    &nbsp;&raquo;
            </h3>
                                                    <h3 class="post_title"><a href="http://www.cubadebate.cu/opinion/2018/03/18/venezuela-y-el-genio-escapo-de-la-botella/">Venezuela: Y el genio escapó de la botella&#8230;.</a></h3>
                            <div id="taxonomies">
            <div>
                <strong>Por: </strong>
                                                                                                    
                <a href="http://www.cubadebate.cu/autor/angeles-maestro/" rel="category tag taxonomy" title="Ver todos los art&iacute;culos de Angeles Maestro">Angeles Maestro</a>
            </div>
        </div>
                            <div class="meta">
    <time datetime="2018-03-18 09:14:00">18 marzo 2018</time>
        <span class="separator">|</span>
        <span class="comment_count">
            <a href="http://www.cubadebate.cu/opinion/2018/03/18/venezuela-y-el-genio-escapo-de-la-botella/#respond" title="Comentarios en Venezuela: Y el genio escapó de la botella&#8230;." >7</a>
        </span>
            </div>                    </div>
                </div>
                                    <br/>
                    <div class="op_excerpt">
                        <p>Hay una guerra abierta contra el pueblo venezolano y nos concierne directamente. El silencio o la pasividad son cómplices necesarios del crimen. El imperialismo yanki está reproduciendo contra el pueblo venezolano los mismos mecanismos, la misma lógica que le ha llevado a generar 327 golpes de Estado en 25 países latinoamericanos entre 1902 y 2002. Una y otra vez, el gobierno del país que invierte en gasto militar el 40% del total mundial, el triple que China y 9 veces más que Rusia, ha lanzado todo su arsenal desestabilizador sobre cada pueblo que ha intentado hacerse dueño de sus recursos&nbsp;&raquo;</p>
                    </div>
                            </li>
                    <li class="op-posts-item-1033935">
                <div class="header">
                    <div class="image_wrapper cycle-slideshow" data-timeout="8000" data-speed="1000">
                                                                            <img class="authorphoto" align="left" src="http://media.cubadebate.cu/wp-content/gallery/autores/papo-coss.jpg" alt="Papo Coss" title="Articulos de Papo Coss publicados en Cubadebate" width="60" height="80" />
                                            </div>
                    <div class="op_wrapper" style="margin-left: 5px;">
                                                                                            <h3 class='cat_title'>
                            <a href="http://www.cubadebate.cu/columna/el-ritmo-boricua/" rel="category tag" title="Ver todos los art&iacute;culos de El Ritmo Boricua">El Ritmo Boricua</a>
    &nbsp;&raquo;
            </h3>
                                                    <h3 class="post_title"><a href="http://www.cubadebate.cu/opinion/2018/03/17/puerto-rico-vencio-al-imperio/">Puerto Rico venció al Imperio (+ Video)</a></h3>
                            <div id="taxonomies">
            <div>
                <strong>Por: </strong>
                                                                                                    
                <a href="http://www.cubadebate.cu/autor/papo-coss/" rel="category tag taxonomy" title="Ver todos los art&iacute;culos de Papo Coss">Papo Coss</a>
            </div>
        </div>
                            <div class="meta">
    <time datetime="2018-03-17 21:17:00">17 marzo 2018</time>
        <span class="separator">|</span>
        <span class="comment_count">
            <a href="http://www.cubadebate.cu/opinion/2018/03/17/puerto-rico-vencio-al-imperio/#respond" title="Comentarios en Puerto Rico venció al Imperio (+ Video)" >4</a>
        </span>
            </div>                    </div>
                </div>
                                    <br/>
                    <div class="op_excerpt">
                        <p>La afirmación categórica de Fidel, de que es en la cultura que se define el futuro de la humanidad en este siglo 21, adquiere hoy tanta pertinencia, sobretodo para la heroica nación boricua. Es que en ese contexto ideológico, al que se refiere Fidel, el aguerrido pueblo puertorriqueño, ya venció al imperio más poderoso de la historia del planeta tierra. ¿Saben porqué?&nbsp;&raquo;</p>
                    </div>
                            </li>
                    <li class="op-posts-item-1033653">
                <div class="header">
                    <div class="image_wrapper cycle-slideshow" data-timeout="8000" data-speed="1000">
                                                                            <img class="authorphoto" align="left" src="http://media.cubadebate.cu/wp-content/gallery/autores/antonio-rodriguez-salvador.jpg" alt="Antonio Rodríguez Salvador" title="Articulos de Antonio Rodríguez Salvador publicados en Cubadebate" width="60" height="80" />
                                            </div>
                    <div class="op_wrapper" style="margin-left: 5px;">
                                                                                            <h3 class='cat_title'>
                                        <a href="http://www.cubadebate.cu/categoria/opinion/" rel="category tag" title="Ver todos los art&iacute;culos de Opinión">Opinión</a>, <a href="http://www.cubadebate.cu/categoria/temas/cultura-temas/" rel="category tag" title="Ver todos los art&iacute;culos de Cultura">Cultura</a>
    &nbsp;&raquo;
            </h3>
                                                    <h3 class="post_title"><a href="http://www.cubadebate.cu/opinion/2018/03/17/elogio-de-la-lectura/">Elogio de la lectura</a></h3>
                            <div id="taxonomies">
            <div>
                <strong>Por: </strong>
                                                                                                    
                <a href="http://www.cubadebate.cu/autor/antonio-rodriguez-salvador/" rel="category tag taxonomy" title="Ver todos los art&iacute;culos de Antonio Rodríguez Salvador">Antonio Rodríguez Salvador</a>
            </div>
        </div>
                            <div class="meta">
    <time datetime="2018-03-17 01:03:00">17 marzo 2018</time>
        <span class="separator">|</span>
        <span class="comment_count">
            <a href="http://www.cubadebate.cu/opinion/2018/03/17/elogio-de-la-lectura/#respond" title="Comentarios en Elogio de la lectura" >2</a>
        </span>
            </div>                    </div>
                </div>
                                    <br/>
                    <div class="op_excerpt">
                        <p>Llegué de la Feria del Libro, es ahora que estoy en la Feria del Libro. No ensayo un retruécano: en realidad, la verdadera fiesta comienza cuando, ya en casa, empiezas a leer los ejemplares adquiridos. En la Feria del Libro no hay tiempo para la lectura: saludas a aquel, conversas con el otro, asistes a la presentación de una obra… Yo voy a las presentaciones de libros más por compromiso con el autor que por necesidades literarias. Prefiero hojear con calma, de pie frente al estante, ver por mí mismo si el texto azuza la imaginación.</p>
                    </div>
                            </li>
                    <li class="op-posts-item-1033721 last_li_item">
                <div class="header">
                    <div class="image_wrapper cycle-slideshow" data-timeout="8000" data-speed="1000">
                                                                            <img class="authorphoto" align="left" src="http://media.cubadebate.cu/wp-content/gallery/autores/jose-luis-rodriguez.jpg" alt="José Luis Rodríguez" title="Articulos de José Luis Rodríguez publicados en Cubadebate" width="60" height="80" />
                                            </div>
                    <div class="op_wrapper" style="margin-left: 5px;">
                                                                                            <h3 class='cat_title'>
                                        <a href="http://www.cubadebate.cu/categoria/opinion/" rel="category tag" title="Ver todos los art&iacute;culos de Opinión">Opinión</a>, <a href="http://www.cubadebate.cu/categoria/temas/economia-temas/" rel="category tag" title="Ver todos los art&iacute;culos de Economía">Economía</a>
    &nbsp;&raquo;
            </h3>
                                                    <h3 class="post_title"><a href="http://www.cubadebate.cu/opinion/2018/03/17/cuba-y-su-economia-el-2017-recien-concluido-y-un-2018-que-apenas-comienza-iv/">Cuba y su economía: El 2017 recién concluido y un 2018 que apenas comienza (IV)</a></h3>
                            <div id="taxonomies">
            <div>
                <strong>Por: </strong>
                                                                                                    
                <a href="http://www.cubadebate.cu/autor/jose-luis-rodriguez/" rel="category tag taxonomy" title="Ver todos los art&iacute;culos de José Luis Rodríguez">José Luis Rodríguez</a>
            </div>
        </div>
                            <div class="meta">
    <time datetime="2018-03-17 00:33:00">17 marzo 2018</time>
        <span class="separator">|</span>
        <span class="comment_count">
            <a href="http://www.cubadebate.cu/opinion/2018/03/17/cuba-y-su-economia-el-2017-recien-concluido-y-un-2018-que-apenas-comienza-iv/#respond" title="Comentarios en Cuba y su economía: El 2017 recién concluido y un 2018 que apenas comienza (IV)" >41</a>
        </span>
            </div>                    </div>
                </div>
                                    <br/>
                    <div class="op_excerpt">
                        <p>Las cifras de crecimiento del 2017, que ya fueron informadas, cubrieron solamente el 26% del PIB, por lo que cabe suponer un desempeño satisfactorio de los servicios sociales básicos que apoyaron el crecimiento global de 1,6% en el PIB. Adicionalmente apuntaron en sentido positivo el crecimiento logrado en el primer semestre del año (1,1%), las labores de reconstrucción a partir de los negativos efectos del huracán Irma así como el crecimiento alcanzado por el turismo de 4,4% en términos de valor agregado.</p>
                    </div>
                            </li>
            </ul>
    <div class="read-more"><a title="Opiniones" href="http://www.cubadebate.cu/categoria/opinion/">Ver más</a></div>
</aside>        </section>
        <div id="sidebar">
<aside id="soldado_ideas_widget-2" class="widget widget_conteiner widget_soldado_ideas_widget"><header class="title_wrapper"><span class="icon_class"></span><h2 class="widget-title seccion_title"><a href="http://www.fidelcastro.cu/es/" target="_blank" title="Fidel Soldado de las Ideas">Fidel Soldado de las Ideas</a></h2></header>            <div class="content">
                <div class="frase">
                    &nbsp;&ldquo;La lucha por la paz significa tambi&eacute;n la lucha por la independencia de los pueblos, significa la lucha por la libertad de las colonias, significa la lucha por el desarrollo econ&oacute;mico de los pa&iacute;ses m&aacute;s pobres, significa la lucha por librar a los pueblos de la explotaci&oacute;n y el dominio imperialistas.&rdquo;  
                </div>
                <div class="info">
                    <i>i</i>    
                    Discurso en el Acto en que le fuera entregado el Premio “Lenin" de la Paz, en el Teatro “Chaplin", 19 de Marzo de 1962  
                </div>
            </div>    
            <div class="enlace">
                <a href="http://www.fidelcastro.cu/es/" target="_blank" title="Fidel Soldado de las Ideas">www.fidelcastro.cu</a>
            </div>
            </aside><aside id="custom_html-2" class="widget_text widget widget_conteiner widget_custom_html"><div class="textwidget custom-html-widget"><a title="Razones de Cuba" href="http://razonesdecuba.cubadebate.cu/" target="_blanck"><img width="300" height="50" alt="Razones de Cuba" title="Razones de Cuba"  src="http://media.cubadebate.cu/wp-content/gallery/noticias/razonesdecuba.jpg" /></a>
<a title="Mesa Redonda de la Televisión Cubana" href="http://mesaredonda.cubadebate.cu/" target="_blanck"><img width="300" height="50" alt="Mesa Redonda de la Televisión Cubana" title="Mesa Redonda de la Televisión Cubana"  src="http://media.cubadebate.cu/wp-content/gallery/noticias/mesaredonda.jpg" /></a>
<iframe src="http://cartel.cubadebate.cu/cartel300.php?alb=12" style="border: 0pt solid black; height: 250px; overflow: hidden;" scrolling="no" frameborder="0" allowtransparency="true"></iframe></div></aside><aside id="col_tax-2" class="widget widget_conteiner col_tax"><header class="title_wrapper"><span class="icon_class columna_black"></span><h2 class="widget-title seccion_title">Columnas</h2></header><ul><li class="coltax-posts-item-105191"><a href="http://www.cubadebate.cu/columna/podcast/">Podcast</a></li><li class="coltax-posts-item-107787"><a href="http://www.cubadebate.cu/columna/goles-son-amores/">Goles son amores</a></li><li class="coltax-posts-item-106589"><a href="http://www.cubadebate.cu/columna/para-pensar/">Para Pensar...</a></li><li class="coltax-posts-item-108113"><a href="http://www.cubadebate.cu/columna/el-ritmo-boricua/">El Ritmo Boricua</a></li><li class="coltax-posts-item-101579"><a href="http://www.cubadebate.cu/columna/la-opinion-del-lector/">La Opinión del Lector</a></li><li class="coltax-posts-item-7907"><a href="http://www.cubadebate.cu/columna/la-imagen-del-dia/">La imagen del día</a></li><li class="coltax-posts-item-8135"><a href="http://www.cubadebate.cu/columna/con-humor/">Con Humor</a></li><li class="coltax-posts-item-105127"><a href="http://www.cubadebate.cu/columna/canal-usb/">Canal USB</a></li><li class="coltax-posts-item-107761"><a href="http://www.cubadebate.cu/columna/american-curios/">American Curios</a></li><li class="coltax-posts-item-108081"><a href="http://www.cubadebate.cu/columna/sabor-tradicion/">Sabor y Tradición</a></li></ul></aside><aside id="autores-4" class="widget widget_conteiner autor_list"><header class="title_wrapper"><span class="icon_class"></span><h2 class="widget-title seccion_title">Autores más publicados</h2></header><div class="list_wrapper"><ul class="first_col"><li class="cat-item"><a href="/autor/ismael-francisco">Ismael Francisco</a></li><li class="cat-item"><a href="/autor/fidel-castro-ruz">Fidel Castro Ruz</a></li><li class="cat-item"><a href="/autor/rosa-miriam-elizalde">Rosa Miriam Elizalde</a></li><li class="cat-item"><a href="/autor/hugo-rius">Hugo Ríus</a></li><li class="cat-item"><a href="/autor/jean-guy-allard">Jean-Guy Allard</a></li><li class="cat-item"><a href="/autor/paquita-de-armas">Paquita Armas Fonseca</a></li><li class="cat-item"><a href="/autor/oscar-figueredo-reinaldo">Oscar Figueredo Reinaldo</a></li><li class="cat-item"><a href="/autor/randy-alonso">Randy Alonso Falcón</a></li></ul><ul class="second_col"><li class="cat-item"><a href="/autor/michel-contreras">Michel Contreras</a></li><li class="cat-item"><a href="/autor/angel-guerra-cabrera">Angel Guerra Cabrera</a></li><li class="cat-item"><a href="/autor/irene-perez">Irene Pérez</a></li><li class="cat-item"><a href="/autor/patricio-montesinos">Patricio Montesinos</a></li><li class="cat-item"><a href="/autor/david-brooks">David Brooks</a></li><li class="cat-item"><a href="/autor/iroel-sanchez">Iroel Sánchez</a></li><li class="cat-item"><a href="/autor/lisandro-otero">Lisandro Otero</a></li></ul></div><select class="subcat-dropdown_class" id="subcat-dropdown_id" onchange="document.location.href=this.options[this.selectedIndex].value;" name="subcat-dropdown" data-placeholder="Buscar Autor"> <option></option><option value="/autor/10k"> 10K </option><option value="/autor/abdiel-bermudez-bermudez"> Abdiel Bermudez Bermudez </option><option value="/autor/abel-ernesto-rubio-estrada"> Abel Ernesto Rubio Estrada </option><option value="/autor/abel-gonzalez-santamaria"> Abel González Santamaría </option><option value="/autor/abel-padron-padilla"> Abel Padrón Padilla </option><option value="/autor/abel-prieto-jimenez"> Abel Prieto Jiménez </option><option value="/autor/abelardo-castillo"> Abelardo Castillo </option><option value="/autor/abner-barrera-rivera"> Abner Barrera Rivera </option><option value="/autor/adan-chavez-frias"> Adán Chávez Frías </option><option value="/autor/adan-iglesias-toledo"> Adán Iglesias Toledo </option><option value="/autor/adolfo-perez-esquivel"> Adolfo Pérez Esquivel </option><option value="/autor/agustin-lage-davila"> Agustín Lage Dávila </option><option value="/autor/aixa-hevia"> Aixa Hevia </option><option value="/autor/alberto-cruz"> Alberto Cruz </option><option value="/autor/alberto-juantorena"> Alberto Juantorena </option><option value="/autor/alberto-muller-rojas"> Alberto Müller Rojas </option><option value="/autor/albor-ruiz"> Albor Ruiz </option><option value="/autor/aleida-godinez-soler"> Aleida Godínez Soler </option><option value="/autor/alejandra-conti"> Alejandra Conti </option><option value="/autor/alejandra-garcia"> Alejandra García </option><option value="/autor/alejandro-castro-espin"> Alejandro Castro Espín </option><option value="/autor/alejandro-dausa"> Alejandro Dausá </option><option value="/autor/alejandro-ernesto"> Alejandro Ernesto </option><option value="/autor/alejandro-nadal"> Alejandro Nadal </option><option value="/autor/alejandro-ramirez-anderson"> Alejandro Ramírez Anderson </option><option value="/autor/alexcastro"> Alex Castro </option><option value="/autor/alexis-schlachter"> Alexis Schlachter </option><option value="/autor/alfredo-guevara"> Alfredo Guevara </option><option value="/autor/alfredo-jalife"> Alfredo Jalife </option><option value="/autor/alfredo-prieto"> Alfredo Prieto </option><option value="/autor/alfredo-rada-velez"> Alfredo Rada Vélez </option><option value="/autor/alfredo-serrano-mancilla"> Alfredo Serrano Mancilla </option><option value="/autor/alfredo-vera"> Alfredo Vera </option><option value="/autor/ali-rodriguez-araque"> Alí Rodríguez Araque </option><option value="/autor/alianet-beltran-alvarez"> Alianet Beltrán Álvarez </option><option value="/autor/alice-walker"> Alice Walker </option><option value="/autor/alicia-alonso"> Alicia Alonso </option><option value="/autor/alicia-barcenas"> Alicia Bárcenas Ibarra </option><option value="/autor/alicia-jrapko"> Alicia Jrapko </option><option value="/autor/aliet-arzola-lima"> Aliet Arzola Lima </option><option value="/autor/alina-perera"> Alina Perera Robbio </option><option value="/autor/allan-g-greenberg"> Allan G. Greenberg </option><option value="/autor/allan-mcdonald"> Allan McDonald </option><option value="/autor/amado-del-pino"> Amado del Pino </option><option value="/autor/amado-riol-pirez"> Amado Riol Pírez </option><option value="/autor/amauri-chamorro"> Amauri Chamorro </option><option value="/autor/amauris-betancourt"> Amauris Betancourt </option><option value="/autor/amaury-perez-vidal"> Amaury Pérez Vidal </option><option value="/autor/ambrosio-fornet"> Ambrosio Fornet </option><option value="/autor/amelia-duarte-de-la-rosa"> Amelia Duarte de la Rosa </option><option value="/autor/amy-goodman"> Amy Goodman </option><option value="/autor/ana-cairo"> Ana Cairo </option><option value="/autor/ana-esther-cecena"> Ana Esther Ceceña </option><option value="/autor/ana-maria-radaelli"> Ana María Radaelli </option><option value="/autor/analeida-puerto"> Analeida Puerto </option><option value="/autor/andres-gomez"> Andrés Gómez </option><option value="/autor/andres-sallari"> Andrés Sallari </option><option value="/autor/andres-sorel"> Andrés Sorel </option><option value="/autor/andy-bermellon-campos"> Andy Bermellón Campos </option><option value="/autor/andy-jorge-blanco"> Andy Jorge Blanco </option><option value="/autor/andy-robinson"> Andy Robinson </option><option value="/autor/angel-guerra-cabrera"> Angel Guerra Cabrera </option><option value="/autor/angeles-diez"> Ángeles Diez </option><option value="/autor/angeles-maestro"> Angeles Maestro </option><option value="/autor/angelica-paredes"> Angélica Paredes </option><option value="/autor/anneris-ivette-leyva"> Anneris Ivette Leyva </option><option value="/autor/antonio-diaz-susavila"> Antonio Díaz Susavila </option><option value="/autor/antonio-guerrero-rodriguez"> Antonio Guerrero Rodríguez </option><option value="/autor/antonio-hernandez-mena"> Antonio Hernández Mena </option><option value="/autor/antonio-maira"> Antonio Maira </option><option value="/autor/antonio-martorell"> Antonio Martorell </option><option value="/autor/antonio-molto"> Antonio Moltó </option><option value="/autor/antonio-muniz"> Antonio Muñiz </option><option value="/autor/antonio-nunez-jimenez"> Antonio Núñez Jimenez </option><option value="/autor/antonio-peredo-leigue"> Antonio Peredo Leigue </option><option value="/autor/antonio-rodriguez-salvador"> Antonio Rodríguez Salvador </option><option value="/autor/antonio-skarmeta"> Antonio Skármeta </option><option value="/autor/aram-aharonian"> Aram Aharonian </option><option value="/autor/arelys-maria-echevarria"> Arelys María Echevarría </option><option value="/autor/ariel-coya"> Ariel B. Coya </option><option value="/autor/ariel-dorfman"> Ariel Dorfman </option><option value="/autor/ariel-noyola-rodriguez"> Ariel Noyola Rodríguez </option><option value="/autor/ariel-terrero"> Ariel Terrero </option><option value="/autor/arleen-rodriguez-derivet"> Arleen Rodríguez Derivet </option><option value="/autor/arlet-castillo-gonzalez"> Arlet Castillo González </option><option value="/autor/armando-franco"> Armando Franco </option><option value="/autor/armando-hart-davalos"> Armando Hart Dávalos </option><option value="/autor/arnaldo-coro-antich"> Arnaldo Coro Antich </option><option value="/autor/arnaldo-perez-guerra"> Arnaldo Pérez Guerra </option><option value="/autor/arnold-august"> Arnold August </option><option value="/autor/arundhati-roy"> Arundhati Roy </option><option value="/autor/atilio-boron"> Atilio Borón </option><option value="/autor/aurelio-alonso"> Aurelio Alonso </option><option value="/autor/aurora-fernandez-gonzalez"> Aurora Fernández González </option><option value="/autor/aylin-febles"> Aylín Febles </option><option value="/autor/aynel-martinez-hernandez"> Aynel Martínez Hernández </option><option value="/autor/beatriz-albert-pino"> Beatriz Albert Pino </option><option value="/autor/belen-gopegui"> Belén Gopegui </option><option value="/autor/belkys-perez-cruz"> Belkys Pérez Cruz </option><option value="/autor/benigno-daquinta"> Benigno Daquinta </option><option value="/autor/benito-joaquin-milanes"> Benito Joaquín Milanés </option><option value="/autor/benjamin-torres-gotay"> Benjamín Torres Gotay </option><option value="/autor/bernardo-alvarez-herrera"> Bernardo Alvarez Herrera </option><option value="/autor/bernie-dwyer"> Bernie Dwyer </option><option value="/autor/bertha-mojena"> Bertha Mojena </option><option value="/autor/bill-hackwell"> Bill Hackwell </option><option value="/autor/blanche-petrich"> Blanche Petrich </option><option value="/autor/boaventura-de-sousa-santos"> Boaventura de Sousa Santos </option><option value="/autor/bruce-e-levine"> Bruce E. Levine </option><option value="/autor/bruno-rodriguez-parrilla"> Bruno Rodríguez Parrilla </option><option value="/autor/camila-vallejo"> Camila Vallejo </option><option value="/autor/caridad-carrobello"> Caridad Carrobello </option><option value="/autor/carlo-frabetti"> Carlo Frabetti </option><option value="/autor/carlos-alberto-cremata"> Carlos Alberto Cremata </option><option value="/autor/carlos-aznarez"> Carlos Aznárez </option><option value="/autor/carlos-carnicero"> Carlos Carnicero </option><option value="/autor/carlos-cespedes-piedra"> Carlos Céspedes Piedra </option><option value="/autor/carlos-del-porto"> Carlos del Porto </option><option value="/autor/carlos-fazio"> Carlos Fazio </option><option value="/autor/carlos-fernandez-liria"> Carlos Fernández Liria </option><option value="/autor/carlos-fernandez-vega"> Carlos Fernandez-Vega </option><option value="/autor/carlos-fresneda"> Carlos Fresneda </option><option value="/autor/carlos-lage-davila"> Carlos Lage Dávila </option><option value="/autor/carlos-luque-zayas-bazan"> Carlos Luque Zayas Bazán </option><option value="/autor/carlos-manuel-alvarez"> Carlos Manuel Álvarez </option><option value="/autor/carlos-mario-castro"> Carlos Mario Castro </option><option value="/autor/carlos-martinez"> Carlos Martínez </option><option value="/autor/carlos-miguel-pereira-hernandez"> Carlos Miguel Pereira Hernández </option><option value="/autor/carlos-muniz-perez"> Carlos Muñiz Pérez </option><option value="/autor/carlos-rodriguez-almaguer"> Carlos Rodríguez Almaguer </option><option value="/autor/carlos-rodriguez-castellanos"> Carlos Rodríguez Castellanos </option><option value="/autor/carlos-tena"> Carlos Tena </option><option value="/autor/carmen-lira-saade"> Carmen Lira Saade </option><option value="/autor/carmen-rosa-baez"> Carmen Rosa Báez </option><option value="/autor/celia-hart"> Celia María Hart Santamaría </option><option value="/autor/cesar-gomez-chacon"> César Gómez Chacón </option><option value="/autor/charo-martin"> Charo Martín </option><option value="/autor/chelsea-manning"> Chelsea Manning </option><option value="/autor/cindy-sheehan"> Cindy Sheehan </option><option value="/autor/cinthyagarcia-casanas"> Cinthya García Casañas </option><option value="/autor/cintio-vitier"> Cintio Vitier </option><option value="/autor/ciro-benemelis"> Ciro Benemelis </option><option value="/autor/ciro-bianchi-ross"> Ciro Bianchi Ross </option><option value="/autor/claudia-montes-de-oca"> Claudia Montes de Oca </option><option value="/autor/claudia-yilen"> Claudia Yilén Paz Joa </option><option value="/autor/claudio-katz"> Claudio Katz </option><option value="/autor/concepcion-badillo"> Concepción Badillo </option><option value="/autor/cristina-escobar"> Cristina Escobar </option><option value="/autor/cristina-fernandez-de-kirchner"> Cristina Fernández de Kirchner </option><option value="/autor/cuauhtemoc-cardenas"> Cuauhtémoc Cárdenas </option><option value="/autor/cusco-tarradell"> Cuscó Tarradell </option><option value="/autor/daily-sanchez-lemus"> Daily Sánchez Lemus </option><option value="/autor/dalia-gonzalez-delgado"> Dalia González Delgado </option><option value="/autor/damien-barroso-vega"> Damien Barroso Vega </option><option value="/autor/daniel-chavarria"> Daniel Chavarría </option><option value="/autor/daniel-ortega-saavedra"> Daniel Ortega Saavedra </option><option value="/autor/danier-ernesto-gonzalez"> Danier Ernesto González </option><option value="/autor/danny-glover"> Danny Glover </option><option value="/autor/danny-gutierrez"> Danny Gutiérrez </option><option value="/autor/danny-rivera"> Danny Rivera </option><option value="/autor/dario-delgado-cura"> Darío Delgado Cura </option><option value="/autor/dario-gabriel-sanchez-garcia"> Darío Gabriel Sánchez García </option><option value="/autor/dario-machado-rodriguez"> Darío Machado Rodríguez </option><option value="/autor/darsi-fernandez"> Darsi Fernández </option><option value="/autor/dave-lindorff"> Dave Lindorff </option><option value="/autor/david-brooks"> David Brooks </option><option value="/autor/david-cole"> David Cole </option><option value="/autor/david-e-sanger"> David E. Sanger </option><option value="/autor/david-vazquez-abella"> David Vázquez Abella </option><option value="/autor/dax-toscano-segovia"> Dax Toscano Segovia </option><option value="/autor/daylen-vega-muguercia"> Daylén Vega Muguercia </option><option value="/autor/deisy-francis-mexidor"> Deisy Francis Mexidor </option><option value="/autor/delcy-rodriguez"> Delcy Rodríguez </option><option value="/autor/denis-moynihan"> Denis Moynihan </option><option value="/autor/desiderio-navarro"> Desiderio Navarro </option><option value="/autor/desmond-boylan"> Desmond Boylan </option><option value="/autor/diana-nyad"> Diana Nyad </option><option value="/autor/dianet-doimeadios"> Dianet Doimeadios Guerrero </option><option value="/autor/dianik-flores"> Dianik Flores </option><option value="/autor/dick-emanuelsson"> Dick Emanuelsson </option><option value="/autor/diego-armando-maradona"> Diego Armando Maradona </option><option value="/autor/diego-m-vidal"> Diego M. Vidal </option><option value="/autor/dilma-rousseff"> Dilma Rousseff </option><option value="/autor/dinella-garcia-acosta"> Dinella García Acosta </option><option value="/autor/dixie-edith"> Dixie Edith </option><option value="/autor/domingo-alberto-rangel"> Domingo Alberto Rangel </option><option value="/autor/domingo-amuchastegui"> Domingo Amuchastegui </option><option value="/autor/dr-jose-rubiera"> Dr. José Rubiera </option><option value="/autor/dr-lorenzo-somarriba-lopez"> Dr. Lorenzo Somarriba López </option><option value="/autor/edgardo-ramirez"> Edgardo Ramírez </option><option value="/autor/edmundo-garcia"> Edmundo García </option><option value="/autor/eduardo-a-prieto"> Eduardo A. Prieto </option><option value="/autor/eduardo-aliverti"> Eduardo Aliverti </option><option value="/autor/eduardo-dimas"> Eduardo Dimas </option><option value="/autor/eduardo-febbro"> Eduardo Febbro </option><option value="/autor/eduardo-galeano"> Eduardo Galeano </option><option value="/autor/eduardo-heras-leon"> Eduardo Heras León </option><option value="/autor/eduardo-mora-basart"> Eduardo Mora Basart </option><option value="/autor/eduardo-puente-fernandez"> Eduardo Puente Fernández </option><option value="/autor/eduardo-rothe"> Eduardo Rothe </option><option value="/autor/eduardo-torres-cuevas"> Eduardo Torres Cuevas </option><option value="/autor/eduardo-tato-pavlovsky"> Eduardo “Tato” Pavlovsky </option><option value="/autor/efrain-otano-gerardo"> Efraín Otaño Gerardo </option><option value="/autor/elda-cento-gomez"> Elda Cento Gómez </option><option value="/autor/eleazar-diaz-rangel"> Eleazar Díaz Rangel </option><option value="/autor/elena-poniatowska"> Elena Poniatowska </option><option value="/autor/eliades-acosta-matos"> Eliades Acosta Matos </option><option value="/autor/elias-jaua"> Elias Jaua </option><option value="/autor/elier-ramirez-canedo"> Elier Ramírez Cañedo </option><option value="/autor/elma-beatriz-rosado"> Elma Beatriz Rosado </option><option value="/autor/elsa-claro"> Elsa Claro </option><option value="/autor/elson-concepcion-perez"> Elson Concepción Pérez </option><option value="/autor/emiliano-mariscal"> Emiliano Mariscal </option><option value="/autor/emilio-marin"> Emilio Marín </option><option value="/autor/emir-sader"> Emir Sader </option><option value="/autor/ena-elsa-velazquez-cobiella"> Ena Elsa Velázquez Cobiella </option><option value="/autor/enrique-arturo-antonio-houston-touceda"> Enrique Arturo Antonio Houston Touceda </option><option value="/autor/enrique-atienzar-rivero"> Enrique Atiénzar Rivero </option><option value="/autor/enrique-bethencourt"> Enrique Bethencourt </option><option value="/autor/enrique-de-la-osa"> Enrique de la Osa </option><option value="/autor/enrique-ojito"> Enrique Ojito </option><option value="/autor/enrique-ubieta"> Enrique Ubieta </option><option value="/autor/eric-nepomuceno"> Eric Nepomuceno </option><option value="/autor/eric-toussaint"> Eric Toussaint </option><option value="/autor/erika-ortega-sanoja"> Érika Ortega Sanoja </option><option value="/autor/ernesto-cardenal"> Ernesto Cardenal </option><option value="/autor/ernesto-carmona"> Ernesto Carmona </option><option value="/autor/ernesto-che-guevara"> Ernesto Che Guevara </option><option value="/autor/ernesto-estevez-rams"> Ernesto Estévez Rams </option><option value="/autor/ernesto-juan-castellanos"> Ernesto Juan Castellanos </option><option value="/autor/ernesto-limia-diaz"> Ernesto Limia Díaz </option><option value="/autor/ernesto-mastrascusa"> Ernesto Mastrascusa </option><option value="/autor/ernesto-montero-acuna"> Ernesto Montero Acuña </option><option value="/autor/ernesto-vera"> Ernesto Vera </option><option value="/autor/esteban-lazo-hernandez"> Esteban Lazo Hernandez </option><option value="/autor/esteban-morales-dominguez"> Esteban Morales Domínguez </option><option value="/autor/estrella-galan"> Estrella Galán </option><option value="/autor/eugene-robinson"> Eugene Robinson </option><option value="/autor/eugenio-martinez"> Eugenio Martínez </option><option value="/autor/eugenio-suarez-perez"> Eugenio Suárez Pérez </option><option value="/autor/eusebio-leal"> Eusebio Leal </option><option value="/autor/eva-golinger"> Eva Golinger </option><option value="/autor/eva-sastre-forest"> Eva Sastre Forest </option><option value="/autor/evgeny-morozov"> Evgeny Morozov </option><option value="/autor/evo-morales-ayma"> Evo Morales Ayma </option><option value="/autor/eyleen-rios-lopez"> Eyleen Ríos López </option><option value="/autor/f-vladimir-perez-casal"> F. Vladimir Pérez Casal </option><option value="/autor/fabian-escalante"> Fabián Escalante </option><option value="/autor/falco"> Falco </option><option value="/autor/fanny-palacios"> Fanny Palacios </option><option value="/autor/federico-mayor-zaragoza"> Federico Mayor Zaragoza </option><option value="/autor/felipe-perez-roque"> Felipe Pérez Roque </option><option value="/autor/fernando-buen-abad"> Fernando Buen Abad </option><option value="/autor/fernando-gonzalez-llort"> Fernando González Llort </option><option value="/autor/fernando-leon-jacomino"> Fernando León Jacomino </option><option value="/autor/fernando-martinez-heredia"> Fernando Martínez Heredia </option><option value="/autor/fernando-medina"> Fernando Medina </option><option value="/autor/fernando-perez"> Fernando Pérez </option><option value="/autor/fidel-castro-smirnov"> Fidel Antonio Castro Smirnov </option><option value="/autor/fidel-castro-diaz-balart"> Fidel Castro Díaz-Balart </option><option value="/autor/fidel-castro-ruz"> Fidel Castro Ruz </option><option value="/autor/fidel-diaz-castro"> Fidel Díaz Castro </option><option value="/autor/fina-garcia-marruz"> Fina García Marruz </option><option value="/autor/flor-de-paz"> Flor de Paz </option><option value="/autor/francisco-arias-fernandez"> Francisco Arias Fernández </option><option value="/autor/francisco-rodriguez-cruz"> Francisco Rodríguez Cruz </option><option value="/autor/francois-houtart-2"> François Houtart </option><option value="/autor/frank-gonzalez-garcia"> Frank González García </option><option value="/autor/frank-padron"> Frank Padrón </option><option value="/autor/freddy-bernal"> Freddy Bernal </option><option value="/autor/frei-betto"> Frei Betto </option><option value="/autor/frida-berrigan"> Frida Berrigan </option><option value="/autor/frida-modak"> Frida Modak </option><option value="/autor/gabriel-davalos"> Gabriel Dávalos </option><option value="/autor/gabriel-garcia-marquez"> Gabriel García Márquez </option><option value="/autor/gabriela-rivadeneira"> Gabriela Rivadeneira </option><option value="/autor/gaspar-velasquez-morillo"> Gaspar Velásquez Morillo </option><option value="/autor/gennaro-carotenuto"> Gennaro Carotenuto </option><option value="/autor/genoveva-lopez-morales"> Genoveva López Morales </option><option value="/autor/gerardo-hernandez-nordelo"> Gerardo Hernández Nordelo </option><option value="/autor/german-sanchez-otero"> Germán Sánchez Otero </option><option value="/autor/gerson-borrero"> Gerson Borrero </option><option value="/autor/gideon-levy"> Gideon Levy </option><option value="/autor/gilad-atzmon"> Gilad Atzmon </option><option value="/autor/gilberto-lopez-y-rivas"> Gilberto López y Rivas </option><option value="/autor/gilberto-santa-rosa"> Gilberto Santa Rosa </option><option value="/autor/giorgio-trucchi"> Giorgio Trucchi </option><option value="/autor/giraldo-mazola"> Giraldo Mazola </option><option value="/autor/gladys-cecilia-hernandez-pedraza"> Gladys Cecilia Hernández Pedraza </option><option value="/autor/glenn-greenwald"> Glenn Greenwald </option><option value="/autor/gloria-gaitan"> Gloria Gaitán </option><option value="/autor/gloria-leon"> Gloria León </option><option value="/autor/godefroid-tchamlesso-diur"> Godefroid Tchamlesso Diur </option><option value="/autor/graciela-ramirez"> Graciela Ramírez </option><option value="/autor/gracziella-pogolotti"> Graziella Pogolotti </option><option value="/autor/guille-vilar"> Guille Vilar </option><option value="/autor/guillermo-cabrera"> Guillermo Cabrera </option><option value="/autor/guillermo-jimenez-melgarejo"> Guillermo Jiménez Melgarejo </option><option value="/autor/guillermo-nova"> Guillermo Nova </option><option value="/autor/guillermo-rodriguez-rivera"> Guillermo Rodríguez Rivera </option><option value="/autor/guillermo-toledo"> Guillermo Toledo </option><option value="/autor/gustavo-arcos"> Gustavo Arcos </option><option value="/autor/hamlet-hermann"> Hamlet Hermann </option><option value="/autor/hassan-perez-casabona"> Hassán Pérez Casabona </option><option value="/autor/hector-rodriguez-llompart"> Héctor Rodríguez Llompart </option><option value="/autor/hedelberto-lopez-blanch"> Hedelberto López Blanch </option><option value="/autor/heidi-calderon-sanchez"> Heidi Calderón Sánchez </option><option value="/autor/helmo-hernandez"> Helmo Hernández </option><option value="/autor/henning-mankell"> Henning Mankell </option><option value="/autor/heriberto-rosabal"> Heriberto Rosabal </option><option value="/autor/hernan-uribe"> Hernán Uribe </option><option value="/autor/hernando-calvo-ospina"> Hernando Calvo Ospina </option><option value="/autor/homero-acosta-alvarez"> Homero Acosta Álvarez </option><option value="/autor/horacio-labastida"> Horacio Labastida </option><option value="/autor/howard-zinn"> Howard Zinn </option><option value="/autor/hugo-alberto-de-pedro"> Hugo Alberto de Pedro </option><option value="/autor/hugo-chavez-frias"> Hugo Chávez Frías </option><option value="/autor/hugo-moldiz-mercado"> Hugo Moldiz Mercado </option><option value="/autor/hugo-morales-karell"> Hugo Morales Karell </option><option value="/autor/hugo-rius"> Hugo Ríus </option><option value="/autor/humberto-de-la-calle"> Humberto de la Calle </option><option value="/autor/humberto-perez-gonzalez"> Humberto Pérez González </option><option value="/autor/ibrahim-hidalgo"> Ibrahim Hidalgo </option><option value="/autor/ida-garberi"> Ida Garberi </option><option value="/autor/ignacio-echevarria"> Ignacio Echevarría </option><option value="/autor/ignacio-ramonet"> Ignacio Ramonet </option><option value="/autor/ilka-oliva-corado"> Ilka Oliva Corado </option><option value="/autor/immanuel-wallerstein"> Immanuel Wallerstein </option><option value="/autor/iraida-calzadilla-rodriguez"> Iraida Calzadilla Rodríguez </option><option value="/autor/irene-perez"> Irene Pérez </option><option value="/autor/irma-gonzalez-salanueva"> Irma González Salanueva </option><option value="/autor/iroel-sanchez"> Iroel Sánchez </option><option value="/autor/isabel-moya"> Isabel Moya </option><option value="/autor/isabela-rodriguez"> Isabela Rodríguez </option><option value="/autor/ismael-clark-arxer"> Ismael Clark Arxer </option><option value="/autor/ismael-francisco"> Ismael Francisco </option><option value="/autor/ismael-serrano"> Ismael Serrano </option><option value="/autor/israel-rojas"> Israel Rojas </option><option value="/autor/ivan-barreto-gelles"> Ivan Barreto Gelles </option><option value="/autor/ivan-marquez"> Iván Márquez </option><option value="/autor/ivan-soca"> Iván Soca </option><option value="/autor/ivyliet-ventura-kessel"> Ivyliet Ventura Kessel </option><option value="/autor/jackson-de-la-cruz"> Jackson de la Cruz </option><option value="/autor/jacques-diouf"> Jacques Diouf </option><option value="/autor/james-petras"> James Petras </option><option value="/autor/jamp"> JAMP </option><option value="/autor/jane-franklin"> Jane Franklin </option><option value="/autor/javier-couso"> Javier Couso </option><option value="/autor/javier-montenegro"> Javier Montenegro </option><option value="/autor/javier-ortiz"> Javier Ortiz </option><option value="/autor/javier-parra"> Javier Parra </option><option value="/autor/javier-rodriguez"> Javier Rodríguez </option><option value="/autor/javier-sanzo"> Javier Sanzo </option><option value="/autor/jean-guy-allard"> Jean-Guy Allard </option><option value="/autor/jean-luc-melenchon"> Jean-Luc Mélenchon </option><option value="/autor/jefferson-morley"> Jefferson Morley </option><option value="/autor/jennifer-romero-andreu"> Jennifer Romero Andreu </option><option value="/autor/jennifer-veliz-gutierrez"> Jennifer Veliz Gutierrez </option><option value="/autor/jennifer-zubizarreta-arias"> Jennifer Zubizarreta Arias </option><option value="/autor/jesus-arboleya-cervera"> Jesús Arboleya Cervera </option><option value="/autor/jesus-arencibia-lorenzo"> Jesús Arencibia Lorenzo </option><option value="/autor/jesus-g-bayolo"> Jesus G. Bayolo </option><option value="/autor/jesus-martin-barbero"> Jesús Martín Barbero </option><option value="/autor/jesus-martinez-alfonso"> Jesús Martínez Alfonso </option><option value="/autor/jesus-suarez-valmana"> Jesús Suárez Valmaña </option><option value="/autor/jhonah-diaz-gonzalez"> Jhonah Díaz González </option><option value="/autor/jim-cason"> Jim Cason </option><option value="/autor/jim-lobe"> Jim Lobe </option><option value="/autor/joan-manuel-serrat"> Joan Manuel Serrat </option><option value="/autor/joao-pedro-stedile"> Joao Pedro Stedile </option><option value="/autor/joaquin-borges-triana"> Joaquín Borges Triana </option><option value="/autor/joaquin-rivery-tur"> Joaquín Rivery Tur </option><option value="/autor/joel-del-rio"> Joel del Río </option><option value="/autor/joel-garcia"> Joel García </option><option value="/autor/johana-tablada"> Johana Tablada </option><option value="/autor/john-banville"> John Banville </option><option value="/autor/john-gray"> John Gray </option><option value="/autor/john-hemingway"> John Hemingway </option><option value="/autor/john-pilger"> John Pilger </option><option value="/autor/john-saxe-fernandez"> John Saxe-Fernández </option><option value="/autor/jorge-aguirre-nunez"> Jorge Aguirre Núñez </option><option value="/autor/jorge-angel-hernandez"> Jorge Ángel Hernández </option><option value="/autor/jorge-bolanos"> Jorge Bolaños </option><option value="/autor/jorge-de-la-nuez-orozco"> Jorge de la Nuez Orozco </option><option value="/autor/jorge-enrique-adoum"> Jorge Enrique Adoum </option><option value="/autor/jorge-enrique-jerez-belisario"> Jorge Enrique Jerez Belisario </option><option value="/autor/jorge-fernandez-era"> Jorge Fernández Era </option><option value="/autor/jorge-fornet"> Jorge Fornet </option><option value="/autor/jorge-gomez-barata"> Jorge Gómez Barata </option><option value="/autor/jorge-gomez-barranco"> Jorge Gómez Barranco </option><option value="/autor/jorge-leganoa-alonso"> Jorge Legañoa Alonso </option><option value="/autor/jorge-luis-sanchez-rivera"> Jorge Luis Sánchez Rivera </option><option value="/autor/jorge-luis-ubertalli"> Jorge Luis Ubertalli </option><option value="/autor/jorge-majfud"> Jorge Majfud </option><option value="/autor/jorge-manzaneda"> Jorge Manzaneda </option><option value="/autor/jorge-nunez-jover"> Jorge Núñez Jover </option><option value="/autor/jorge-risquet-valdes"> Jorge Risquet Valdés </option><option value="/autor/jorge-rodriguez"> Jorge Rodríguez </option><option value="/autor/jose-pepe-mujica"> José "Pepe" Mujica </option><option value="/autor/jose-a-martin"> José A. Martín </option><option value="/autor/jose-alejandro-rodriguez"> José Alejandro Rodríguez </option><option value="/autor/jose-antonio-bacigalupo"> José Antonio Bacigalupo </option><option value="/autor/jose-antonio-rigual"> José Antonio Rigual </option><option value="/autor/jose-armando-fernandez-salazar"> José Armando Fernández Salazar </option><option value="/autor/jose-blanco"> José Blanco </option><option value="/autor/jose-daniel-fierro"> José Daniel Fierro </option><option value="/autor/jose-dos-santos"> José Dos Santos </option><option value="/autor/jose-luis-centella"> José Luis Centella </option><option value="/autor/jose-luis-cuza-tellez-de-giron"> José Luis Cuza Téllez de Girón </option><option value="/autor/jose-luis-farinas"> José Luis Fariñas </option><option value="/autor/jose-luis-mendez-mendez"> José Luis Méndez Méndez </option><option value="/autor/jose-luis-rodriguez"> José Luis Rodríguez </option><option value="/autor/jose-manuel-zelaya-rosales"> José Manuel Zelaya Rosales </option><option value="/autor/jose-manzaneda"> José Manzaneda </option><option value="/autor/jose-maria-perez-gay"> José María Pérez Gay </option><option value="/autor/jose-maria-vitier"> José María Vitier </option><option value="/autor/jose-pertierra"> José Pertierra </option><option value="/autor/jose-oro"> José R. Oro </option><option value="/autor/jose-ramon-machado-ventura"> José Ramón Machado Ventura </option><option value="/autor/jose-ramon-rodriguez-ruiz"> José Ramón Rodríguez Ruiz </option><option value="/autor/jose-raul-concepcion-llanes"> José Raúl Concepción </option><option value="/autor/jose-saramago"> Jose Saramago </option><option value="/autor/jose-steinsleger"> José Steinsleger </option><option value="/autor/jose-vicente-rangel"> José Vicente Rangel </option><option value="/autor/josefina-vidal"> Josefina Vidal </option><option value="/autor/joseph-e-stiglitz"> Joseph E. Stiglitz </option><option value="/autor/josh-r-nelson"> Josh R. Nelson </option><option value="/autor/juan-a-martinez-de-osaba-y-goenaga"> Juan A. Martínez de Osaba y Goenaga </option><option value="/autor/juan-alfonso-fernandez-gonzalez"> Juan Alfonso Fernández González </option><option value="/autor/juan-almeida-bosque"> Juan Almeida Bosque </option><option value="/autor/juan-almendares"> Juan Almendares </option><option value="/autor/juan-antonio-fernandez"> Juan Antonio Fernández </option><option value="/autor/juan-barahona"> Juan Barahona </option><option value="/autor/juan-carlos-camano"> Juan Carlos Camaño </option><option value="/autor/juan-carlos-monedero"> Juan Carlos Monedero </option><option value="/autor/juan-fernandez-lopez"> Juan Fernández López </option><option value="/autor/juan-gelman"> Juan Gelman </option><option value="/autor/juan-jacomino"> Juan Jacomino </option><option value="/autor/juan-luis-vallina-ariznavarreta"> Juan Luis Vallina Ariznavarreta </option><option value="/autor/juan-manuel-karg"> Juan Manuel Karg </option><option value="/autor/juan-marrero"> Juan Marrero </option><option value="/autor/juan-miguel-cruz-suarez"> Juan Miguel Cruz Suárez </option><option value="/autor/juan-nuiry-sanchez"> Juan Nuiry  Sánchez </option><option value="/autor/juan-pablo-carreras"> Juan Pablo Carreras </option><option value="/autor/juan-padron"> Juan Padrón </option><option value="/autor/juan-paz-mino-cepeda"> Juan Paz y Miño Cepeda </option><option value="/autor/juan-ramon-quintana-taborga"> Juan Ramón Quintana Taborga </option><option value="/autor/juan-torres-lopez"> Juan Torres López </option><option value="/autor/juan-tortosa"> Juan Tortosa </option><option value="/autor/juana-carrasco-martin"> Juana Carrasco Martín </option><option value="/autor/julio-a-garcia-oliveras"> Julio A. García Oliveras </option><option value="/autor/julio-batista"> Julio Batista </option><option value="/autor/julio-c-gambina"> Julio C. Gambina </option><option value="/autor/julio-cesar-hernandez-perera"> Julio César Hernández Perera </option><option value="/autor/julio-cortazar"> Julio Cortázar </option><option value="/autor/julio-ferreira"> Julio Ferreira </option><option value="/autor/julio-garcia-espinosa"> Julio García Espinosa </option><option value="/autor/julio-garcia-luis"> Julio García Luis </option><option value="/autor/justo-cruz"> Justo Cruz </option><option value="/autor/justo-gonzalez-ortega"> Justo González Ortega </option><option value="/autor/juvenal-balan"> Juvenal Balán </option><option value="/autor/kaloian"> Kaloian Santos Cabrera </option><option value="/autor/karen-alonso"> Karen Alonso </option><option value="/autor/karina-marron-gonzalez"> Karina Marrón González </option><option value="/autor/katheryn-felipe"> Katheryn Felipe </option><option value="/autor/kathleen-kennedy-townsend"> Kathleen Kennedy Townsend </option><option value="/autor/katiuska-blanco"> Katiuska Blanco </option><option value="/autor/katrina-vanden-heuvel"> Katrina vanden Heuvel </option><option value="/autor/katu-arkonada"> Katu Arkonada </option><option value="/autor/kenia-a-agramonte"> Kenia A. Agramonte </option><option value="/autor/keyttia"> Keyttia Pinton Almenares </option><option value="/autor/kintto-lucas"> Kintto Lucas </option><option value="/autor/l-e-dominguez"> L Eduardo Domínguez </option><option value="/autor/l-alberto-rodriguez"> L. Alberto Rodríguez </option><option value="/autor/laidi-fernandez-de-juan"> Laidi Fernández de Juan </option><option value="/autor/laura-becquer-paseiro"> Laura Bécquer Paseiro </option><option value="/autor/laura-david"> Laura David </option><option value="/autor/laura-melo"> Laura Melo </option><option value="/autor/lawrence-wilkerson"> Lawrence Wilkerson </option><option value="/autor/lazaro-barredo"> Lázaro Barredo </option><option value="/autor/leandro-grille"> Leandro Grille </option><option value="/autor/leonard-weinglass"> Leonard Weinglass </option><option value="/autor/leonardo-boff"> Leonardo Boff </option><option value="/autor/leonardo-padura-fuentes"> Leonardo Padura Fuentes </option><option value="/autor/leonel-fernandez"> Leonel Fernández </option><option value="/autor/leonel-nodal"> Leonel Nodal </option><option value="/autor/leslie-monserrat"> Leslie Díaz Monserrat </option><option value="/autor/leticia-martinez"> Leticia Martínez </option><option value="/autor/leyde-e-rodriguez-hernandez"> Leyde E. Rodríguez Hernández </option><option value="/autor/leyla-carrillo"> Leyla Carrillo </option><option value="/autor/leysi-rubio"> Leysi Rubio A. </option><option value="/autor/lianet-hernandez"> Lianet Hernández </option><option value="/autor/liborio-noval"> Liborio Noval </option><option value="/autor/lil-rodriguez"> Lil Rodríguez </option><option value="/autor/liliam-marrero"> Liliam Marrero </option><option value="/autor/lilian-cid"> Lilian Cid </option><option value="/autor/lillian-lechuga"> Lillian Lechuga </option><option value="/autor/lisandro-otero"> Lisandro Otero </option><option value="/autor/lisset-isabel-ricardo"> Lisset Isabel Ricardo </option><option value="/autor/liudmila-pena-herrera"> Liudmila Peña Herrera </option><option value="/autor/livia-reyes"> Livia Reyes </option><option value="/autor/llanisca-lugo"> Llanisca Lugo </option><option value="/autor/luciano-castillo"> Luciano Castillo </option><option value="/autor/luis-a-montero-cabrera"> Luis A. Montero Cabrera </option><option value="/autor/luis-baez"> Luis Báez </option><option value="/autor/luis-barrios"> Luis Barrios </option><option value="/autor/luis-bilbao"> Luis Bilbao </option><option value="/autor/luis-britto-garcia"> Luis Britto García </option><option value="/autor/luis-bruschtein"> Luis Bruschtein </option><option value="/autor/luis-hernandez-navarro"> Luis Hernández Navarro </option><option value="/autor/luis-lopez-viera"> Luis López Viera </option><option value="/autor/luis-manuel-arce-isaac"> Luis Manuel Arce Isaac </option><option value="/autor/luis-mario-rodriguez-sunol"> Luis Mario Rodríguez Suñol </option><option value="/autor/luis-rumbaut"> Luis Rumbaut </option><option value="/autor/luis-sepulveda"> Luis Sepúlveda </option><option value="/autor/luis-sexto"> Luis Sexto </option><option value="/autor/luis-toledo-sande"> Luis Toledo Sande </option><option value="/autor/m-h-lagarde"> M. H. Lagarde </option><option value="/autor/magda-resik-aguirre"> Magda Resik Aguirre </option><option value="/autor/manuel-alejandro-hernandez-barrios"> Manuel Alejandro Hernández Barrios </option><option value="/autor/manuel-cabieses-donoso"> Manuel Cabieses Donoso </option><option value="/autor/manuel-calvino"> Manuel Calviño </option><option value="/autor/manuel-castells"> Manuel Castells </option><option value="/autor/manuel-david-orrio"> Manuel David Orrio </option><option value="/autor/manuel-e-yepe"> Manuel E. Yepe </option><option value="/autor/marcelino-vazquez-hernandez"> Marcelino Vázquez Hernández </option><option value="/autor/marcelo-colussi"> Marcelo Colussi </option><option value="/autor/marco-aurelio-garcia"> Marco Aurelio García </option><option value="/autor/marco-teruggi"> Marco Teruggi </option><option value="/autor/marcos-roitman-rosenmann"> Marcos Roitman Rosenmann </option><option value="/autor/margaret-chan"> Margaret Chan </option><option value="/autor/margarita-alarcon-perea"> Margarita Alarcón Perea </option><option value="/autor/margarita-gonzalez-fernandez"> Margarita González Fernández </option><option value="/autor/maria-carla-oconnor"> María Carla OˊConnor </option><option value="/autor/maria-del-carmen-pupo-martinez"> María del Carmen Pupo Martínez </option><option value="/autor/maria-del-carmen-ramon"> María del Carmen Ramón </option><option value="/autor/maria-elena-alvarez-acosta"> María Elena Álvarez Acosta </option><option value="/autor/maria-lucia-exposito"> María Lucía Expósito </option><option value="/autor/maria-rojas"> María Rojas </option><option value="/autor/marianela-dufflar"> Marianela Dufflar </option><option value="/autor/maribel-acosta-damas"> Maribel Acosta Damas </option><option value="/autor/mariela-castro-espin"> Mariela Castro Espín </option><option value="/autor/marina-menendez"> Marina Menéndez </option><option value="/autor/mario-benedetti"> Mario Benedetti </option><option value="/autor/mario-masvidal-saavedra"> Mario Masvidal Saavedra </option><option value="/autor/marisol-ruiz-soto"> Marisol Ruiz Soto </option><option value="/autor/marjorie-cohn"> Marjorie Cohn </option><option value="/autor/mark-weisbrot"> Mark Weisbrot </option><option value="/autor/marta-harnecker"> Marta Harnecker </option><option value="/autor/marta-o-carreras-rivery"> Marta O. Carreras Rivery </option><option value="/autor/marta-rojas"> Marta Rojas </option><option value="/autor/marta-valdes"> Marta Valdés </option><option value="/autor/martha-isabel-andres-roman"> Martha Isabel Andrés Román </option><option value="/autor/maurice-lemoine"> Maurice Lemoine </option><option value="/autor/mauricio-aira"> Mauricio Aira </option><option value="/autor/mauricio-rodriguez"> Mauricio Rodríguez </option><option value="/autor/max-castro"> Max Castro </option><option value="/autor/max-lesnik"> Max Lesnik </option><option value="/autor/maya-wind"> Maya Wind </option><option value="/autor/mayangdi-inzaulgarat"> Mayangdi Inzaulgarat </option><option value="/autor/mayra-montero"> Mayra Montero </option><option value="/autor/mempo-giardinelli"> Mempo Giardinelli </option><option value="/autor/mercedes-cardoso"> Mercedes Cardoso </option><option value="/autor/mercedes-de-armas-garcia-chachi"> Mercedes de Armas García (Chachi) </option><option value="/autor/michael-moore"> Michael Moore </option><option value="/autor/michel-chossudovsky"> Michel Chossudovsky </option><option value="/autor/michel-collon"> Michel Collon </option><option value="/autor/michel-contreras"> Michel Contreras </option><option value="/autor/miguel-angel-untoria-pedroso"> Miguel Ángel Untoria Pedroso </option><option value="/autor/miguel-barnet"> Miguel Barnet </option><option value="/autor/miguel-barreto"> Miguel Barreto </option><option value="/autor/miguel-bonasso"> Miguel Bonasso </option><option value="/autor/miguel-descoto"> Miguel D'Escoto </option><option value="/autor/miguel-del-padron"> Miguel del Padrón </option><option value="/autor/miguel-diaz-canel"> Miguel Díaz-Canel </option><option value="/autor/miguel-marin-bosch"> Miguel Marín Bosch </option><option value="/autor/miguel-rubiera-justiz"> Miguel Rubiera Justiz </option><option value="/autor/miguel-terry-valdespino"> Miguel Terry Valdespino </option><option value="/autor/miguel-torres"> Miguel Torres </option><option value="/autor/mikis-theodorakis"> Míkis Theodorákis </option><option value="/autor/mireya-baltra"> Mireya Baltra </option><option value="/autor/modesto-gutierrez-cabo"> Modesto Gutierrez Cabo </option><option value="/autor/moises-saab"> Moisés Saab </option><option value="/autor/monica-corrieri"> Mónica Corrieri </option><option value="/autor/monica-montes-medina"> Mónica Montes Medina </option><option value="/autor/monica-rivero"> Mónica Rivero </option><option value="/autor/monserrat-ponsa"> Monserrat Ponsa </option><option value="/autor/mumia-abu-jamal"> Mumia Abu-Jamal </option><option value="/autor/n-esteban-martinez"> N. Esteban Martínez </option><option value="/autor/nancy-morejon"> Nancy Morejón </option><option value="/autor/naomi-klein"> Naomi Klein </option><option value="/autor/nazanin-armanian"> Nazanín Armanian </option><option value="/autor/nelson-p-valdes"> Nelson P Valdés </option><option value="/autor/nelson-rodriguez-roque"> Nelson Rodríguez Roque </option><option value="/autor/nestor-del-prado"> Néstor del Prado </option><option value="/autor/nestor-garcia-iturbe"> Néstor Garía Iturbe </option><option value="/autor/nestor-kohan"> Néstor Kohan </option><option value="/autor/nicanor-leon-cotayo"> Nicanor León Cotayo </option><option value="/autor/nicholas-carr"> Nicholas Carr </option><option value="/autor/nicolas-guillen"> Nicolás Guillén </option><option value="/autor/nicolas-maduro-moros"> Nicolás Maduro Moros </option><option value="/autor/nidia-diaz"> Nidia Díaz </option><option value="/autor/nils-castro"> Nils Castro </option><option value="/autor/noam-chomsky"> Noam Chomsky </option><option value="/autor/noel-e-martinez"> Noel E. Martínez </option><option value="/autor/noel-martinez-martinez"> Noel Martínez Martínez </option><option value="/autor/noelio-tiuna"> Noelio Tiuna </option><option value="/autor/norge-espinosa-mendoza"> Norge Espinosa Mendoza </option><option value="/autor/norland-rosendo-gonzalez"> Norland Rosendo González </option><option value="/autor/octavio-fraga-guerra"> Octavio Fraga Guerra </option><option value="/autor/olga-fernandez-rios"> Olga Fernández Ríos </option><option value="/autor/olga-marta-carreras"> Olga Marta Carreras </option><option value="/autor/oliver-zamora-oria"> Oliver Zamora Oria </option><option value="/autor/omar-george-carpi"> Omar George Capri </option><option value="/autor/omar-gonzalez"> Omar González </option><option value="/autor/omar-olazabal-rodriguez"> Omar Olazábal Rodríguez </option><option value="/autor/omar-perez-salomon"> Omar Pérez Salomón </option><option value="/autor/omar-valino"> Omar Valiño </option><option value="/autor/omara-garcia-mederos"> Omara García Mederos </option><option value="/autor/oni-acosta-llerena"> Oni Acosta Llerena </option><option value="/autor/orestes-perez-perez"> Orestes Pérez Pérez </option><option value="/autor/oriol-de-la-cruz-atencio"> Oriol de la Cruz Atencio </option><option value="/autor/orlando-duran-hernandez"> Orlando Durán Hernández </option><option value="/autor/orlando-oramas-leon"> Orlando Oramas León </option><option value="/autor/oscar-alfonso-sosa"> Oscar Alfonso Sosa </option><option value="/autor/oscar-figueredo-reinaldo"> Oscar Figueredo Reinaldo </option><option value="/autor/oscar-lopez-rivera"> Oscar López Rivera </option><option value="/autor/oscar-sanchez-serra"> Oscar Sánchez Serra </option><option value="/autor/oscar-ugarteche"> Oscar Ugarteche </option><option value="/autor/osmany-torres"> Osmany Torres </option><option value="/autor/osvaldo-bayer"> Osvaldo Bayer </option><option value="/autor/osvaldo-doimeadios-2"> Osvaldo Doimeadios </option><option value="/autor/osvaldo-gutierrez-gomez"> Osvaldo Gutiérrez Gómez </option><option value="/autor/osvaldo-gutierrez-gomez-osval"> Osvaldo Gutiérrez Gómez (Osval) </option><option value="/autor/osvaldo-martinez"> Osvaldo Martínez </option><option value="/autor/otmaro-rodriguez"> Otmaro Rodríguez </option><option value="/autor/pablo-gentili"> Pablo Gentili </option><option value="/autor/pablo-gonzalez-casanova"> Pablo González Casanova </option><option value="/autor/pablo-milanes"> Pablo Milanés </option><option value="/autor/pablo-urbano"> Pablo Urbano </option><option value="/autor/paco-ignacio-taibo-ii"> Paco Ignacio Taibo II </option><option value="/autor/padre-michael-lapsley"> Padre Michael Lapsley </option><option value="/autor/palmiro-soria-saucedo"> Palmiro Soria Saucedo </option><option value="/autor/paola-cabrera-rodriguez"> Paola Cabrera Rodríguez </option><option value="/autor/papo-coss"> Papo Coss </option><option value="/autor/paquita-de-armas"> Paquita Armas Fonseca </option><option value="/autor/pascual-serrano"> Pascual Serrano </option><option value="/autor/pastor-batista"> Pastor Batista </option><option value="/autor/patricio-hugo-viveros"> Patricio Hugo Viveros </option><option value="/autor/patricio-montesinos"> Patricio Montesinos </option><option value="/autor/patricio-zamorano"> Patricio Zamorano </option><option value="/autor/patrick-cockburn"> Patrick Cockburn </option><option value="/autor/paul-craig-roberts"> Paul Craig Roberts </option><option value="/autor/paul-krugman"> Paul Krugman </option><option value="/autor/pedro-de-la-hoz"> Pedro de la Hoz </option><option value="/autor/pedro-miguel"> Pedro Miguel </option><option value="/autor/pedro-nunez-mosquera"> Pedro Núñez Mosquera </option><option value="/autor/pedro-pablo-rodriguez"> Pedro Pablo Rodríguez </option><option value="/autor/pedro-prada"> Pedro Prada </option><option value="/autor/pedro-urra-medina"> Pedro Urra Medina </option><option value="/autor/pepe-escobar"> Pepe Escobar </option><option value="/autor/percy-alvarado"> Percy Alvarado </option><option value="/autor/peter-kornbluh"> Peter Kornbluh </option><option value="/autor/peter-s-goodman"> Peter S. Goodman </option><option value="/autor/piero-gleijeses"> Piero Gleijeses </option><option value="/autor/rafael-arzuaga"> Rafael Arzuaga </option><option value="/autor/rafael-cancel-miranda"> Rafael Cancel Miranda </option><option value="/autor/rafael-correa"> Rafael Correa </option><option value="/autor/rafael-cruz-ramos"> Rafael Cruz Ramos </option><option value="/autor/rafael-de-la-osa"> Rafael de la Osa </option><option value="/autor/rafael-gonzalez-escalona"> Rafael González Escalona </option><option value="/autor/rafael-gonzalez-vazquez"> Rafael González Vázquez </option><option value="/autor/rafael-grillo"> Rafael Grillo </option><option value="/autor/rafael-hernandez"> Rafael Hernández </option><option value="/autor/rafael-rodriguez-cruz"> Rafael Rodríguez Cruz </option><option value="/autor/ramon-barreras-ferran"> Ramón Barreras Ferrán </option><option value="/autor/ramon-chao"> Ramón Chao </option><option value="/autor/ramon-espinosa"> Ramón Espinosa </option><option value="/autor/ramon-frontera-nieves"> Ramón Frontera Nieves </option><option value="/autor/ramon-guerra-diaz"> Ramón Guerra Díaz </option><option value="/autor/ramon-labanino-salazar"> Ramón Labañino Salazar </option><option value="/autor/ramon-sanchez-parodi-montoto"> Ramón Sánchez-Parodi Montoto </option><option value="/autor/randy-alonso"> Randy Alonso Falcón </option><option value="/autor/raul-alzaga-manresa"> Raúl Alzaga Manresa </option><option value="/autor/raul-antonio-capote"> Raúl Antonio Capote </option><option value="/autor/raul-castro-ruz"> Raúl Castro Ruz </option><option value="/autor/raul-fergo"> Raúl Fergo </option><option value="/autor/raul-garces"> Raul Garcés </option><option value="/autor/raul-gomez"> Raúl Gómez </option><option value="/autor/raul-nogues"> Raúl Nogués </option><option value="/autor/raul-pupo"> Raúl Pupo </option><option value="/autor/raul-roa-kouri"> Raúl Roa Kourí </option><option value="/autor/raul-rodriguez-la-o"> Raúl Rodríguez La O </option><option value="/autor/raynier-pellon-azopardo"> Raynier Pellón Azopardo </option><option value="/autor/rebeca-chavez"> Rebeca Chávez </option><option value="/autor/reinaldo-taladrid-herrero"> Reinaldo Taladrid Herrero </option><option value="/autor/remy-herrera"> Rémy Herrera </option><option value="/autor/renato-recio"> Renato Recio </option><option value="/autor/rene-gonzalez-barrios"> René González Barrios </option><option value="/autor/rene-gonzalez-sehwerert"> René González Sehwerert </option><option value="/autor/rene-navarro-arbelo"> René Navarro Arbelo </option><option value="/autor/rene-vazquez-diaz"> René Vázquez Díaz </option><option value="/autor/rev-raul-suarez"> Rev. Raúl Suárez </option><option value="/autor/reydel-reyes-torres"> Reydel Reyes Torres </option><option value="/autor/reinaldo-gonzalez"> Reynaldo González </option><option value="/autor/ricardo-alarcon-de-quesada"> Ricardo Alarcón de Quesada </option><option value="/autor/ricardo-arturo-salgado"> Ricardo Arturo Salgado </option><option value="/autor/ricardo-lopez-hevia"> Ricardo López Hevia </option><option value="/autor/ricardo-patino-aroca"> Ricardo Patiño Aroca </option><option value="/autor/ricardo-ronquillo"> Ricardo Ronquillo </option><option value="/autor/richard-stone"> Richard Stone </option><option value="/autor/rick-rozoff"> Rick Rozoff </option><option value="/autor/rigoberta-menchu"> Rigoberta Menchú </option><option value="/autor/robert-fisk"> Robert Fisk </option><option value="/autor/roberto-bardini"> Roberto Bardini </option><option value="/autor/roberto-cabrera-catasus"> Roberto Cabrera Catasús </option><option value="/autor/roberto-chile"> Roberto Chile </option><option value="/autor/roberto-fernandez-retamar"> Roberto Fernández Retamar </option><option value="/autor/roberto-garaycoa-martinez"> Roberto Garaycoa Martínez </option><option value="/autor/roberto-merino"> Roberto Meriño </option><option value="/autor/roberto-montoya"> Roberto Montoya </option><option value="/autor/roberto-morales-ojeda"> Roberto Morales Ojeda </option><option value="/autor/roberto-morejon-rodriguez"> Roberto Morejón Rodríguez </option><option value="/autor/roberto-regalado"> Roberto Regalado </option><option value="/autor/roberto-suarez"> Roberto Suárez </option><option value="/autor/roberto-verrier-castro"> Roberto Verrier Castro </option><option value="/autor/rodolfo-blanco-cue"> Rodolfo Blanco Cue </option><option value="/autor/rodolfo-reyes-rodriguez"> Rodolfo Reyes Rodríguez </option><option value="/autor/rodolfo-romero-reyes"> Rodolfo Romero Reyes </option><option value="/autor/rodrigo-malmierca"> Rodrigo Malmierca </option><option value="/autor/rogelio-polanco-fuentes"> Rogelio Polanco Fuentes </option><option value="/autor/roger-ricardo-luis"> Roger Ricardo Luis </option><option value="/autor/rolando-perez-betancourt"> Rolando Pérez Betancourt </option><option value="/autor/rolando-rodriguez"> Rolando Rodríguez </option><option value="/autor/rolando-segura"> Rolando Segura </option><option value="/autor/ronald-hernandez-torres"> Ronald Hernández Torres </option><option value="/autor/rootcd"> root@cd </option><option value="/autor/rosa-maria-de-lahaye-guerra"> Rosa María de Lahaye Guerra </option><option value="/autor/rosa-maria-fernandez"> Rosa María Fernández </option><option value="/autor/rosa-miriam-elizalde"> Rosa Miriam Elizalde </option><option value="/autor/rosa-regas"> Rosa Regás </option><option value="/autor/rosana-berjaga"> Rosana Berjaga </option><option value="/autor/rosario-murillo"> Rosario Murillo </option><option value="/autor/roy-chaderton-matos"> Roy Chaderton Matos </option><option value="/autor/ruben-abelenda"> Rubén Abelenda </option><option value="/autor/ruben-berrios-martinez"> Rubén Berríos Martínez </option><option value="/autor/ruben-sanchez-rivero"> Rubén Sánchez Rivero </option><option value="/autor/sacha-llorenti"> Sacha Llorenti </option><option value="/autor/salam-mousa-reyes"> Salam Mousa Reyes </option><option value="/autor/salim-lamrani"> Salim Lamrani </option><option value="/autor/salvador-capote"> Salvador Capote </option><option value="/autor/salvador-valdes-mesa"> Salvador Valdés Mesa </option><option value="/autor/sandra-russo"> Sandra Russo </option><option value="/autor/santiago-alba-rico"> Santiago Alba Rico </option><option value="/autor/santiago-odonnell"> Santiago O'Donnell </option><option value="/autor/santiago-perez-benitez"> Santiago Pérez Benítez </option><option value="/autor/sara-gonzalez"> Sara González </option><option value="/autor/sarah-stephens"> Sarah Stephens </option><option value="/autor/saul-landau"> Saul Landau </option><option value="/autor/senel-paz"> Senel Paz </option><option value="/autor/sergio-abel-reyes"> Sergio Abel Reyes </option><option value="/autor/sergio-alejandro-gomez"> Sergio Alejandro Gómez </option><option value="/autor/sergio-pena-martinez"> Sergio Peña Martínez </option><option value="/autor/seth-stephens-davidowitz"> Seth Stephens-Davidowitz </option><option value="/autor/silvia-mayra-gomez-farinas"> Silvia Mayra Gómez Fariñas </option><option value="/autor/silvio-rodriguez"> Silvio Rodríguez </option><option value="/autor/simon-wollers"> Simon Wollers </option><option value="/autor/sophie-perchelett"> Sophie Perchelett </option><option value="/autor/stella-calloni"> Stella Calloni </option><option value="/autor/subcomandante-marcos"> Subcomandante Marcos </option><option value="/autor/susana-tesoro"> Susana Tesoro </option><option value="/autor/talia-gonzalez"> Talía González </option><option value="/autor/tariq-ali"> Tariq Alí </option><option value="/autor/teodoro-renteria-arroyave"> Teodoro Rentería Arróyave </option><option value="/autor/teresa-melo"> Teresa Melo </option><option value="/autor/thalia-fuentes-puebla"> Thalia Fuentes Puebla </option><option value="/autor/thalia-fung"> Thalía Fung </option><option value="/autor/theotonio-dos-santos"> Theotonio Dos Santos </option><option value="/autor/thierry-meyssan"> Thierry Meyssan </option><option value="/autor/thom-shanker"> Thom Shanker </option><option value="/autor/tim-anderson"> Tim Anderson </option><option value="/autor/tom-crumpacker"> Tom Crumpacker </option><option value="/autor/tom-engelhardt"> Tom Engelhardt </option><option value="/autor/tom-hayden"> Tom Hayden </option><option value="/autor/tomas-borge"> Tomás Borge </option><option value="/autor/tomas-roncero"> Tomás Roncero </option><option value="/autor/tony-pinelli"> Tony Pinelli </option><option value="/autor/tubal-paez"> Tubal Páez </option><option value="/autor/ulises-guilarte-de-nacimiento"> Ulises Guilarte De Nacimiento </option><option value="/autor/umberto-eco"> Umberto Eco </option><option value="/autor/veronica-moreira"> Verónica Moreira </option><option value="/autor/vicenc-navarro"> Vicenç Navarro </option><option value="/autor/vicente-feliu"> Vicente Feliú </option><option value="/autor/vicky-pelaez"> Vicky Peláez </option><option value="/autor/victor-casaus"> Víctor Casaus </option><option value="/autor/victor-flores-olea"> Víctor Flores Olea </option><option value="/autor/victor-gonzalez-martinez"> Victor Gonzalez Martínez </option><option value="/autor/victor-joaquin-ortega"> Víctor Joaquín Ortega </option><option value="/autor/vincenzo-basile"> Vincenzo Basile </option><option value="/autor/vladia-rubio"> Vladia Rubio </option><option value="/autor/wafy-ibrahim"> Wafy Ibrahim </option><option value="/autor/wayne-madsen"> Wayne Madsen </option><option value="/autor/wayne-smith"> Wayne Smith </option><option value="/autor/wilfredo-gonzalez-vidal"> Wilfredo González Vidal </option><option value="/autor/wilkie-delgado-correa"> Wilkie Delgado Correa </option><option value="/autor/william-blum"> William Blum </option><option value="/autor/william-fisher"> William Fisher </option><option value="/autor/william-m-leogrande"> William M. LeoGrande </option><option value="/autor/william-ospina"> William Ospina </option><option value="/autor/wilmer-rodriguez-fernandez"> Wilmer Rodríguez Fernández </option><option value="/autor/winston-orrillo"> Winston Orrillo </option><option value="/autor/yaciel-pena-de-la-pena"> Yaciel Peña de la Peña </option><option value="/autor/yadira-escobar"> Yadira Escobar </option><option value="/autor/yaifred-ron"> Yaifred Ron </option><option value="/autor/yailin-orta-rivera"> Yailin Orta Rivera </option><option value="/autor/yaima-puig-meneses"> Yaima Puig Meneses </option><option value="/autor/yanet-munoz-hernandez"> Yanet Muñoz Hernández </option><option value="/autor/yasel-toledo-garnache"> Yasel Toledo Garnache </option><option value="/autor/yeniel-cabrera-duardo"> Yeniel Cabrera Duardo </option><option value="/autor/yenisei-garcia"> Yenisei García </option><option value="/autor/yirsandy-rodriguez"> Yirsandy Rodríguez </option><option value="/autor/yoandry-avila-guerra"> Yoandry Avila Guerra </option><option value="/autor/yoel-cordovi-nunez"> Yoel Cordoví Núñez </option><option value="/autor/yoerky-sanchez-cuellar"> Yoerky Sánchez Cuéllar </option><option value="/autor/yosbel-bullain-viltres"> Yosbel Bullaín Viltres </option><option value="/autor/yosvani-montano"> Yosvani Montano </option><option value="/autor/yuliat-danay-acosta"> Yuliat Danay Acosta </option><option value="/autor/yuniasky-crespo-baquero"> Yuniasky Crespo Baquero </option><option value="/autor/yunier-javier-sifonte-diaz"> Yunier Javier Sifonte Díaz </option><option value="/autor/yusmary-romero-cruz"> Yusmary Romero Cruz </option><option value="/autor/yusuam-palacios-ortega"> Yusuam Palacios Ortega </option><option value="/autor/zuleica-roman"> Zuleica Romay </option> </select></aside>
<!-- sidebar cached 78270b5bdf040295b1d64341973f43aa --></div>
    </section>
    
    <section id="bottom">
    <section id="sidebar_top" class="generic">
    <aside id="rldtriple_widget" class="widget widget_conteiner widget_rldtriple_widget">
                    <section id="green">
                <header class="title_wrapper">
                    <span class="icon_class"></span>
                    <h2 class="widget-title seccion_title">
                        <a href="http://www.cubadebate.cu/autor/fidel-castro-ruz/" title="Reflexiones de Fidel">
                            Reflexiones de Fidel                        </a>
                    </h2>
                </header>
                <ul class="catlist" id="catlist-">
                                            <li class="encat-posts-item-1003647">
                            <a href="http://www.cubadebate.cu/especiales/2018/01/08/fidel-el-8-de-enero-de-1959-este-un-momento-decisivo-de-nuestra-historia/" title="Fidel el 8 de enero de 1959: &#8220;&#8230; este un momento decisivo de nuestra historia&#8221; (+Video)">Fidel el 8 de enero de 1959: &#8220;&#8230; este un momento decisivo de nuestra historia&#8221; (+Video)</a>
                            <div class="meta">
    <time datetime="2018-01-08 02:32:00">8 enero 2018</time>
        <span class="separator">|</span>
        <span class="comment_count">
            <a href="http://www.cubadebate.cu/especiales/2018/01/08/fidel-el-8-de-enero-de-1959-este-un-momento-decisivo-de-nuestra-historia/#respond" title="Comentarios en Fidel el 8 de enero de 1959: &#8220;&#8230; este un momento decisivo de nuestra historia&#8221; (+Video)" >17</a>
        </span>
            </div>                        </li>
                                            <li class="encat-posts-item-971683">
                            <a href="http://www.cubadebate.cu/especiales/2017/10/18/fidel-hace-50-anos-queremos-que-sean-como-el-che/" title="Fidel hace 50 años: &#8220;¡Queremos que sean como el Che!&#8221; (+ Videos)">Fidel hace 50 años: &#8220;¡Queremos que sean como el Che!&#8221; (+ Videos)</a>
                            <div class="meta">
    <time datetime="2017-10-18 07:33:00">18 octubre 2017</time>
        <span class="separator">|</span>
        <span class="comment_count">
            <a href="http://www.cubadebate.cu/especiales/2017/10/18/fidel-hace-50-anos-queremos-que-sean-como-el-che/#respond" title="Comentarios en Fidel hace 50 años: &#8220;¡Queremos que sean como el Che!&#8221; (+ Videos)" >11</a>
        </span>
            </div>                        </li>
                                            <li class="encat-posts-item-971275">
                            <a href="http://www.cubadebate.cu/especiales/2017/10/17/fidel-hace-20-anos-gracias-che-por-tu-historia-tu-vida-y-tu-ejemplo/" title="Fidel hace 20 años: &#8220;¡Gracias, Che, por tu historia, tu vida y tu ejemplo!&#8221; (`Videos)">Fidel hace 20 años: &#8220;¡Gracias, Che, por tu historia, tu vida y tu ejemplo!&#8221; (`Videos)</a>
                            <div class="meta">
    <time datetime="2017-10-17 01:42:00">17 octubre 2017</time>
        <span class="separator">|</span>
        <span class="comment_count">
            <a href="http://www.cubadebate.cu/especiales/2017/10/17/fidel-hace-20-anos-gracias-che-por-tu-historia-tu-vida-y-tu-ejemplo/#respond" title="Comentarios en Fidel hace 20 años: &#8220;¡Gracias, Che, por tu historia, tu vida y tu ejemplo!&#8221; (`Videos)" >10</a>
        </span>
            </div>                        </li>
                                            <li class="encat-posts-item-892031">
                            <a href="http://www.cubadebate.cu/especiales/2017/04/13/la-llamada-de-fidel-y-chavez-hace-15-anos-tras-la-derrota-del-golpe-en-venezuela-video-y-pdf/" title="La llamada de Fidel y Chavez hace 15 años tras la derrota del Golpe en Venezuela (+ Video y PDF)">La llamada de Fidel y Chavez hace 15 años tras la derrota del Golpe en Venezuela (+ Video y PDF)</a>
                            <div class="meta">
    <time datetime="2017-04-13 14:38:00">13 abril 2017</time>
        <span class="separator">|</span>
        <span class="comment_count">
            <a href="http://www.cubadebate.cu/especiales/2017/04/13/la-llamada-de-fidel-y-chavez-hace-15-anos-tras-la-derrota-del-golpe-en-venezuela-video-y-pdf/#respond" title="Comentarios en La llamada de Fidel y Chavez hace 15 años tras la derrota del Golpe en Venezuela (+ Video y PDF)" >15</a>
        </span>
            </div>                        </li>
                                            <li class="encat-posts-item-859547">
                            <a href="http://www.cubadebate.cu/noticias/2017/01/27/intervencion-inedita-de-fidel-en-1988-durante-el-primer-consejo-nacional-de-la-ahs/" title="Intervención inédita de Fidel en 1988 durante el primer Consejo Nacional de la AHS">Intervención inédita de Fidel en 1988 durante el primer Consejo Nacional de la AHS</a>
                            <div class="meta">
    <time datetime="2017-01-27 00:01:00">27 enero 2017</time>
        <span class="separator">|</span>
        <span class="comment_count">
            <a href="http://www.cubadebate.cu/noticias/2017/01/27/intervencion-inedita-de-fidel-en-1988-durante-el-primer-consejo-nacional-de-la-ahs/#respond" title="Comentarios en Intervención inédita de Fidel en 1988 durante el primer Consejo Nacional de la AHS" >38</a>
        </span>
            </div>                        </li>
                                    </ul>
                <div class="read-more"><a title="Reflexiones de Fidel" href="http://www.cubadebate.cu/autor/fidel-castro-ruz/">Ver más</a></div>
            </section>
                    <section id="blue">
                <header class="title_wrapper">
                    <span class="icon_class"></span>
                    <h2 class="widget-title seccion_title">
                        <a href="http://www.cubadebate.cu/autor/raul-castro-ruz/" title="Discursos de Raúl">
                            Discursos de Raúl                        </a>
                    </h2>
                </header>
                <ul class="catlist" id="catlist-">
                                            <li class="encat-posts-item-1028927">
                            <a href="http://www.cubadebate.cu/especiales/2018/03/05/raul-castro-proclamamos-el-invariable-respaldo-a-la-revolucion-bolivariana/" title="Raúl Castro: Proclamamos el invariable respaldo a la Revolución Bolivariana">Raúl Castro: Proclamamos el invariable respaldo a la Revolución Bolivariana</a>
                            <div class="meta">
    <time datetime="2018-03-05 23:23:00">5 marzo 2018</time>
        <span class="separator">|</span>
        <span class="comment_count">
            <a href="http://www.cubadebate.cu/especiales/2018/03/05/raul-castro-proclamamos-el-invariable-respaldo-a-la-revolucion-bolivariana/#respond" title="Comentarios en Raúl Castro: Proclamamos el invariable respaldo a la Revolución Bolivariana" >25</a>
        </span>
            </div>                        </li>
                                            <li class="encat-posts-item-1026011">
                            <a href="http://www.cubadebate.cu/opinion/2018/02/25/raul-castro-un-homenaje-a-tres-mambises-de-estos-tiempos/" title="Raúl Castro: Un homenaje a tres mambises de estos tiempos">Raúl Castro: Un homenaje a tres mambises de estos tiempos</a>
                            <div class="meta">
    <time datetime="2018-02-25 16:25:00">25 febrero 2018</time>
        <span class="separator">|</span>
        <span class="comment_count">
            <a href="http://www.cubadebate.cu/opinion/2018/02/25/raul-castro-un-homenaje-a-tres-mambises-de-estos-tiempos/#respond" title="Comentarios en Raúl Castro: Un homenaje a tres mambises de estos tiempos" >24</a>
        </span>
            </div>                        </li>
                                            <li class="encat-posts-item-999279">
                            <a href="http://www.cubadebate.cu/especiales/2017/12/22/raul-en-la-asamblea-nacional-aqui-estamos-y-estaremos-libres-soberanos-e-independientes/" title="Raúl en la Asamblea Nacional: &#8220;La Revolución Cubana ha resistido los embates de 11 administraciones de EEUU&#8221;">Raúl en la Asamblea Nacional: &#8220;La Revolución Cubana ha resistido los embates de 11 administraciones de EEUU&#8221;</a>
                            <div class="meta">
    <time datetime="2017-12-22 00:50:00">22 diciembre 2017</time>
        <span class="separator">|</span>
        <span class="comment_count">
            <a href="http://www.cubadebate.cu/especiales/2017/12/22/raul-en-la-asamblea-nacional-aqui-estamos-y-estaremos-libres-soberanos-e-independientes/#respond" title="Comentarios en Raúl en la Asamblea Nacional: &#8220;La Revolución Cubana ha resistido los embates de 11 administraciones de EEUU&#8221;" >50</a>
        </span>
            </div>                        </li>
                                            <li class="encat-posts-item-994333">
                            <a href="http://www.cubadebate.cu/especiales/2017/12/09/raul-en-asamblea-de-oeco-el-caribe-siempre-podra-contar-con-cuba/" title="Raúl en Asamblea de OECO: &#8220;El Caribe siempre podrá contar con Cuba&#8221;">Raúl en Asamblea de OECO: &#8220;El Caribe siempre podrá contar con Cuba&#8221;</a>
                            <div class="meta">
    <time datetime="2017-12-09 23:27:00">9 diciembre 2017</time>
        <span class="separator">|</span>
        <span class="comment_count">
            <a href="http://www.cubadebate.cu/especiales/2017/12/09/raul-en-asamblea-de-oeco-el-caribe-siempre-podra-contar-con-cuba/#respond" title="Comentarios en Raúl en Asamblea de OECO: &#8220;El Caribe siempre podrá contar con Cuba&#8221;" >1</a>
        </span>
            </div>                        </li>
                                            <li class="encat-posts-item-993947">
                            <a href="http://www.cubadebate.cu/especiales/2017/12/08/raul-castro-un-caribe-cada-vez-mas-prospero-equitativo-seguro-sostenible-y-unido-es-posible/" title="Raúl Castro: Un Caribe cada vez más próspero, equitativo, seguro, sostenible y unido es posible">Raúl Castro: Un Caribe cada vez más próspero, equitativo, seguro, sostenible y unido es posible</a>
                            <div class="meta">
    <time datetime="2017-12-08 17:34:00">8 diciembre 2017</time>
        <span class="separator">|</span>
        <span class="comment_count">
            <a href="http://www.cubadebate.cu/especiales/2017/12/08/raul-castro-un-caribe-cada-vez-mas-prospero-equitativo-seguro-sostenible-y-unido-es-posible/#respond" title="Comentarios en Raúl Castro: Un Caribe cada vez más próspero, equitativo, seguro, sostenible y unido es posible" >7</a>
        </span>
            </div>                        </li>
                                    </ul>
                <div class="read-more"><a title="Discursos de Raúl" href="http://www.cubadebate.cu/autor/raul-castro-ruz/">Ver más</a></div>
            </section>
                <section id='red'>
            <header class="title_wrapper">
                <span class="icon_class"></span>
                <h2 class="widget-title seccion_title">
                    <a href="http://www.cubadebate.cu/serie/chavez-vivira-por-siempre/" title="Chávez por Siempre">
                        Chávez por Siempre                    </a>
                </h2>
            </header>
            <ul>
                                    <li class="entax-posts-item-100103}">
                        <a href="http://www.cubadebate.cu/serie/fotos-chavez/">Fotos</a>
                    </li>
                                    <li class="entax-posts-item-100099}">
                        <a href="http://www.cubadebate.cu/serie/lineas-chavez/">Las líneas de Chávez</a>
                    </li>
                                    <li class="entax-posts-item-100105}">
                        <a href="http://www.cubadebate.cu/serie/libros-chavez/">Libros</a>
                    </li>
                                    <li class="entax-posts-item-100101}">
                        <a href="http://www.cubadebate.cu/serie/notas-retaguardia/">Notas de Retaguardia</a>
                    </li>
                            </ul>
        </section>
        <section id="serie">
            <header class="title_wrapper">
                <span class="icon_class ' . $icon . '"></span>
                <h2 class="widget-title seccion_title">Series</h2>
            </header>
            <ul>
                                    <li class="entax-posts-item-106897">
                        <a href="http://www.cubadebate.cu/serie/la-columna-del-che/">La columna del Che</a>
                    </li>
                                    <li class="entax-posts-item-344">
                        <a href="http://www.cubadebate.cu/serie/historia-de-cuba/">Historia de Cuba</a>
                    </li>
                                    <li class="entax-posts-item-325">
                        <a href="http://www.cubadebate.cu/serie/democracia-en-cuba/">Democracia en Cuba</a>
                    </li>
                                    <li class="entax-posts-item-340">
                        <a href="http://www.cubadebate.cu/serie/cuba-deporte/">Cuba en el Deporte</a>
                    </li>
                            </ul>
        </section>
    </aside>
</section>    <section id="bottom-sidebars">
    <div id="sidebar_left">
    <aside id="col_tax-3" class="widget widget_conteiner col_tax"><header class="title_wrapper"><span class="icon_class canal-usb"></span><h2 class="widget-title seccion_title"><a href="http://www.cubadebate.cu/columna/canal-usb/" title="Ver art&iacute;culos de Canal USB">Canal USB</a></h2></header><ul><li class="coltax-posts-item-1033235"><a href="http://www.cubadebate.cu/noticias/2018/03/16/invita-la-uci-a-participar-en-pena-tecnologica-y-hackathon/">Invita la UCI a participar en Peña Tecnológica y Hackathon</a><div class="meta">16 marzo 2018<span class="separator">|</span><span class="comment_count"><a href="http://www.cubadebate.cu/noticias/2018/03/16/invita-la-uci-a-participar-en-pena-tecnologica-y-hackathon/#respond" title="Comentarios en Invita la UCI a participar en Peña Tecnológica y Hackathon" >6</a></span></li><li class="coltax-posts-item-1030043"><a href="http://www.cubadebate.cu/noticias/2018/03/09/la-apk-de-canal-habana-tuitsaojo-y-el-regreso-de-black-ops/">La apk de Canal Habana, #TuitsAOjo y el regreso de Black Ops</a><div class="meta">09 marzo 2018<span class="separator">|</span><span class="comment_count"><a href="http://www.cubadebate.cu/noticias/2018/03/09/la-apk-de-canal-habana-tuitsaojo-y-el-regreso-de-black-ops/#respond" title="Comentarios en La apk de Canal Habana, #TuitsAOjo y el regreso de Black Ops" >4</a></span></li><li class="coltax-posts-item-1028969"><a href="http://www.cubadebate.cu/especiales/2018/03/06/one-piece-mas-de-20-anos-de-una-travesia-impagable/">One Piece: Más de 20 años de una travesía impagable</a><div class="meta">06 marzo 2018<span class="separator">|</span><span class="comment_count"><a href="http://www.cubadebate.cu/especiales/2018/03/06/one-piece-mas-de-20-anos-de-una-travesia-impagable/#respond" title="Comentarios en One Piece: Más de 20 años de una travesía impagable" >69</a></span></li><li class="coltax-posts-item-1027709"><a href="http://www.cubadebate.cu/noticias/2018/03/02/premios-tu-android-2018-la-subasta-nerd-cubana/">Premios Tu Android 2018, la subasta nerd cubana</a><div class="meta">02 marzo 2018<span class="separator">|</span><span class="comment_count"><a href="http://www.cubadebate.cu/noticias/2018/03/02/premios-tu-android-2018-la-subasta-nerd-cubana/#respond" title="Comentarios en Premios Tu Android 2018, la subasta nerd cubana" >44</a></span></li></ul><div class="read-more"><a title="Canal USB" href="http://www.cubadebate.cu/columna/canal-usb/">Ver más</a></div></aside><aside id="col_tax-4" class="widget widget_conteiner col_tax"><header class="title_wrapper"><span class="icon_class rafaga-340"></span><h2 class="widget-title seccion_title"><a href="http://www.cubadebate.cu/columna/rafaga-340/" title="Ver art&iacute;culos de Ráfaga 340">Ráfaga 340</a></h2></header><h3 class="author_tax"><a href="http://www.cubadebate.cu/autor/danier-ernesto-gonzalez/" title="Ver art&iacute;culos de Danier Ernesto González">Danier Ernesto González</a></h3><ul><li class="coltax-posts-item-1032007"><a href="http://www.cubadebate.cu/especiales/2018/03/13/el-frio-en-cuba-el-declive-de-la-nina-y-algunas-notas-del-mal-tiempo/">El frío en Cuba, el declive de La Niña y algunas notas del mal tiempo</a><div class="meta">13 marzo 2018<span class="separator">|</span><span class="comment_count"><a href="http://www.cubadebate.cu/especiales/2018/03/13/el-frio-en-cuba-el-declive-de-la-nina-y-algunas-notas-del-mal-tiempo/#respond" title="Comentarios en El frío en Cuba, el declive de La Niña y algunas notas del mal tiempo" >7</a></span></li><li class="coltax-posts-item-1028931"><a href="http://www.cubadebate.cu/especiales/2018/03/06/la-verdadera-historia-de-la-tormenta-del-siglo-fotos/">La verdadera historia de la “Tormenta del Siglo" (+ Fotos)</a><div class="meta">06 marzo 2018<span class="separator">|</span><span class="comment_count"><a href="http://www.cubadebate.cu/especiales/2018/03/06/la-verdadera-historia-de-la-tormenta-del-siglo-fotos/#respond" title="Comentarios en La verdadera historia de la “Tormenta del Siglo" (+ Fotos)" >94</a></span></li><li class="coltax-posts-item-1026173"><a href="http://www.cubadebate.cu/especiales/2018/02/27/el-rarisimo-huracan-de-marzo-de-1908/">El rarísimo huracán de marzo de 1908</a><div class="meta">27 febrero 2018<span class="separator">|</span><span class="comment_count"><a href="http://www.cubadebate.cu/especiales/2018/02/27/el-rarisimo-huracan-de-marzo-de-1908/#respond" title="Comentarios en El rarísimo huracán de marzo de 1908" >10</a></span></li><li class="coltax-posts-item-1023689"><a href="http://www.cubadebate.cu/especiales/2018/02/20/el-anticiclon-sigue-pegao-y-los-frentes-frios/">El Anticiclón sigue pega'o, ¿y los frentes fríos?</a><div class="meta">20 febrero 2018<span class="separator">|</span><span class="comment_count"><a href="http://www.cubadebate.cu/especiales/2018/02/20/el-anticiclon-sigue-pegao-y-los-frentes-frios/#respond" title="Comentarios en El Anticiclón sigue pega'o, ¿y los frentes fríos?" >18</a></span></li></ul><div class="read-more"><a title="Ráfaga 340" href="http://www.cubadebate.cu/columna/rafaga-340/">Ver más</a></div></aside>
<!-- sidebar cached 01e36df22cad0375bb694c6ddad351ec -->    </div>
    <div id="sidebar_center">
    <aside id="time_machine-2" class="widget widget_conteiner time_machine"><header class="title_wrapper"><span class="icon_class"></span><h2 class="widget-title seccion_title">Hoy en el Archivo</h2></header>    <ul>
                    <li class="">
                <div class="post_title"><a href="http://www.cubadebate.cu/opinion/2017/03/19/estados-unidos-presencia-una-confrontacion-letal-entre-dos-imperialismos/">La creciente oleada de militarismo estadounidense en el siglo XXI</a></div>
                <div class="meta">
                    19 marzo 2017                                            <span class="separator">|</span>
                        <span class="comment_count">
                        <a href="http://www.cubadebate.cu/opinion/2017/03/19/estados-unidos-presencia-una-confrontacion-letal-entre-dos-imperialismos/#respond" title="Comentarios en ' La creciente oleada de militarismo estadounidense en el siglo XXI" >7</a>
                        </span>
                                    </div>
            </li>
                        <li class="">
                <div class="post_title"><a href="http://www.cubadebate.cu/opinion/2016/03/19/guantanamo-y-la-base-la-cuba-que-recibira-hoy-a-obama/">Guantánamo y la Base: La Cuba que recibirá hoy a Obama</a></div>
                <div class="meta">
                    19 marzo 2016                                            <span class="separator">|</span>
                        <span class="comment_count">
                        <a href="http://www.cubadebate.cu/opinion/2016/03/19/guantanamo-y-la-base-la-cuba-que-recibira-hoy-a-obama/#respond" title="Comentarios en ' Guantánamo y la Base: La Cuba que recibirá hoy a Obama" >41</a>
                        </span>
                                    </div>
            </li>
                        <li class="">
                <div class="post_title"><a href="http://www.cubadebate.cu/opinion/2016/03/19/una-guerra-del-siglo-xxi/">Una guerra del siglo XXI</a></div>
                <div class="meta">
                    19 marzo 2016                                            <span class="separator">|</span>
                        <span class="comment_count">
                        <a href="http://www.cubadebate.cu/opinion/2016/03/19/una-guerra-del-siglo-xxi/#respond" title="Comentarios en ' Una guerra del siglo XXI" >19</a>
                        </span>
                                    </div>
            </li>
                        <li class="">
                <div class="post_title"><a href="http://www.cubadebate.cu/opinion/2016/03/19/adoramos-a-tu-mama/">Adoramos a tu mamá</a></div>
                <div class="meta">
                    19 marzo 2016                                            <span class="separator">|</span>
                        <span class="comment_count">
                        <a href="http://www.cubadebate.cu/opinion/2016/03/19/adoramos-a-tu-mama/#respond" title="Comentarios en ' Adoramos a tu mamá" >103</a>
                        </span>
                                    </div>
            </li>
                        <li class="">
                <div class="post_title"><a href="http://www.cubadebate.cu/opinion/2016/03/19/a-proposito-de-la-visita-de-obama-sociedad-civil-y-derechos-humanos-en-cuba-ii-y-final/">A propósito de la  visita de Obama: Sociedad Civil y derechos humanos en Cuba (II y final)</a></div>
                <div class="meta">
                    19 marzo 2016                                            <span class="separator">|</span>
                        <span class="comment_count">
                        <a href="http://www.cubadebate.cu/opinion/2016/03/19/a-proposito-de-la-visita-de-obama-sociedad-civil-y-derechos-humanos-en-cuba-ii-y-final/#respond" title="Comentarios en ' A propósito de la  visita de Obama: Sociedad Civil y derechos humanos en Cuba (II y final)" >10</a>
                        </span>
                                    </div>
            </li>
                        <li class="">
                <div class="post_title"><a href="http://www.cubadebate.cu/opinion/2015/03/19/strike-3-fabula-de-la-sardina-y-el-escualo/">Strike 3: Fábula de la sardina y el escualo</a></div>
                <div class="meta">
                    19 marzo 2015                                            <span class="separator">|</span>
                        <span class="comment_count">
                        <a href="http://www.cubadebate.cu/opinion/2015/03/19/strike-3-fabula-de-la-sardina-y-el-escualo/#respond" title="Comentarios en ' Strike 3: Fábula de la sardina y el escualo" >165</a>
                        </span>
                                    </div>
            </li>
                        <li class="">
                <div class="post_title"><a href="http://www.cubadebate.cu/opinion/2015/03/19/con-tres-que-se-quieran-2/">Con tres que se quieran</a></div>
                <div class="meta">
                    19 marzo 2015                                            <span class="separator">|</span>
                        <span class="comment_count">
                        <a href="http://www.cubadebate.cu/opinion/2015/03/19/con-tres-que-se-quieran-2/#respond" title="Comentarios en ' Con tres que se quieran" >97</a>
                        </span>
                                    </div>
            </li>
                        <li class="">
                <div class="post_title"><a href="http://www.cubadebate.cu/opinion/2015/03/19/petimetres-contra-jose-marti/">¿Petimetres contra José Martí?</a></div>
                <div class="meta">
                    19 marzo 2015                                            <span class="separator">|</span>
                        <span class="comment_count">
                        <a href="http://www.cubadebate.cu/opinion/2015/03/19/petimetres-contra-jose-marti/#respond" title="Comentarios en ' ¿Petimetres contra José Martí?" >23</a>
                        </span>
                                    </div>
            </li>
                        <li class="">
                <div class="post_title"><a href="http://www.cubadebate.cu/opinion/2015/03/19/venezuela-obama-topa-con-pared/">Venezuela: Obama topa con la pared</a></div>
                <div class="meta">
                    19 marzo 2015                                            <span class="separator">|</span>
                        <span class="comment_count">
                        <a href="http://www.cubadebate.cu/opinion/2015/03/19/venezuela-obama-topa-con-pared/#respond" title="Comentarios en ' Venezuela: Obama topa con la pared" >9</a>
                        </span>
                                    </div>
            </li>
                        <li class=" last_li_item">
                <div class="post_title"><a href="http://www.cubadebate.cu/opinion/2015/03/19/con-tony-en-el-submarino-amarillo/">Con Tony en el Submarino Amarillo</a></div>
                <div class="meta">
                    19 marzo 2015                                            <span class="separator">|</span>
                        <span class="comment_count">
                        <a href="http://www.cubadebate.cu/opinion/2015/03/19/con-tony-en-el-submarino-amarillo/#respond" title="Comentarios en ' Con Tony en el Submarino Amarillo" >19</a>
                        </span>
                                    </div>
            </li>
                </ul>
    </aside>
<!-- sidebar cached 152355a64ebe1741592df4451819504d -->    </div>
    <div id="sidebar_right">
        <aside id="subscribe2" class="widget widget_conteiner s2class_widget_subscribe2widget"><a name="subscribe2"></a><header class="title_wrapper"><span class="icon_class"></span><h2 class="widget-title seccion_title">Suscribirse</h2></header><div class="search2"><form method="post" action="?subscribe=active#subscribe2"><p>Para registrarse o modificar el servicio de envío por correo electrónico de mensajes desde el sitio de Cubadebate, escriba su dirección en el recuadro que aparece a continuación y complete las informaciones que siguen.<br /><br />
		<input type="text" style="width: 99%;" name="email" value="Dirección de correo..." size="20" onfocus="if (this.value == 'Dirección de correo...') {this.value = '';}" onblur="if (this.value == '') {this.value = 'Dirección de correo...';}" /></p>
		<p><strong>Tr&aacute;mite</strong><br/><input type="radio" checked="checked" name="subscribe" value="1" />&nbsp;Suscribirse&nbsp;(comenzar a recibir correos)<br/><input type="radio" name="subscribe" value="2" /> Anular suscripci&oacute;n (dejar de recibir correos)</p>
		<p><strong>Contenido</strong><br/><input type="checkbox" checked="checked" name="subscribecat[]" value="1" /> Resumen diario de los art&iacute;culos publicados </p>
		<p><input type="submit" name="save" value="Enviar" /></p>
		</form>
</div></aside><aside id="custom_html-6" class="widget_text widget widget_conteiner widget_custom_html"><div class="textwidget custom-html-widget"><div id="fb-root"></div>
<script>(function(d, s, id) {
  var js, fjs = d.getElementsByTagName(s)[0];
  if (d.getElementById(id)) return;
  js = d.createElement(s); js.id = id;
  js.async=true; js.src = 'https://connect.facebook.net/es_LA/sdk.js#xfbml=1&version=v2.12&appId=569940219797389&autoLogAppEvents=1';
  fjs.parentNode.insertBefore(js, fjs);
}(document, 'script', 'facebook-jssdk'));</script>
<div class="fb-page" data-href="https://www.facebook.com/cubadebate/" data-small-header="false" data-adapt-container-width="true" data-hide-cover="false" data-show-facepile="true"><blockquote cite="https://www.facebook.com/cubadebate/" class="fb-xfbml-parse-ignore"><a href="https://www.facebook.com/cubadebate/">Cubadebate</a></blockquote></div></div></aside>    </div>
</section>
<section id="sidebar_bottom" class="generic">
    <aside id="youchannel-447272992" class="widget widget_conteiner youchannel"><div><header class="title_wrapper"><span class="icon_class"></span><h2 class="widget-title seccion_title">Desde Youtube</h2></header><div class='iframe-container'><div id="youchannel_inner"><a href="https://www.youtube.com/watch?v=LgEynqoFZUU" target="_blank" title="Ver en YouTube.com"><img width="100%" src="http://media.cubadebate.cu/wp-content/gallery/youtubechannel/default.jpg?v=LgEynqoFZUU" title="Ver en YouTube.com"></a></div></div></div>
			<script>
				
			  var tag = document.createElement('script');
			  tag.src = "https://www.youtube.com/player_api";
			  tag.defer = "defer";
			  var firstScriptTag = document.getElementsByTagName('script')[0];
			  firstScriptTag.parentNode.insertBefore(tag, firstScriptTag);

			  var player;
			  function onYouTubePlayerAPIReady() {
				jQuery('#youchannel_inner a').remove();
				player = new YT.Player('youchannel_inner', {
				  height: '100%',
				  width: '100%',
				  videoId: 'LgEynqoFZUU',
				  autoplay: 0,
				  playerVars: {rel: 0}
				});
			  }
			</script>
		</aside><aside id="ngg-images-2" class="widget widget_conteiner ngg_images">
<div class="hslice" id="ngg-webslice" >
<header class="entry-title title_wrapper"><span class="entry-title icon_class"></span><h2 class="entry-title widget-title seccion_title">Galería de Fotos</h2></header>
<div class="ngg-widget entry-content">
<a href="http://media.cubadebate.cu/wp-content/gallery/abierto-al-publico-el-capitolio-nacional-de-cuba/capitolio-cuba-4.jpg" title="Sobresale en el Capitolio la majestuosidad del Salón de los Pasos Perdidos. Foto: Irene Pérez/ Cubadebate." class="fancybox" rel="ngg-images-2"><img src="http://media.cubadebate.cu/wp-content/gallery/abierto-al-publico-el-capitolio-nacional-de-cuba/thumbs/thumbs_capitolio-cuba-4.jpg" width="157" height="117" title="capitolio-cuba-4" alt="capitolio-cuba-4" /></a>
<a href="http://media.cubadebate.cu/wp-content/gallery/abierto-al-publico-el-capitolio-nacional-de-cuba/capitolio-cuba-2.jpg" title="Este 1 de marzo, en horas de la mañana, se ha abierto al público el primer recorrido interior en el Capitolio de La Habana. Foto: Irene Pérez/ Cubadebate." class="fancybox" rel="ngg-images-2"><img src="http://media.cubadebate.cu/wp-content/gallery/abierto-al-publico-el-capitolio-nacional-de-cuba/thumbs/thumbs_capitolio-cuba-2.jpg" width="157" height="117" title="capitolio-cuba-2" alt="capitolio-cuba-2" /></a>
<a href="http://media.cubadebate.cu/wp-content/gallery/abierto-al-publico-el-capitolio-nacional-de-cuba/capitolio-cuba-14.jpg" title="Las visitas públicas serán siempre guiadas y estarán organizadas en grupos de 15 personas. Foto: Irene Pérez/ Cubadebate." class="fancybox" rel="ngg-images-2"><img src="http://media.cubadebate.cu/wp-content/gallery/abierto-al-publico-el-capitolio-nacional-de-cuba/thumbs/thumbs_capitolio-cuba-14.jpg" width="157" height="117" title="capitolio-cuba-14" alt="capitolio-cuba-14" /></a>
<a href="http://media.cubadebate.cu/wp-content/gallery/abierto-al-publico-el-capitolio-nacional-de-cuba/capitolio-cuba-5.jpg" title="La escultura La República, de 15 metros de alto es considerada la tercera mayor entre las que se encuentran bajo techo en el mundo. Foto: Irene Pérez/ Cubadebate." class="fancybox" rel="ngg-images-2"><img src="http://media.cubadebate.cu/wp-content/gallery/abierto-al-publico-el-capitolio-nacional-de-cuba/thumbs/thumbs_capitolio-cuba-5.jpg" width="157" height="117" title="capitolio-cuba-5" alt="capitolio-cuba-5" /></a>
<a href="http://media.cubadebate.cu/wp-content/gallery/abierto-al-publico-el-capitolio-nacional-de-cuba/capitolio-cuba-8.jpg" title="Cripta que guarda los restos del mambí desconocido. Foto: Irene Pérez/ Cubadebate." class="fancybox" rel="ngg-images-2"><img src="http://media.cubadebate.cu/wp-content/gallery/abierto-al-publico-el-capitolio-nacional-de-cuba/thumbs/thumbs_capitolio-cuba-8.jpg" width="157" height="117" title="capitolio-cuba-8" alt="capitolio-cuba-8" /></a>
<a href="http://media.cubadebate.cu/wp-content/gallery/abierto-al-publico-el-capitolio-nacional-de-cuba/capitolio-cuba-7.jpg" title="En la cripta que guarda los restos del mambí desconocido se pueden apreciar las banderas de algunas naciones del continente americano. Foto: Irene Pérez/ Cubadebate." class="fancybox" rel="ngg-images-2"><img src="http://media.cubadebate.cu/wp-content/gallery/abierto-al-publico-el-capitolio-nacional-de-cuba/thumbs/thumbs_capitolio-cuba-7.jpg" width="157" height="117" title="capitolio-cuba-7" alt="capitolio-cuba-7" /></a>
<a href="http://media.cubadebate.cu/wp-content/gallery/abierto-al-publico-el-capitolio-nacional-de-cuba/capitolio-cuba-10.jpg" title="El Capitolio es sede de la Asamblea Nacional del Poder Popular. Foto: Irene Pérez/ Cubadebate." class="fancybox" rel="ngg-images-2"><img src="http://media.cubadebate.cu/wp-content/gallery/abierto-al-publico-el-capitolio-nacional-de-cuba/thumbs/thumbs_capitolio-cuba-10.jpg" width="157" height="117" title="capitolio-cuba-10" alt="capitolio-cuba-10" /></a>
<a href="http://media.cubadebate.cu/wp-content/gallery/abierto-al-publico-el-capitolio-nacional-de-cuba/capitolio-cuba-13.jpg" title="El Hemiciclo del Capitolio de La Habana. Foto: Irene Pérez/ Cubadebate." class="fancybox" rel="ngg-images-2"><img src="http://media.cubadebate.cu/wp-content/gallery/abierto-al-publico-el-capitolio-nacional-de-cuba/thumbs/thumbs_capitolio-cuba-13.jpg" width="157" height="117" title="capitolio-cuba-13" alt="capitolio-cuba-13" /></a>
<a href="http://media.cubadebate.cu/wp-content/gallery/abierto-al-publico-el-capitolio-nacional-de-cuba/capitolio-cuba-12.jpg" title="Detalle de una luminaria. Foto: Irene Pérez/ Cubadebate." class="fancybox" rel="ngg-images-2"><img src="http://media.cubadebate.cu/wp-content/gallery/abierto-al-publico-el-capitolio-nacional-de-cuba/thumbs/thumbs_capitolio-cuba-12.jpg" width="157" height="117" title="capitolio-cuba-12" alt="capitolio-cuba-12" /></a>
</div>
</div>
</aside>
<!-- sidebar cached 7987e245657b764d337c617d7295f94a --></section>
    </section>
          
</section>
<section id="front-popular" class="generic">
    <!--<div id="footer-top" class="generic">-->
        <aside id="estadisticasconsumo-2" class="widget widget_conteiner widget_estadisticasconsumo"><section class="estadistica_ga"><header class="title_wrapper"><span class="icon_class"></span><h2 class="widget-title seccion_title" title="Más Leidas en los últimos 30 días"><a href="http://www.cubadebate.cu/internet/mas-leidas/" title="Más leídas">Más Leídas</a></h2></header><div class="estadisticas_grupo"><ul><li class="nombre_entrada"><a href="http://www.cubadebate.cu/noticias/2018/03/16/laura-pausini-cantara-en-cuba-invitada-por-gente-de-zona-video/">Laura Pausini desea ofrecer concierto en Cuba (+ Video)</a><div class="meta">16 marzo 2018<span class="separator">|</span><span class="comment_count"><a href="http://www.cubadebate.cu/noticias/2018/03/16/laura-pausini-cantara-en-cuba-invitada-por-gente-de-zona-video/#respond" title="Comentarios en Laura Pausini desea ofrecer concierto en Cuba (+ Video)" >128</a></span></div></li><li class="nombre_entrada"><a href="http://www.cubadebate.cu/noticias/2018/03/07/fracaso-provocacion-anticubana/">Fracasó provocación anticubana</a><div class="meta">07 marzo 2018<span class="separator">|</span><span class="comment_count"><a href="http://www.cubadebate.cu/noticias/2018/03/07/fracaso-provocacion-anticubana/#respond" title="Comentarios en Fracasó provocación anticubana" >212</a></span></div></li><li class="nombre_entrada"><a href="http://www.cubadebate.cu/opinion/2018/03/02/lo-que-le-gusta-a-los-menores/">Lo que le gusta a los ¿menores?</a><div class="meta">02 marzo 2018<span class="separator">|</span><span class="comment_count"><a href="http://www.cubadebate.cu/opinion/2018/03/02/lo-que-le-gusta-a-los-menores/#respond" title="Comentarios en Lo que le gusta a los ¿menores?" >290</a></span></div></li><li class="nombre_entrada"><a href="http://www.cubadebate.cu/especiales/2018/02/27/el-rarisimo-huracan-de-marzo-de-1908/">El rarísimo huracán de marzo de 1908</a><div class="meta">27 febrero 2018<span class="separator">|</span><span class="comment_count"><a href="http://www.cubadebate.cu/especiales/2018/02/27/el-rarisimo-huracan-de-marzo-de-1908/#respond" title="Comentarios en El rarísimo huracán de marzo de 1908" >10</a></span></div></li><li class="nombre_entrada"><a href="http://www.cubadebate.cu/especiales/2018/02/26/corrupcion-peligro-que-nos-afecta-a-todos/">Corrupción: Peligro que nos afecta a todos</a><div class="meta">26 febrero 2018<span class="separator">|</span><span class="comment_count"><a href="http://www.cubadebate.cu/especiales/2018/02/26/corrupcion-peligro-que-nos-afecta-a-todos/#respond" title="Comentarios en Corrupción: Peligro que nos afecta a todos" >347</a></span></div></li><li class="nombre_entrada"><a href="http://www.cubadebate.cu/noticias/2018/02/19/muerte-de-sylvester-stallone-causa-revuelo-en-las-redes-sociales/">Falsa muerte de Sylvester Stallone causa revuelo en las redes</a><div class="meta">19 febrero 2018<span class="separator">|</span><span class="comment_count"><a href="http://www.cubadebate.cu/noticias/2018/02/19/muerte-de-sylvester-stallone-causa-revuelo-en-las-redes-sociales/#respond" title="Comentarios en Falsa muerte de Sylvester Stallone causa revuelo en las redes" >84</a></span></div></li><li class="nombre_entrada"><a href="http://www.cubadebate.cu/especiales/2018/02/18/como-es-posible-la-florida-celebra-feria-de-armas-tres-dias-despues-de-la-masacre-en-parkland/">¿Cómo es posible? La Florida celebra Feria de Armas tres días después de la masacre en Parkland</a><div class="meta">18 febrero 2018<span class="separator">|</span><span class="comment_count"><a href="http://www.cubadebate.cu/especiales/2018/02/18/como-es-posible-la-florida-celebra-feria-de-armas-tres-dias-despues-de-la-masacre-en-parkland/#respond" title="Comentarios en ¿Cómo es posible? La Florida celebra Feria de Armas tres días después de la masacre en Parkland" >77</a></span></div></li><li class="nombre_entrada"><a href="http://www.cubadebate.cu/especiales/2018/02/18/noam-chomsky-explica-que-esconden-las-ayudas-humanitarias/">Noam Chomsky explica qué esconden las &#8220;Ayudas Humanitarias”</a><div class="meta">18 febrero 2018<span class="separator">|</span><span class="comment_count"><a href="http://www.cubadebate.cu/especiales/2018/02/18/noam-chomsky-explica-que-esconden-las-ayudas-humanitarias/#respond" title="Comentarios en Noam Chomsky explica qué esconden las &#8220;Ayudas Humanitarias”" >12</a></span></div></li><li class="nombre_entrada"><a href="http://www.cubadebate.cu/especiales/2017/12/05/que-es-el-petro-la-nueva-criptomoneda-de-venezuela/">¿Qué es &#8220;El Petro&#8221;, la nueva criptomoneda de Venezuela?</a><div class="meta">05 diciembre 2017<span class="separator">|</span><span class="comment_count"><a href="http://www.cubadebate.cu/especiales/2017/12/05/que-es-el-petro-la-nueva-criptomoneda-de-venezuela/#respond" title="Comentarios en ¿Qué es &#8220;El Petro&#8221;, la nueva criptomoneda de Venezuela?" >68</a></span></div></li><li class="nombre_entrada"><a href="http://www.cubadebate.cu/noticias/2009/12/21/unicef-confirma-que-cuba-tiene-0-de-desnutricion-infantil/">UNICEF confirma que Cuba tiene cero por ciento de Desnutrición Infantil</a><div class="meta">21 diciembre 2009<span class="separator">|</span><span class="comment_count"><a href="http://www.cubadebate.cu/noticias/2009/12/21/unicef-confirma-que-cuba-tiene-0-de-desnutricion-infantil/#respond" title="Comentarios en UNICEF confirma que Cuba tiene cero por ciento de Desnutrición Infantil" >107</a></span></div></li></ul></div></section><section class="estadistica_addthis"><header class="title_wrapper"><span class="icon_class"></span><h2 class="widget-title seccion_title" title="Más Compartidas en los últimos 30 días"><a href="http://www.cubadebate.cu/internet/mas-compartidas/" title="Más compartidas">Más Compartidas</a></h2></header><div class="estadisticas_grupo"><ul><li class="nombre_entrada"><a href="http://www.cubadebate.cu/noticias/2018/03/16/laura-pausini-cantara-en-cuba-invitada-por-gente-de-zona-video/">Laura Pausini desea ofrecer concierto en Cuba (+ Video)</a><div class="meta">16 marzo 2018<span class="separator">|</span><span class="comment_count"><a href="http://www.cubadebate.cu/noticias/2018/03/16/laura-pausini-cantara-en-cuba-invitada-por-gente-de-zona-video/#respond" title="Comentarios en Laura Pausini desea ofrecer concierto en Cuba (+ Video)" >128</a></span></div></li><li class="nombre_entrada"><a href="http://www.cubadebate.cu/noticias/2018/03/12/elecciones-en-cuba-elegidos-los-605-diputados-a-la-asamblea-nacional/">Elecciones en Cuba: Elegidos 605 diputados a la Asamblea Nacional (+ Infografía y Video)</a><div class="meta">12 marzo 2018<span class="separator">|</span><span class="comment_count"><a href="http://www.cubadebate.cu/noticias/2018/03/12/elecciones-en-cuba-elegidos-los-605-diputados-a-la-asamblea-nacional/#respond" title="Comentarios en Elecciones en Cuba: Elegidos 605 diputados a la Asamblea Nacional (+ Infografía y Video)" >153</a></span></div></li><li class="nombre_entrada"><a href="http://www.cubadebate.cu/noticias/2018/03/12/unicef-cuba-es-campeon-en-derechos-de-la-ninez/">Unicef: Cuba es campeón en derechos de la niñez</a><div class="meta">12 marzo 2018<span class="separator">|</span><span class="comment_count"><a href="http://www.cubadebate.cu/noticias/2018/03/12/unicef-cuba-es-campeon-en-derechos-de-la-ninez/#respond" title="Comentarios en Unicef: Cuba es campeón en derechos de la niñez" >12</a></span></div></li><li class="nombre_entrada"><a href="http://www.cubadebate.cu/noticias/2018/03/08/minrex-abortada-provocacion-anticubana-en-visperas-de-las-elecciones/">Minrex: Abortada provocación anticubana en vísperas de las elecciones</a><div class="meta">08 marzo 2018<span class="separator">|</span><span class="comment_count"><a href="http://www.cubadebate.cu/noticias/2018/03/08/minrex-abortada-provocacion-anticubana-en-visperas-de-las-elecciones/#respond" title="Comentarios en Minrex: Abortada provocación anticubana en vísperas de las elecciones" >100</a></span></div></li><li class="nombre_entrada"><a href="http://www.cubadebate.cu/noticias/2018/03/07/fracaso-provocacion-anticubana/">Fracasó provocación anticubana</a><div class="meta">07 marzo 2018<span class="separator">|</span><span class="comment_count"><a href="http://www.cubadebate.cu/noticias/2018/03/07/fracaso-provocacion-anticubana/#respond" title="Comentarios en Fracasó provocación anticubana" >212</a></span></div></li><li class="nombre_entrada"><a href="http://www.cubadebate.cu/especiales/2018/02/26/corrupcion-peligro-que-nos-afecta-a-todos/">Corrupción: Peligro que nos afecta a todos</a><div class="meta">26 febrero 2018<span class="separator">|</span><span class="comment_count"><a href="http://www.cubadebate.cu/especiales/2018/02/26/corrupcion-peligro-que-nos-afecta-a-todos/#respond" title="Comentarios en Corrupción: Peligro que nos afecta a todos" >347</a></span></div></li><li class="nombre_entrada"><a href="http://www.cubadebate.cu/noticias/2018/02/21/elegida-varadero-entre-las-mejores-playas-del-mundo-por-travellerschoice-2018/">Elegida Varadero entre las mejores playas del mundo por Travellers´Choice 2018</a><div class="meta">21 febrero 2018<span class="separator">|</span><span class="comment_count"><a href="http://www.cubadebate.cu/noticias/2018/02/21/elegida-varadero-entre-las-mejores-playas-del-mundo-por-travellerschoice-2018/#respond" title="Comentarios en Elegida Varadero entre las mejores playas del mundo por Travellers´Choice 2018" >89</a></span></div></li><li class="nombre_entrada"><a href="http://www.cubadebate.cu/noticias/2018/02/19/muerte-de-sylvester-stallone-causa-revuelo-en-las-redes-sociales/">Falsa muerte de Sylvester Stallone causa revuelo en las redes</a><div class="meta">19 febrero 2018<span class="separator">|</span><span class="comment_count"><a href="http://www.cubadebate.cu/noticias/2018/02/19/muerte-de-sylvester-stallone-causa-revuelo-en-las-redes-sociales/#respond" title="Comentarios en Falsa muerte de Sylvester Stallone causa revuelo en las redes" >84</a></span></div></li><li class="nombre_entrada"><a href="http://www.cubadebate.cu/especiales/2018/02/18/como-es-posible-la-florida-celebra-feria-de-armas-tres-dias-despues-de-la-masacre-en-parkland/">¿Cómo es posible? La Florida celebra Feria de Armas tres días después de la masacre en Parkland</a><div class="meta">18 febrero 2018<span class="separator">|</span><span class="comment_count"><a href="http://www.cubadebate.cu/especiales/2018/02/18/como-es-posible-la-florida-celebra-feria-de-armas-tres-dias-despues-de-la-masacre-en-parkland/#respond" title="Comentarios en ¿Cómo es posible? La Florida celebra Feria de Armas tres días después de la masacre en Parkland" >77</a></span></div></li><li class="nombre_entrada"><a href="http://www.cubadebate.cu/especiales/2018/02/18/noam-chomsky-explica-que-esconden-las-ayudas-humanitarias/">Noam Chomsky explica qué esconden las &#8220;Ayudas Humanitarias”</a><div class="meta">18 febrero 2018<span class="separator">|</span><span class="comment_count"><a href="http://www.cubadebate.cu/especiales/2018/02/18/noam-chomsky-explica-que-esconden-las-ayudas-humanitarias/#respond" title="Comentarios en Noam Chomsky explica qué esconden las &#8220;Ayudas Humanitarias”" >12</a></span></div></li></ul></div></section><section class="estadistica_comments consumo_last_section"><header class="title_wrapper"><span class="icon_class"></span><h2 class="widget-title seccion_title" title="Más Comentadas en los últimos 30 días"><a href="http://www.cubadebate.cu/internet/mas-comentadas/" title="Más comentadas">Más Comentadas</a></h2></header><a class="item_separator"></a><div class="estadisticas_grupo last_grupo"><ul><li class="nombre_entrada"><a href="http://www.cubadebate.cu/noticias/2018/03/05/pon-a-prueba-tu-mente-el-99-no-puede-resolver-este-acertijo/">Pon a prueba tu mente con el acertijo sobre hexágonos, plátanos y relojes</a><div class="meta">05 marzo 2018<span class="separator">|</span><span class="comment_count"><a href="http://www.cubadebate.cu/noticias/2018/03/05/pon-a-prueba-tu-mente-el-99-no-puede-resolver-este-acertijo/#respond" title="Comentarios en Pon a prueba tu mente con el acertijo sobre hexágonos, plátanos y relojes" >1835</a></span></div></li><li class="nombre_entrada"><a href="http://www.cubadebate.cu/especiales/2018/02/26/corrupcion-peligro-que-nos-afecta-a-todos/">Corrupción: Peligro que nos afecta a todos</a><div class="meta">26 febrero 2018<span class="separator">|</span><span class="comment_count"><a href="http://www.cubadebate.cu/especiales/2018/02/26/corrupcion-peligro-que-nos-afecta-a-todos/#respond" title="Comentarios en Corrupción: Peligro que nos afecta a todos" >347</a></span></div></li><li class="nombre_entrada"><a href="http://www.cubadebate.cu/opinion/2018/03/02/lo-que-le-gusta-a-los-menores/">Lo que le gusta a los ¿menores?</a><div class="meta">02 marzo 2018<span class="separator">|</span><span class="comment_count"><a href="http://www.cubadebate.cu/opinion/2018/03/02/lo-que-le-gusta-a-los-menores/#respond" title="Comentarios en Lo que le gusta a los ¿menores?" >290</a></span></div></li><li class="nombre_entrada"><a href="http://www.cubadebate.cu/noticias/2018/02/28/foro-debate-la-inversion-extranjera-en-cuba/">Foro Debate: La Inversión Extranjera en Cuba</a><div class="meta">28 febrero 2018<span class="separator">|</span><span class="comment_count"><a href="http://www.cubadebate.cu/noticias/2018/02/28/foro-debate-la-inversion-extranjera-en-cuba/#respond" title="Comentarios en Foro Debate: La Inversión Extranjera en Cuba" >214</a></span></div></li><li class="nombre_entrada"><a href="http://www.cubadebate.cu/noticias/2018/03/07/fracaso-provocacion-anticubana/">Fracasó provocación anticubana</a><div class="meta">07 marzo 2018<span class="separator">|</span><span class="comment_count"><a href="http://www.cubadebate.cu/noticias/2018/03/07/fracaso-provocacion-anticubana/#respond" title="Comentarios en Fracasó provocación anticubana" >212</a></span></div></li><li class="nombre_entrada"><a href="http://www.cubadebate.cu/especiales/2018/03/14/transporte-en-la-capital-los-p-se-quedan-sin-choferes/">Transporte en la capital: ¿Los P se quedan sin choferes?</a><div class="meta">14 marzo 2018<span class="separator">|</span><span class="comment_count"><a href="http://www.cubadebate.cu/especiales/2018/03/14/transporte-en-la-capital-los-p-se-quedan-sin-choferes/#respond" title="Comentarios en Transporte en la capital: ¿Los P se quedan sin choferes?" >208</a></span></div></li><li class="nombre_entrada"><a href="http://www.cubadebate.cu/especiales/2018/02/27/cuba-revelaciones-sobre-la-adulteracion-de-embutidos/">Cuba: Revelaciones sobre la adulteración de embutidos</a><div class="meta">27 febrero 2018<span class="separator">|</span><span class="comment_count"><a href="http://www.cubadebate.cu/especiales/2018/02/27/cuba-revelaciones-sobre-la-adulteracion-de-embutidos/#respond" title="Comentarios en Cuba: Revelaciones sobre la adulteración de embutidos" >190</a></span></div></li><li class="nombre_entrada"><a href="http://www.cubadebate.cu/noticias/2018/03/16/apagon-parcial-de-la-tv-analogica-en-cuba-comenzara-a-partir-de-julio/">Apagón parcial de la TV analógica en Cuba comenzará a partir de julio</a><div class="meta">16 marzo 2018<span class="separator">|</span><span class="comment_count"><a href="http://www.cubadebate.cu/noticias/2018/03/16/apagon-parcial-de-la-tv-analogica-en-cuba-comenzara-a-partir-de-julio/#respond" title="Comentarios en Apagón parcial de la TV analógica en Cuba comenzará a partir de julio" >173</a></span></div></li><li class="nombre_entrada"><a href="http://www.cubadebate.cu/especiales/2018/02/22/la-voz-de-un-pais-la-voz-de-una-revolucion-video/">La voz de un país, la voz de una Revolución (+ Video)</a><div class="meta">22 febrero 2018<span class="separator">|</span><span class="comment_count"><a href="http://www.cubadebate.cu/especiales/2018/02/22/la-voz-de-un-pais-la-voz-de-una-revolucion-video/#respond" title="Comentarios en La voz de un país, la voz de una Revolución (+ Video)" >161</a></span></div></li><li class="nombre_entrada"><a href="http://www.cubadebate.cu/especiales/2018/03/09/regueton-gusto-o-intereses/">Reguetón: ¿Gusto o intereses?</a><div class="meta">09 marzo 2018<span class="separator">|</span><span class="comment_count"><a href="http://www.cubadebate.cu/especiales/2018/03/09/regueton-gusto-o-intereses/#respond" title="Comentarios en Reguetón: ¿Gusto o intereses?" >159</a></span></div></li></ul></section></aside>
<!-- sidebar cached 9ae26f13dc509808bb156a402876c7a6 --><!--</div>--></section>
<footer class="main-footer">
    <section id="footer_menu">
        <section id="footer_left">
            <header>Sitios de Cubadebate</header>
            <nav id="wrapper_menu_blogs_footer" class="menu"><ul id="menu-sitios-cubadebate" class="menu"><li id="menu-item-380109" class="cden menu-item menu-item-type-custom menu-item-object-custom menu-item-380109"><a title="Cubadebate Inglés (english)" target="_blank" href="http://en.cubadebate.cu">Cubadebate Inglés</a></li>
<li id="menu-item-380111" class="cdit menu-item menu-item-type-custom menu-item-object-custom menu-item-380111"><a target="_blank" href="http://it.cubadebate.cu">Cubadebate Italiano</a></li>
<li id="menu-item-380113" class="cdfr menu-item menu-item-type-custom menu-item-object-custom menu-item-380113"><a target="_blank" href="http://fr.cubadebate.cu">Cubadebate Francés</a></li>
<li id="menu-item-380115" class="cdde menu-item menu-item-type-custom menu-item-object-custom menu-item-380115"><a target="_blank" href="http://de.cubadebate.cu">Cubadebate Alemán</a></li>
<li id="menu-item-380117" class="cdru menu-item menu-item-type-custom menu-item-object-custom menu-item-380117"><a target="_blank" href="http://ru.cubadebate.cu">Cubadebate Ruso</a></li>
<li id="menu-item-380119" class="cdpt menu-item menu-item-type-custom menu-item-object-custom menu-item-380119"><a target="_blank" href="http://pt.cubadebate.cu">Cubadebate Portugués</a></li>
<li id="menu-item-380121" class="cdar menu-item menu-item-type-custom menu-item-object-custom menu-item-380121"><a target="_blank" href="http://ar.cubadebate.cu">Cubadebate Árabe</a></li>
<li id="menu-item-380123" class="mesaredonda menu-item menu-item-type-custom menu-item-object-custom menu-item-380123"><a target="_blank" href="http://mesaredonda.cubadebate.cu">Mesa Redonda</a></li>
<li id="menu-item-380125" class="razones menu-item menu-item-type-custom menu-item-object-custom menu-item-380125"><a target="_blank" href="http://razonesdecuba.cubadebate.cu">Razones de Cuba</a></li>
</ul></nav>
<!-- menu cached ff8cf93e1f9d9914e22e0db7c2929dd4 -->        </section>
        <section id="footer_right">
            <header>Cubadebate</header>
            <nav id="wrapper_menu_extra_footer" class="menu"><ul id="menu-enlaces" class="menu"><li id="menu-item-380147" class="menu-item menu-item-type-post_type menu-item-object-page menu-item-380147"><a href="/editores/">Consejo Editorial</a></li>
<li id="menu-item-380151" class="menu-item menu-item-type-post_type menu-item-object-page menu-item-380151"><a href="/editores/terminos-condiciones/">Términos y Condiciones</a></li>
<li id="menu-item-380153" class="menu-item menu-item-type-post_type menu-item-object-page menu-item-380153"><a href="/editores/webmasters/">Para webmasters y desarrolladores</a></li>
<li id="menu-item-380155" class="menu-item menu-item-type-custom menu-item-object-custom menu-item-380155"><a href="/editores/preguntas-frecuentes/#enlaces">Sugerencias para referenciar www.cubadebate.cu</a></li>
</ul></nav>
<!-- menu cached 9a0a9f0a31dd37e78d2215f07e6ee083 -->        </section>
    </section>
    <section id="footer_bottom">
        <ul>
            <li>Cubadebate. Contra el Terrorismo Medi&aacute;tico</li>
            <li class="separator">|</li>
            <li>C&iacute;rculo de Periodistas Cubanos contra el Terrorismo</li>
            <li class="separator">|</li>
            <li>&copy; 2014, <a href="http://www.uci.cu" title="Sitio Web de la Universidad de las Ciencias Inform&aacute;ticas" target="_blank">UCI</a>, La Habana, Cuba</li>
        </ul>
    </section>
    <script type='text/javascript'>
/* <![CDATA[ */
var snb_ajax_object = {"ajax_url":"http:\/\/www.cubadebate.cu\/wp-admin\/admin-ajax.php"};
/* ]]> */
</script>
<script type='text/javascript' src='http://www.cubadebate.cu/wp-content/plugins/bwp-minify/min/?f=wp-includes/js/jquery/jquery-migrate.min.js,wp-content/plugins/fancybox-for-wordpress/fancybox/jquery.fancybox.js,wp-content/plugins/fancybox-for-wordpress/js/jquery.mousewheel.pack.js,wp-includes/js/jquery/ui/core.min.js,wp-includes/js/jquery/ui/widget.min.js,wp-includes/js/jquery/ui/tabs.min.js,wp-content/plugins/serie-nacional-beisbol/js/widget-snb.js&#038;ver=3.95'></script>
<script type='text/javascript' src='http://www.cubadebate.cu/wp-content/plugins/portada/js/cycle2/build/jquery.cycle2.min.js?ver=4.9.4'></script>
<script type='text/javascript'>
/* <![CDATA[ */
var fps_ajax_object = {"ajax_url":"http:\/\/www.cubadebate.cu\/wp-admin\/admin-ajax.php"};
/* ]]> */
</script>
<script type='text/javascript'>
/* <![CDATA[ */
var mam_ajax_object = {"ajax_url":"http:\/\/www.cubadebate.cu\/wp-admin\/admin-ajax.php"};
/* ]]> */
</script>
<script type='text/javascript'>
/* <![CDATA[ */
var pollsL10n = {"ajax_url":"http:\/\/www.cubadebate.cu\/wp-admin\/admin-ajax.php?plugin=wppolls","text_wait":"Tu \u00faltima petici\u00f3n est\u00e1 a\u00fan proces\u00e1ndose. Por favor, espera un momento \u2026","text_valid":"Por favor, elige una respuesta v\u00e1lida para la encuesta.","text_multiple":"N\u00famero m\u00e1ximo de opciones permitidas:","show_loading":"1","show_fading":"1"};
/* ]]> */
</script>
<script type='text/javascript'>
/* <![CDATA[ */
var menuArgs = null;menuArgs = {"action":"get_posts_from_category","ajaxurl":"http:\/\/www.cubadebate.cu\/wp-admin\/admin-ajax.php","loaderSrc":"http:\/\/www.cubadebate.cu\/wp-content\/themes\/cd2.1\/images\/loading_menu.gif"};
/* ]]> */
</script>
<script type='text/javascript' src='http://www.cubadebate.cu/wp-content/plugins/bwp-minify/min/?f=wp-content/plugins/wp-anexos/js/scripts.js,wp-content/plugins/wp-fidel-siempre/js/scripts.js,wp-content/plugins/wp-minuto-minuto/js/scripts.js,wp-content/plugins/wp-polls/polls-js.js,wp-content/themes/cd2.1/scripts/select2/select2.min.js,wp-content/themes/cd2.1/scripts/main.js,wp-includes/js/wp-embed.min.js&#038;ver=3.95'></script>
<script type='text/javascript' src='http://www.cubadebate.cu/wp-content/themes/cd2.1/scripts/modernizr.js?ver=4.9.4'></script>

<!-- Fancybox for WordPress -->
<script type="text/javascript">
jQuery(function(){

jQuery.fn.getTitle = function() { // Copy the title of every IMG tag and add it to its parent A so that fancybox can show titles
	var arr = jQuery("a.fancybox");
	jQuery.each(arr, function() {
		var title = jQuery(this).children("img").attr("title");
		jQuery(this).attr('title',title);
	})
}

// Supported file extensions
var thumbnails = jQuery("a:has(img)").not(".nolightbox").filter( function() { return /\.(jpe?g|png|gif|bmp)$/i.test(jQuery(this).attr('href')) });

thumbnails.addClass("fancybox").attr("rel","fancybox").getTitle();
jQuery("a.fancybox").fancybox({
	'cyclic': true,
	'autoScale': true,
	'padding': 10,
	'opacity': true,
	'speedIn': 500,
	'speedOut': 500,
	'changeSpeed': 300,
	'overlayShow': true,
	'overlayOpacity': "0.3",
	'overlayColor': "#666666",
	'titleShow': true,
	'titlePosition': 'over',
	'enableEscapeButton': true,
	'showCloseButton': true,
	'showNavArrows': true,
	'hideOnOverlayClick': true,
	'hideOnContentClick': false,
	'width': 560,
	'height': 340,
	'transitionIn': "elastic",
	'transitionOut': "elastic",
	'centerOnScroll': true
});


})
</script>
<!-- END Fancybox for WordPress -->
</footer>

    </body>
</html>
<!-- Dynamic page generated in 3.948 seconds. -->
<!-- Cached page generated by WP-Super-Cache on 2018-03-19 23:27:51 -->

<!-- super cache -->